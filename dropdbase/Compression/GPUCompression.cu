#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "dropdbase/Compression/feathergpu/fl/containers.cuh"
#include "dropdbase/Compression/feathergpu/fl/default.cuh"
#include "GPUCompression.h"
#include "dropdbase/QueryEngine/Context.h" 
#include "dropdbase/QueryEngine/QueryEngineError.h" 
#include <memory>
#include <string>
#include <limits>
#include "dropdbase/Types/ComplexPolygon.pb.h"
#include "dropdbase/Types/Point.pb.h"
#include "dropdbase/QueryEngine/QueryEngineError.h"

/// <summary>
/// Compresses input data and fills output vector with compressed data
/// </summary>
/// <param name="CWARP_SIZE">Warp size</param>
/// <param name="hostUncompressed">Poiter to uncompressed data stored in host memory</param>
/// <param name="uncompressedElementsCount">Number of elements of uncompressed data</param>
/// <param name="hostCompressed">Compressed data vector in host memory</param>
/// <param name="compressedElementsCount">Number of elements of compressed data</param>
/// <param name="minValue">Minimum value of uncompressed data</param>
/// <param name="maxValue">Maximum value of uncompressed data</param>
/// <returns>Output parameter representing result of compression</returns>
template<typename T>
bool compressAAFL(const int CWARP_SIZE, T* const hostUncompressed, int64_t uncompressedElementsCount, std::vector<T>& hostCompressed, int64_t& compressedElementsCount, T minValue, T maxValue)
{
	// Sets offset for data transformation (subtracting minimal value), it checkes if it is possible to transform within range of type T
	T offset = minValue;
	if (minValue < 0 && maxValue > 0)
	{
		if (std::numeric_limits<T>::max() - maxValue < -minValue)
			offset = 0;
	}

	int64_t uncompressedDataSize = uncompressedElementsCount * sizeof(T); // size in bytes
	int64_t compressionBlocksCount = (uncompressedDataSize + (sizeof(T) * CWARP_SIZE) - 1) / (sizeof(T) * CWARP_SIZE);

	// Device pointers to compression data and metadata
	T *deviceUncompressed;
	T *deviceCompressed;
	unsigned char *deviceBitLength;
	unsigned long *devicePositionId;
	unsigned long *deviceCompressedElementsCount;

	// Device allocations for compression
	auto& cudaAllocator = Context::getInstance().GetAllocatorForCurrentDevice();
	deviceUncompressed = reinterpret_cast<T*>(cudaAllocator.allocate(uncompressedDataSize));
	deviceCompressed = reinterpret_cast<T*>(cudaAllocator.allocate(uncompressedDataSize)); // first we do not know what will be the size, therfore data_size	
	deviceBitLength = reinterpret_cast<unsigned char*>(cudaAllocator.allocate(compressionBlocksCount * sizeof(unsigned char)));
	devicePositionId = reinterpret_cast<unsigned long*>(cudaAllocator.allocate(compressionBlocksCount * sizeof(unsigned long)));
	deviceCompressedElementsCount = reinterpret_cast<unsigned long*>(cudaAllocator.allocate(sizeof(unsigned long)));
		
	// Copy data CPU->GPU
	hipMemcpy(deviceUncompressed, hostUncompressed, uncompressedDataSize, hipMemcpyHostToDevice);
	QueryEngineError::setCudaError(hipGetLastError());

	// Set before compression
	hipMemset(deviceCompressed, 0, uncompressedDataSize);
	hipMemset(deviceCompressedElementsCount, 0, sizeof(unsigned long));
	hipMemset(deviceBitLength, 0, compressionBlocksCount * sizeof(unsigned char));
	hipMemset(devicePositionId, 0, compressionBlocksCount * sizeof(unsigned long));
	QueryEngineError::setCudaError(hipGetLastError());

	// Compression
	container_uncompressed<T> udata = { deviceUncompressed, uncompressedElementsCount };
	container_aafl<T> cdata = { deviceCompressed, uncompressedElementsCount, deviceBitLength, devicePositionId, deviceCompressedElementsCount, offset };
	gpu_fl_naive_launcher_compression<T, 32, container_aafl<T>>::compress(udata, cdata);
	QueryEngineError::setCudaError(hipGetLastError());

	// Gets compression elements (values only without meta data) count into RAM
	unsigned long compressedValuesCount;
	hipMemcpy(&compressedValuesCount, deviceCompressedElementsCount, sizeof(unsigned long), hipMemcpyDeviceToHost);
	int64_t compressedDataSize = (compressedValuesCount) * sizeof(T);

	// Total compression data size (values and meta data)
	unsigned long compressedDataSizeTotal = 
		compressedDataSize + 
		std::max(sizeof(unsigned long) * compressionBlocksCount, sizeof(T)) +
		std::max(sizeof(unsigned char) * compressionBlocksCount, sizeof(T)) +
		(sizeof(int64_t) * 3);
	compressedElementsCount = compressedDataSizeTotal / sizeof(T);

	bool result = false;
	// Does compression make sense?
	if (compressedElementsCount < uncompressedElementsCount)
	{
		// All data are coded into single array of type T
		// We determine positions of partials arrays (meta data) in one T array
		int64_t sizes[3] = { uncompressedElementsCount , compressedElementsCount, compressionBlocksCount };
		T* codedSizes = reinterpret_cast<T*>(sizes);

		int positionCodedDataPositionId = (sizeof(int64_t) / (float)sizeof(T) * 3);
		int positionCodedDataBitLength = positionCodedDataPositionId + std::max((int)(sizeof(unsigned long) / (float)sizeof(T) * compressionBlocksCount), 1);
		int positionHostOut = positionCodedDataBitLength + std::max((int)(sizeof(char) / (float)sizeof(T) * compressionBlocksCount), 1);

		hostCompressed.reserve(compressedDataSizeTotal / sizeof(T));
		
		// Resulting pointer to host compressed data
		std::unique_ptr<T[]> data = std::unique_ptr<T[]>(new T[(compressedDataSizeTotal / sizeof(T))]);

		// Copy all compression data GPU -> CPU
		std::move(codedSizes, codedSizes + (int)(sizeof(int64_t) / (float)sizeof(T) * 3), data.get());
		hipMemcpy(data.get() + positionCodedDataPositionId, devicePositionId, compressionBlocksCount * sizeof(unsigned long), hipMemcpyDeviceToHost);
		hipMemcpy(data.get() + positionCodedDataBitLength, deviceBitLength, compressionBlocksCount * sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipMemcpy(data.get() + positionHostOut, deviceCompressed, compressedDataSize, hipMemcpyDeviceToHost);

		// Assignment into output parameter
		hostCompressed.assign(data.get(), data.get() + compressedElementsCount);

		result = true;
	}
	else
	{
		result = false;
	}

	// Clean up device allocations
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(deviceUncompressed), uncompressedDataSize);
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(deviceCompressed), uncompressedDataSize);
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(deviceBitLength), compressionBlocksCount * sizeof(unsigned char));
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(devicePositionId), compressionBlocksCount * sizeof(unsigned long));
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(deviceCompressedElementsCount), sizeof(long));
	QueryEngineError::setCudaError(hipGetLastError());

	return result;
}





template<>
bool CompressionGPU::compressDataAAFL<int32_t>(int32_t* const host_uncompressed, int64_t size, std::vector<int32_t>& host_compressed, int64_t& compressed_size, int32_t min, int32_t max)
{
	return compressAAFL(32, host_uncompressed, size, host_compressed, compressed_size, min, max);
}

template<>
bool CompressionGPU::compressDataAAFL<int64_t>(int64_t* const host_uncompressed, int64_t size, std::vector<int64_t>& host_compressed, int64_t& compressed_size, int64_t min, int64_t max)
{
	return compressAAFL(32, host_uncompressed, size, host_compressed, compressed_size, min, max);
}

template<>
bool CompressionGPU::compressDataAAFL<int8_t>(int8_t* const host_uncompressed, int64_t size, std::vector<int8_t>& host_compressed, int64_t& compressed_size, int8_t min, int8_t max)
{
	return compressAAFL(32, host_uncompressed, size, host_compressed, compressed_size, min, max);
}

template<>
bool CompressionGPU::compressDataAAFL<double>(double* const host_uncompressed, int64_t size, std::vector<double>& host_compressed, int64_t& compressed_size, double min, double max)
{	
	return 0;
}

template<>
bool CompressionGPU::compressDataAAFL<float>(float* const host_uncompressed, int64_t size, std::vector<float>& host_compressed, int64_t& compressed_size, float min, float max)
{
	if ((min >= 0 && max >= 0) || (min <= 0 && max <= 0))
	{
		int32_t * host_uncompressed_int32 = reinterpret_cast<int32_t*>(host_uncompressed);
		std::vector<int32_t> host_compressed_int32;
		int32_t min_int32 = *(reinterpret_cast<int32_t*>(&min));
		int32_t max_int32 = *(reinterpret_cast<int32_t*>(&max));
		if (min_int32 > max_int32)
			std::swap(min_int32, max_int32);
		bool compressed = compressAAFL(32, host_uncompressed_int32, size, host_compressed_int32, compressed_size, min_int32, max_int32);
		if (compressed) {
			const int32_t *p_host_compressed_int32 = host_compressed_int32.data();
			host_compressed.reserve(compressed_size);
			const float *p_host_compressed = reinterpret_cast<const float *>(p_host_compressed_int32);
			host_compressed.assign(p_host_compressed, p_host_compressed + compressed_size);
			return true;
		}
	}
	
	return 0;
}

template<>
bool CompressionGPU::compressDataAAFL<std::string>(std::string* const host_uncompressed, int64_t size, std::vector<std::string>& host_compressed, int64_t& compressed_size, std::string min, std::string max)
{
	return 0;
}


template<>
bool CompressionGPU::compressDataAAFL<ColmnarDB::Types::ComplexPolygon>(ColmnarDB::Types::ComplexPolygon* const host_uncompressed, int64_t size, std::vector<ColmnarDB::Types::ComplexPolygon>& host_compressed, int64_t& compressed_size, ColmnarDB::Types::ComplexPolygon min, ColmnarDB::Types::ComplexPolygon max)
{
	return 0;
}

template<>
bool CompressionGPU::compressDataAAFL<ColmnarDB::Types::Point>(ColmnarDB::Types::Point* const host_uncompressed, int64_t size, std::vector<ColmnarDB::Types::Point>& host_compressed, int64_t& compressed_size, ColmnarDB::Types::Point min, ColmnarDB::Types::Point max)
{
	return 0;
}




/// <summary>
/// Decompresses input data and fills output vector with decompressed data
/// </summary>
/// <param name="CWARP_SIZE">Warp size</param>
/// <param name="hostCompressed">Pointer to compressed data stored in host memory</param>
/// <param name="compressedElementsCount">Number of elements of compressed data</param>
/// <param name="hostUncompressed">Uncompressed data vector in host memory</param>
/// <param name="uncompressedElementsCount">Number of elements of uncompressed data</param>
/// <param name="minValue">Minimum value of uncompressed data</param>
/// <param name="maxValue">Maximum value of uncompressed data</param>
/// <returns>Value representing result of decompression</returns>
template<typename T>
bool decompressAAFL(const int CWARP_SIZE, T* const hostCompressed, int64_t compressedElementsCount, std::vector<T>& hostUncompressed, int64_t &uncompressedElementsCount, T minValue, T maxValue)
{
	T offset = minValue;

	uncompressedElementsCount = reinterpret_cast<int64_t*>(hostCompressed)[0];
	compressedElementsCount = reinterpret_cast<int64_t*>(hostCompressed)[1];
	int64_t compressionBlocksCount = reinterpret_cast<int64_t*>(hostCompressed)[2];

	int64_t uncompressedDataSize = uncompressedElementsCount * sizeof(T); // size in bytes
	int64_t compressedDataSize = compressedElementsCount * sizeof(T); // size in bytes
	
	T *hostCompressedValuesData; // data of values only without meta data
	
	// Device pointers to compression data and metadata
	T *deviceUncompressed;
	T *deviceCompressed;
	unsigned char *deviceBitLength;
	unsigned long *devicePositionId;

	// Device allocations for decompression
	auto& cudaAllocator = Context::getInstance().GetAllocatorForCurrentDevice();
	deviceUncompressed = reinterpret_cast<T*>(cudaAllocator.allocate(uncompressedDataSize));
	deviceCompressed = reinterpret_cast<T*>(cudaAllocator.allocate(compressedDataSize));
	deviceBitLength = reinterpret_cast<unsigned char*>(cudaAllocator.allocate(compressionBlocksCount * sizeof(unsigned char)));
	devicePositionId = reinterpret_cast<unsigned long*>(cudaAllocator.allocate(compressionBlocksCount * sizeof(unsigned long)));
	QueryEngineError::setCudaError(hipGetLastError());

	// Decoding single array of type T into separate arrays (of compression meta data)
	int positionCodedDataPositionId = (sizeof(int64_t) / (float)sizeof(T) * 3);
	int positionCodedDataBitLength = positionCodedDataPositionId + std::max((int)(sizeof(unsigned long) / (float)sizeof(T) * compressionBlocksCount), 1);
	int positionHostOut = positionCodedDataBitLength + std::max((int)(sizeof(char) / (float)sizeof(T) * compressionBlocksCount), 1);

	unsigned char *hostPositionId = reinterpret_cast<unsigned long*>(&hostCompressed[positionCodedDataPositionId]);
	unsigned long *hostBitLength = reinterpret_cast<unsigned char*>(&hostCompressed[positionCodedDataBitLength]);
	hostCompressedValuesData = &hostCompressed[positionHostOut];

	// Copy data CPU->GPU
	hipMemcpy(deviceCompressed, hostCompressedValuesData, compressedDataSize - (positionHostOut * sizeof(T)), hipMemcpyHostToDevice); // from compression size we need to subtract leading bytes with meta info
	hipMemcpy(devicePositionId, hostPositionId, compressionBlocksCount * sizeof(unsigned long), hipMemcpyHostToDevice);
	hipMemcpy(deviceBitLength, hostBitLength, compressionBlocksCount * sizeof(unsigned char), hipMemcpyHostToDevice);
	QueryEngineError::setCudaError(hipGetLastError());

	// Decompression
	container_uncompressed<T> udata = { deviceUncompressed, uncompressedElementsCount };
	container_aafl<T> cdata = { deviceCompressed, uncompressedElementsCount, deviceBitLength, devicePositionId, NULL, offset };
	gpu_fl_naive_launcher_decompression<T, 32, container_aafl<T>>::decompress(cdata, udata);
	QueryEngineError::setCudaError(hipGetLastError());
	
	// Copy result GPU->CPU into resulting pointer
	std::unique_ptr<T[]> data = std::unique_ptr<T[]>(new T[uncompressedDataSize / sizeof(T)]);
	hipMemcpy(data.get(), deviceUncompressed, uncompressedDataSize, hipMemcpyDeviceToHost);
	QueryEngineError::setCudaError(hipGetLastError());

	// Clean up device allocations
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(deviceUncompressed), uncompressedDataSize);
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(deviceCompressed), uncompressedDataSize);
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(deviceBitLength), compressionBlocksCount * sizeof(unsigned char));
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(devicePositionId), compressionBlocksCount * sizeof(unsigned long));
	QueryEngineError::setCudaError(hipGetLastError());

	// Assignment into output parameter
	hostUncompressed.reserve(uncompressedElementsCount);
	hostUncompressed.assign(data.get(), data.get() + uncompressedElementsCount);

	return true;
}

template<>
bool CompressionGPU::decompressDataAAFL<int32_t>(int32_t* const host_compressed, int64_t compressed_size, std::vector<int32_t>& host_uncompressed, int64_t &size, int32_t min, int32_t max)
{
	return decompressAAFL(32, host_compressed, compressed_size, host_uncompressed, size, min, max);	
}

template<>
bool CompressionGPU::decompressDataAAFL<int64_t>(int64_t* const host_compressed, int64_t compressed_size, std::vector<int64_t>& host_uncompressed, int64_t &size, int64_t min, int64_t max)
{
	return decompressAAFL(32, host_compressed, compressed_size, host_uncompressed, size, min, max);
}

template<>
bool CompressionGPU::decompressDataAAFL<int8_t>(int8_t* const host_compressed, int64_t compressed_size, std::vector<int8_t>& host_uncompressed, int64_t &size, int8_t min, int8_t max)
{
	return decompressAAFL(32, host_compressed, compressed_size, host_uncompressed, size, min, max);
}

template<>
bool CompressionGPU::decompressDataAAFL<float>(float* const host_compressed, int64_t compressed_size, std::vector<float>& host_uncompressed, int64_t &size, float min, float max)
{
	if ((min > 0 && max > 0) || (min < 0 && max < 0))
	{
		int32_t * host_compressed_int32 = reinterpret_cast<int32_t*>(host_compressed);
		std::vector<int32_t> host_uncompressed_int32;
		int32_t min_int32 = *(reinterpret_cast<int32_t*>(&min));
		int32_t max_int32 = *(reinterpret_cast<int32_t*>(&max));
		if (min_int32 > max_int32)
			std::swap(min_int32, max_int32);
		bool decompressed = decompressAAFL(32, host_compressed_int32, compressed_size, host_uncompressed_int32, size, min_int32, max_int32);
		if (decompressed) {
			const int32_t *p_host_uncompressed_int32 = host_uncompressed_int32.data();
			host_uncompressed.reserve(size);
			const float *p_host_uncompressed = reinterpret_cast<const float *>(p_host_uncompressed_int32);
			host_uncompressed.assign(p_host_uncompressed, p_host_uncompressed + size);
			return true;
		}
	}
	return false;
}

template<>
bool CompressionGPU::decompressDataAAFL<double>(double* const host_compressed, int64_t compressed_size, std::vector<double>& host_uncompressed, int64_t &size, double min, double max)
{
	return false;
}

template<>
bool CompressionGPU::decompressDataAAFL<ColmnarDB::Types::ComplexPolygon>(ColmnarDB::Types::ComplexPolygon* const host_compressed, int64_t compressed_size, std::vector<ColmnarDB::Types::ComplexPolygon>& host_uncompressed, int64_t &size, ColmnarDB::Types::ComplexPolygon min, ColmnarDB::Types::ComplexPolygon max)
{
	return false;
}

template<>
bool CompressionGPU::decompressDataAAFL<ColmnarDB::Types::Point>(ColmnarDB::Types::Point* const host_compressed, int64_t compressed_size, std::vector<ColmnarDB::Types::Point>& host_uncompressed, int64_t &size, ColmnarDB::Types::Point min, ColmnarDB::Types::Point max)
{
	return false;
}






template<typename T>
bool decompressAAFLOnDevice(const int CWARP_SIZE, T* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, T* const device_uncompressed, T min, T max)
{
	T offset = min;

	
	T *device_compressed_data;

	unsigned char *device_bit_length;
	unsigned long *device_position_id;
		
	int coded_data_position_id_start = (sizeof(int64_t) / (float)sizeof(T) * 3);
	int coded_data_bit_length_start = coded_data_position_id_start + std::max((int)(sizeof(unsigned long) / (float)sizeof(T) * compression_blocks_count), 1);
	int device_out_start = coded_data_bit_length_start + std::max((int)(sizeof(char) / (float)sizeof(T) * compression_blocks_count), 1);

	device_position_id = reinterpret_cast<unsigned long*>(&device_compressed[coded_data_position_id_start]);
	device_bit_length = reinterpret_cast<unsigned char*>(&device_compressed[coded_data_bit_length_start]);
	device_compressed_data = &device_compressed[device_out_start];

	
	container_uncompressed<T> udata = { device_uncompressed, data_size };
	container_aafl<T> cdata = { device_compressed_data, data_size, device_bit_length, device_position_id, NULL, offset };

	gpu_fl_naive_launcher_decompression<T, 32, container_aafl<T>>::decompress(cdata, udata);
	QueryEngineError::setCudaError(hipGetLastError());

	return true;
}


template<>
bool CompressionGPU::decompressDataAAFLOnDevice<int32_t>(int32_t* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, int32_t* const device_uncompressed, int32_t min, int32_t max)
{
	return decompressAAFLOnDevice(32, device_compressed, data_size, compression_data_size, compression_blocks_count, device_uncompressed, min, max);
}

template<>
bool CompressionGPU::decompressDataAAFLOnDevice<int64_t>(int64_t* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, int64_t* const device_uncompressed, int64_t min, int64_t max)
{
	return decompressAAFLOnDevice(32, device_compressed, data_size, compression_data_size, compression_blocks_count, device_uncompressed, min, max);
}


template<>
bool CompressionGPU::decompressDataAAFLOnDevice<int8_t>(int8_t* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, int8_t* const device_uncompressed, int8_t min, int8_t max)
{
	return decompressAAFLOnDevice(32, device_compressed, data_size, compression_data_size, compression_blocks_count, device_uncompressed, min, max);
}

template<>
bool CompressionGPU::decompressDataAAFLOnDevice<float>(float* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, float* const device_uncompressed, float min, float max)
{
	if ((min > 0 && max > 0) || (min < 0 && max < 0))
	{
		int32_t * device_compressed_int32 = reinterpret_cast<int32_t*>(device_compressed);
		int32_t * device_uncompressed_int32 = reinterpret_cast<int32_t*>(device_uncompressed);
		int32_t min_int32 = *(reinterpret_cast<int32_t*>(&min));
		int32_t max_int32 = *(reinterpret_cast<int32_t*>(&max));
		if (min_int32 > max_int32)
			std::swap(min_int32, max_int32);
		
		bool compressed = decompressAAFLOnDevice(32, device_compressed_int32, data_size, compression_data_size, compression_blocks_count, device_uncompressed_int32, min_int32, max_int32);
		return compressed;
	}
	return false;
}

template<>
bool CompressionGPU::decompressDataAAFLOnDevice<double>(double* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, double* const device_uncompressed, double min, double max)
{
	return false;
}

template<>
bool CompressionGPU::decompressDataAAFLOnDevice<ColmnarDB::Types::ComplexPolygon>(ColmnarDB::Types::ComplexPolygon* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, ColmnarDB::Types::ComplexPolygon* const device_uncompressed, ColmnarDB::Types::ComplexPolygon min, ColmnarDB::Types::ComplexPolygon max)
{
	return false;
}

template<>
bool CompressionGPU::decompressDataAAFLOnDevice<ColmnarDB::Types::Point>(ColmnarDB::Types::Point* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, ColmnarDB::Types::Point* const device_uncompressed, ColmnarDB::Types::Point min, ColmnarDB::Types::Point max)
{
	return false;
}