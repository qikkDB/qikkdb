#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "dropdbase/Compression/feathergpu/fl/containers.cuh"
#include "dropdbase/Compression/feathergpu/fl/default.cuh"
#include "GPUCompression.h"
#include "dropdbase/QueryEngine/Context.h" 
#include <memory>
#include <string>
#include <limits>
#include "dropdbase/Types/ComplexPolygon.pb.h"
#include "dropdbase/Types/Point.pb.h"
#include "dropdbase/QueryEngine/QueryEngineError.h"


template<typename T>
bool compressAAFL(const int CWARP_SIZE, T* const host_uncompressed, int64_t size, std::vector<T>& host_compressed, int64_t& compressed_size, T min, T max)
{
	T offset = min;
	if (min < 0)
	{
		if (std::numeric_limits<T>::max() - max < -min)
			offset = 0;
	}

	int64_t data_size = size * sizeof(T);
	int64_t compression_blocks_count = (data_size + (sizeof(T) * CWARP_SIZE) - 1) / (sizeof(T) * CWARP_SIZE);


	T *device_uncompressed;
	T *device_compressed;
	unsigned char *device_bit_length;
	unsigned long *device_position_id;
	unsigned long *device_compressed_size;

	//allocations
	auto& cudaAllocator = Context::getInstance().GetAllocatorForCurrentDevice();
	device_uncompressed = reinterpret_cast<T*>(cudaAllocator.allocate(data_size));
	device_compressed = reinterpret_cast<T*>(cudaAllocator.allocate(data_size)); // first we do not know what will be the size, therfore data_size
	
	device_bit_length = reinterpret_cast<unsigned char*>(cudaAllocator.allocate(compression_blocks_count * sizeof(unsigned char)));
	device_position_id = reinterpret_cast<unsigned long*>(cudaAllocator.allocate(compression_blocks_count * sizeof(unsigned long)));
	device_compressed_size = reinterpret_cast<unsigned long*>(cudaAllocator.allocate(sizeof(unsigned long)));
		
	//copy M->G
	hipMemcpy(device_uncompressed, host_uncompressed, data_size, hipMemcpyHostToDevice);

	// Clean up before compression
	hipMemset(device_compressed, 0, data_size);
	hipMemset(device_compressed_size, 0, sizeof(unsigned long));
	hipMemset(device_bit_length, 0, compression_blocks_count * sizeof(unsigned char));
	hipMemset(device_position_id, 0, compression_blocks_count * sizeof(unsigned long));

	container_uncompressed<T> udata = { device_uncompressed, size };
	container_aafl<T> cdata = { device_compressed, size, device_bit_length, device_position_id, device_compressed_size, offset };
	gpu_fl_naive_launcher_compression<T, 32, container_aafl<T>>::compress(udata, cdata);

	unsigned long host_compressed_size;
	hipMemcpy(&host_compressed_size, device_compressed_size, sizeof(unsigned long), hipMemcpyDeviceToHost);
	int64_t compressed_data_size = (host_compressed_size) * sizeof(T);

	// coding into single array
	unsigned long compressed_data_size_final = 
		compressed_data_size + 
		std::max(sizeof(unsigned long) * compression_blocks_count, sizeof(T)) +
		std::max(sizeof(unsigned char) * compression_blocks_count, sizeof(T)) +
		(sizeof(int64_t) * 3);
	compressed_size = compressed_data_size_final / sizeof(T);

	bool result = false;
	if (compressed_size < size)
	{
		int64_t sizes[3] = { data_size , compressed_data_size, compression_blocks_count };

		T* coded_sizes = reinterpret_cast<T*>(sizes);

		int coded_data_position_id_start = (sizeof(int64_t) / sizeof(T) * 3);
		int coded_data_bit_length_start = coded_data_position_id_start + std::max((int)(sizeof(unsigned long) / (float)sizeof(T) * compression_blocks_count), 1);
		int host_out_start = coded_data_bit_length_start + std::max((int)(sizeof(char) / (float)sizeof(T) * compression_blocks_count), 1);

		host_compressed.reserve(compressed_data_size_final / sizeof(T));
		
		std::unique_ptr<T[]> data = std::unique_ptr<T[]>(new T[(compressed_data_size_final / sizeof(T))]);

		std::move(coded_sizes, coded_sizes + (int)(sizeof(int64_t) / (float)sizeof(T) * 3), data.get());
		hipMemcpy(data.get() + coded_data_position_id_start, device_position_id, compression_blocks_count * sizeof(unsigned long), hipMemcpyDeviceToHost);
		hipMemcpy(data.get() + coded_data_bit_length_start, device_bit_length, compression_blocks_count * sizeof(unsigned char), hipMemcpyDeviceToHost);
		hipMemcpy(data.get() + host_out_start, device_compressed, compressed_data_size, hipMemcpyDeviceToHost);

		cudaAllocator.deallocate(reinterpret_cast<int8_t*>(device_uncompressed), data_size);
		cudaAllocator.deallocate(reinterpret_cast<int8_t*>(device_compressed), data_size);
		cudaAllocator.deallocate(reinterpret_cast<int8_t*>(device_bit_length), compression_blocks_count * sizeof(unsigned char));
		cudaAllocator.deallocate(reinterpret_cast<int8_t*>(device_position_id), compression_blocks_count * sizeof(unsigned long));
		cudaAllocator.deallocate(reinterpret_cast<int8_t*>(device_compressed_size), sizeof(long));

		host_compressed.assign(data.get(), data.get() + compressed_size);

		result = true;
	}
	else
	{
		result = false;
	}

	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(device_uncompressed), data_size);
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(device_compressed), data_size);
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(device_bit_length), compression_blocks_count * sizeof(unsigned char));
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(device_position_id), compression_blocks_count * sizeof(unsigned long));
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(device_compressed_size), sizeof(long));

	return result;
}





template<>
bool CompressionGPU::compressDataAAFL<int32_t>(int32_t* const host_uncompressed, int64_t size, std::vector<int32_t>& host_compressed, int64_t& compressed_size, int32_t min, int32_t max)
{
	return compressAAFL(32, host_uncompressed, size, host_compressed, compressed_size, min, max);
}

template<>
bool CompressionGPU::compressDataAAFL<int64_t>(int64_t* const host_uncompressed, int64_t size, std::vector<int64_t>& host_compressed, int64_t& compressed_size, int64_t min, int64_t max)
{
	return compressAAFL(32, host_uncompressed, size, host_compressed, compressed_size, min, max);
}

template<>
bool CompressionGPU::compressDataAAFL<int8_t>(int8_t* const host_uncompressed, int64_t size, std::vector<int8_t>& host_compressed, int64_t& compressed_size, int8_t min, int8_t max)
{
	return compressAAFL(32, host_uncompressed, size, host_compressed, compressed_size, min, max);
}

template<>
bool CompressionGPU::compressDataAAFL<double>(double* const host_uncompressed, int64_t size, std::vector<double>& host_compressed, int64_t& compressed_size, double min, double max)
{	
	return 0;
}

template<>
bool CompressionGPU::compressDataAAFL<float>(float* const host_uncompressed, int64_t size, std::vector<float>& host_compressed, int64_t& compressed_size, float min, float max)
{
	return 0;
}

template<>
bool CompressionGPU::compressDataAAFL<std::string>(std::string* const host_uncompressed, int64_t size, std::vector<std::string>& host_compressed, int64_t& compressed_size, std::string min, std::string max)
{
	return 0;
}


template<>
bool CompressionGPU::compressDataAAFL<ColmnarDB::Types::ComplexPolygon>(ColmnarDB::Types::ComplexPolygon* const host_uncompressed, int64_t size, std::vector<ColmnarDB::Types::ComplexPolygon>& host_compressed, int64_t& compressed_size, ColmnarDB::Types::ComplexPolygon min, ColmnarDB::Types::ComplexPolygon max)
{
	return 0;
}

template<>
bool CompressionGPU::compressDataAAFL<ColmnarDB::Types::Point>(ColmnarDB::Types::Point* const host_uncompressed, int64_t size, std::vector<ColmnarDB::Types::Point>& host_compressed, int64_t& compressed_size, ColmnarDB::Types::Point min, ColmnarDB::Types::Point max)
{
	return 0;
}





template<typename T>
bool decompressAAFL(const int CWARP_SIZE, T* const host_compressed, int64_t compressed_size, std::vector<T>& host_uncompressed, T min, T max)
{
	T offset = min;

	int64_t data_size = reinterpret_cast<int64_t*>(host_compressed)[0];
	int64_t compressed_data_size = reinterpret_cast<int64_t*>(host_compressed)[1];
	int64_t compression_blocks_count = reinterpret_cast<int64_t*>(host_compressed)[2];

	int64_t size = data_size / sizeof(T);

	unsigned char *host_bit_length;
	unsigned long *host_position_id;

	T *host_compressed_data;

	T *device_uncompressed;
	T *device_compressed;
	unsigned char *device_bit_length;
	unsigned long *device_position_id;

	//allocations
	auto& cudaAllocator = Context::getInstance().GetAllocatorForCurrentDevice();
	device_uncompressed = reinterpret_cast<T*>(cudaAllocator.allocate(data_size));
	device_compressed = reinterpret_cast<T*>(cudaAllocator.allocate(compressed_data_size));

	device_bit_length = reinterpret_cast<unsigned char*>(cudaAllocator.allocate(compression_blocks_count * sizeof(unsigned char)));
	device_position_id = reinterpret_cast<unsigned long*>(cudaAllocator.allocate(compression_blocks_count * sizeof(unsigned long)));
	
	int coded_data_position_id_start = (sizeof(int64_t) / (float)sizeof(T) * 3);
	int coded_data_bit_length_start = coded_data_position_id_start + (sizeof(unsigned long) / (float)sizeof(T) * compression_blocks_count);
	int host_out_start = coded_data_bit_length_start + (sizeof(char) / (float)sizeof(T) * compression_blocks_count);

	host_position_id = reinterpret_cast<unsigned long*>(&host_compressed[coded_data_position_id_start]);
	host_bit_length = reinterpret_cast<unsigned char*>(&host_compressed[coded_data_bit_length_start]);
	host_compressed_data = &host_compressed[host_out_start];

	//for (int i = 0; i < 10; i++) {
	//	printf("bit2 %d\n", host_bit_length[i]);
	//}

	hipMemcpy(device_compressed, host_compressed_data, compressed_data_size, hipMemcpyHostToDevice);
	hipMemcpy(device_position_id, host_position_id, compression_blocks_count * sizeof(unsigned long), hipMemcpyHostToDevice);
	hipMemcpy(device_bit_length, host_bit_length, compression_blocks_count * sizeof(unsigned char), hipMemcpyHostToDevice);

	container_uncompressed<T> udata = { device_uncompressed, size };
	container_aafl<T> cdata = { device_compressed, size, device_bit_length, device_position_id, NULL, offset };

	gpu_fl_naive_launcher_decompression<T, 32, container_aafl<T>>::decompress(cdata, udata);
	
	std::unique_ptr<T[]> data = std::unique_ptr<T[]>(new T[data_size / sizeof(T)]);
	hipMemcpy(data.get(), device_uncompressed, data_size, hipMemcpyDeviceToHost);
	
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(device_uncompressed), data_size);
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(device_compressed), data_size);
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(device_bit_length), compression_blocks_count * sizeof(unsigned char));
	cudaAllocator.deallocate(reinterpret_cast<int8_t*>(device_position_id), compression_blocks_count * sizeof(unsigned long));
	
	host_uncompressed.reserve(size);
	host_uncompressed.assign(data.get(), data.get() + size);

	return true;
}

template<>
bool CompressionGPU::decompressDataAAFL<int32_t>(int32_t* const host_compressed, int64_t compressed_size, std::vector<int32_t>& host_uncompressed, int32_t min, int32_t max)
{
	return decompressAAFL(32, host_compressed, compressed_size, host_uncompressed, min, max);	
}

template<>
bool CompressionGPU::decompressDataAAFL<int64_t>(int64_t* const host_compressed, int64_t compressed_size, std::vector<int64_t>& host_uncompressed, int64_t min, int64_t max)
{
	return decompressAAFL(32, host_compressed, compressed_size, host_uncompressed, min, max);
}

template<>
bool CompressionGPU::decompressDataAAFL<int8_t>(int8_t* const host_compressed, int64_t compressed_size, std::vector<int8_t>& host_uncompressed, int8_t min, int8_t max)
{
	return decompressAAFL(32, host_compressed, compressed_size, host_uncompressed, min, max);
}

template<>
bool CompressionGPU::decompressDataAAFL<float>(float* const host_compressed, int64_t compressed_size, std::vector<float>& host_uncompressed, float min, float max)
{
	return false;
}

template<>
bool CompressionGPU::decompressDataAAFL<double>(double* const host_compressed, int64_t compressed_size, std::vector<double>& host_uncompressed, double min, double max)
{
	return false;
}

template<>
bool CompressionGPU::decompressDataAAFL<ColmnarDB::Types::ComplexPolygon>(ColmnarDB::Types::ComplexPolygon* const host_compressed, int64_t compressed_size, std::vector<ColmnarDB::Types::ComplexPolygon>& host_uncompressed, ColmnarDB::Types::ComplexPolygon min, ColmnarDB::Types::ComplexPolygon max)
{
	return false;
}

template<>
bool CompressionGPU::decompressDataAAFL<ColmnarDB::Types::Point>(ColmnarDB::Types::Point* const host_compressed, int64_t compressed_size, std::vector<ColmnarDB::Types::Point>& host_uncompressed, ColmnarDB::Types::Point min, ColmnarDB::Types::Point max)
{
	return false;
}






template<typename T>
bool decompressAAFLOnDevice(const int CWARP_SIZE, T* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, T* const device_uncompressed, T min, T max)
{
	T offset = -1;

	int64_t size = data_size / sizeof(T);

	T *device_compressed_data;

	unsigned char *device_bit_length;
	unsigned long *device_position_id;
		
	int coded_data_position_id_start = (sizeof(int64_t) / (float)sizeof(T) * 3);
	int coded_data_bit_length_start = coded_data_position_id_start + std::max((int)(sizeof(unsigned long) / (float)sizeof(T) * compression_blocks_count), 1);
	int device_out_start = coded_data_bit_length_start + std::max((int)(sizeof(char) / (float)sizeof(T) * compression_blocks_count), 1);

	device_position_id = reinterpret_cast<unsigned long*>(&device_compressed[coded_data_position_id_start]);
	device_bit_length = reinterpret_cast<unsigned char*>(&device_compressed[coded_data_bit_length_start]);
	device_compressed_data = &device_compressed[device_out_start];

	
	container_uncompressed<T> udata = { device_uncompressed, size };
	container_aafl<T> cdata = { device_compressed_data, size, device_bit_length, device_position_id, NULL, offset };

	gpu_fl_naive_launcher_decompression<T, 32, container_aafl<T>>::decompress(cdata, udata);
	QueryEngineError::setCudaError(hipGetLastError());

	return true;
}


template<>
bool CompressionGPU::decompressDataAAFLOnDevice<int32_t>(int32_t* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, int32_t* const device_uncompressed, int32_t min, int32_t max)
{
	return decompressAAFLOnDevice(32, device_compressed, data_size, compression_data_size, compression_blocks_count, device_uncompressed, min, max);
}

template<>
bool CompressionGPU::decompressDataAAFLOnDevice<int64_t>(int64_t* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, int64_t* const device_uncompressed, int64_t min, int64_t max)
{
	return decompressAAFLOnDevice(32, device_compressed, data_size, compression_data_size, compression_blocks_count, device_uncompressed, min, max);
}


template<>
bool CompressionGPU::decompressDataAAFLOnDevice<int8_t>(int8_t* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, int8_t* const device_uncompressed, int8_t min, int8_t max)
{
	return decompressAAFLOnDevice(32, device_compressed, data_size, compression_data_size, compression_blocks_count, device_uncompressed, min, max);
}

template<>
bool CompressionGPU::decompressDataAAFLOnDevice<float>(float* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, float* const device_uncompressed, float min, float max)
{
	return false;
}

template<>
bool CompressionGPU::decompressDataAAFLOnDevice<double>(double* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, double* const device_uncompressed, double min, double max)
{
	return false;
}

template<>
bool CompressionGPU::decompressDataAAFLOnDevice<ColmnarDB::Types::ComplexPolygon>(ColmnarDB::Types::ComplexPolygon* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, ColmnarDB::Types::ComplexPolygon* const device_uncompressed, ColmnarDB::Types::ComplexPolygon min, ColmnarDB::Types::ComplexPolygon max)
{
	return false;
}

template<>
bool CompressionGPU::decompressDataAAFLOnDevice<ColmnarDB::Types::Point>(ColmnarDB::Types::Point* const device_compressed, int64_t data_size, int64_t compression_data_size, int64_t compression_blocks_count, ColmnarDB::Types::Point* const device_uncompressed, ColmnarDB::Types::Point min, ColmnarDB::Types::Point max)
{
	return false;
}