#include "hip/hip_runtime.h"
#include "GpuSqlDispatcherVMFunctions.h"
#include <array>
#include "../ParserExceptions.h"
#include "../../PointFactory.h"
#include "../../CudaLogBoost.h"
#include "../../ComplexPolygonFactory.h"
#include "../../StringFactory.h"
#include "DispatcherMacros.h"

BEGIN_UNARY_DISPATCH_TABLE(GpuSqlDispatcher::retFunctions_)
DISPATCHER_UNARY_FUNCTION(GpuSqlDispatcher::Ret, int32_t)
DISPATCHER_UNARY_FUNCTION(GpuSqlDispatcher::Ret, int64_t)
DISPATCHER_UNARY_FUNCTION(GpuSqlDispatcher::Ret, float)
DISPATCHER_UNARY_FUNCTION(GpuSqlDispatcher::Ret, double)
DISPATCHER_UNARY_FUNCTION(GpuSqlDispatcher::Ret, ColmnarDB::Types::Point)
DISPATCHER_UNARY_FUNCTION(GpuSqlDispatcher::Ret, ColmnarDB::Types::ComplexPolygon)
DISPATCHER_UNARY_FUNCTION(GpuSqlDispatcher::Ret, std::string)
DISPATCHER_UNARY_FUNCTION(GpuSqlDispatcher::Ret, int8_t)
END_DISPATCH_TABLE

GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::lockRegisterFunction_ = &GpuSqlDispatcher::LockRegister;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::getLoadSizeFunction_ = &GpuSqlDispatcher::GetLoadSize;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::filFunction_ = &GpuSqlDispatcher::Fil;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::whereEvaluationFunction_ = &GpuSqlDispatcher::WhereEvaluation;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::jmpFunction_ = &GpuSqlDispatcher::Jmp;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::doneFunction_ = &GpuSqlDispatcher::Done;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::showDatabasesFunction_ = &GpuSqlDispatcher::ShowDatabases;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::showTablesFunction_ = &GpuSqlDispatcher::ShowTables;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::showColumnsFunction_ = &GpuSqlDispatcher::ShowColumns;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::showConstraintsFunction_ = &GpuSqlDispatcher::ShowConstraints;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::showQueryColumnTypesFunction_ =
    &GpuSqlDispatcher::ShowQueryColumnTypes;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::insertIntoDoneFunction_ = &GpuSqlDispatcher::InsertIntoDone;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::createDatabaseFunction_ = &GpuSqlDispatcher::CreateDatabase;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::dropDatabaseFunction_ = &GpuSqlDispatcher::DropDatabase;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::createTableFunction_ = &GpuSqlDispatcher::CreateTable;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::dropTableFunction_ = &GpuSqlDispatcher::DropTable;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::alterTableFunction_ = &GpuSqlDispatcher::AlterTable;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::alterDatabaseFunction_ = &GpuSqlDispatcher::AlterDatabase;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::createIndexFunction_ = &GpuSqlDispatcher::CreateIndex;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::isNullFunction_ =
    &GpuSqlDispatcher::NullMaskCol<NullMaskOperations::isNull>;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::isNotNullFunction_ =
    &GpuSqlDispatcher::NullMaskCol<NullMaskOperations::isNotNull>;

template <>
GpuSqlDispatcher::InstructionStatus
GpuSqlDispatcher::LoadCol<ColmnarDB::Types::ComplexPolygon>(std::string& colName)
{
    if (allocatedPointers_.find(colName + "_polyPoints") == allocatedPointers_.end() &&
        !colName.empty() && colName.front() != '$')
    {
        CudaLogBoost::getInstance(CudaLogBoost::debug)
            << "Load: " << colName << " " << typeid(ColmnarDB::Types::ComplexPolygon).name() << '\n';

        std::string table;
        std::string column;

        std::tie(table, column) = SplitColumnName(colName);

        const int32_t blockCount =
            database_->GetTables().at(table).GetColumns().at(column).get()->GetBlockCount();
        GpuSqlDispatcher::deviceCountLimit_ =
            std::min(Context::getInstance().getDeviceCount() - 1, blockCount - 1);
        if (blockIndex_ >= blockCount)
        {
            return InstructionStatus::OUT_OF_BLOCKS;
        }
        if (blockIndex_ >= blockCount - Context::getInstance().getDeviceCount())
        {
            isLastBlockOfDevice_ = true;
        }
        if (blockIndex_ == blockCount - 1)
        {
            isOverallLastBlock_ = true;
        }

        noLoad_ = false;

        if (loadNecessary_ == 0 || loadSize_ <= 0)
        {
            instructionPointer_ = jmpInstructionPosition_;
            return InstructionStatus::LOAD_SKIPPED;
        }

        auto col = dynamic_cast<const ColumnBase<ColmnarDB::Types::ComplexPolygon>*>(
            database_->GetTables().at(table).GetColumns().at(column).get());


        if (!usingJoin_)
        {
            auto block =
                dynamic_cast<BlockBase<ColmnarDB::Types::ComplexPolygon>*>(col->GetBlocksList()[blockIndex_]);
            nullmask_t* nullMaskPtr = nullptr;

            if (block->GetNullBitmask())
            {
                if (allocatedPointers_.find(colName + NULL_SUFFIX) == allocatedPointers_.end())
                {
                    int32_t bitMaskCapacity = NullValues::GetNullBitMaskSize(loadSize_);
                    nullMaskPtr = AllocateRegister<nullmask_t>(colName + NULL_SUFFIX, bitMaskCapacity);

                    if (loadOffset_ > 0)
                    {
                        int32_t offsetBitMaskCapacity = NullValues::GetNullBitMaskSize(loadSize_ + loadOffset_);
                        int32_t maxBitMaskCapacity = NullValues::GetNullBitMaskSize(block->GetSize());

                        offsetBitMaskCapacity = std::min(offsetBitMaskCapacity, maxBitMaskCapacity);

                        std::vector<nullmask_t> maskToOffset(block->GetNullBitmask(),
                                                          block->GetNullBitmask() + offsetBitMaskCapacity);
                        ShiftNullMaskLeft(maskToOffset, loadOffset_);
                        GPUMemory::copyHostToDevice(nullMaskPtr, maskToOffset.data(), bitMaskCapacity);
                    }
                    else
                    {
                        GPUMemory::copyHostToDevice(nullMaskPtr, block->GetNullBitmask(), bitMaskCapacity);
                    }
                }
                else
                {
                    nullMaskPtr =
                        reinterpret_cast<nullmask_t*>(allocatedPointers_.at(colName + NULL_SUFFIX).GpuPtr);
                }
            }
            InsertComplexPolygon(database_->GetName(), colName,
                                 std::vector<ColmnarDB::Types::ComplexPolygon>(block->GetData() + loadOffset_,
                                                                               block->GetData() + loadOffset_ + loadSize_),
                                 loadSize_, false, nullMaskPtr);
            noLoad_ = false;
        }
        else
        {
            CudaLogBoost::getInstance(CudaLogBoost::debug) << "Loading joined block." << '\n';
            int32_t loadSize = joinIndices_->at(table)[blockIndex_].size();
            std::string joinCacheId = colName + "_join";
            for (auto& joinTable : *joinIndices_)
            {
                joinCacheId += "_" + joinTable.first;
            }

            std::vector<ColmnarDB::Types::ComplexPolygon> joinedPolygons;
            nullmask_t* nullMaskPtr = nullptr;

            int32_t outDataSize;
            CPUJoinReorderer::reorderByJI<ColmnarDB::Types::ComplexPolygon>(joinedPolygons, outDataSize,
                                                                            *col, blockIndex_,
                                                                            joinIndices_->at(table),
                                                                            database_->GetBlockSize());

            if (col->GetIsNullable())
            {
                if (allocatedPointers_.find(colName + NULL_SUFFIX) == allocatedPointers_.end())
                {
                    int32_t bitMaskCapacity = NullValues::GetNullBitMaskSize(loadSize);
                    auto cacheMaskEntry =
                        Context::getInstance().getCacheForCurrentDevice().getColumn<nullmask_t>(
                        database_->GetName(), joinCacheId + NULL_SUFFIX, blockIndex_,
                        bitMaskCapacity, loadSize_, loadOffset_);
                    nullMaskPtr = std::get<0>(cacheMaskEntry);
                    if (!std::get<2>(cacheMaskEntry))
                    {
                        int32_t outMaskSize;
                        CPUJoinReorderer::reorderNullMaskByJIPushToGPU<ColmnarDB::Types::ComplexPolygon>(
                            std::get<0>(cacheMaskEntry), outMaskSize, *col, blockIndex_,
                            joinIndices_->at(table), database_->GetBlockSize());
                    }
                }
                else
                {
                    nullMaskPtr =
                        reinterpret_cast<nullmask_t*>(allocatedPointers_.at(colName + NULL_SUFFIX).GpuPtr);
                }
            }

            InsertComplexPolygon(database_->GetName(), colName, joinedPolygons, loadSize, nullMaskPtr);
            noLoad_ = false;
        }
    }
    return InstructionStatus::CONTINUE;
}

template <>
GpuSqlDispatcher::InstructionStatus GpuSqlDispatcher::LoadCol<ColmnarDB::Types::Point>(std::string& colName)
{
    if (allocatedPointers_.find(colName) == allocatedPointers_.end() && !colName.empty() && colName.front() != '$')
    {
        CudaLogBoost::getInstance(CudaLogBoost::debug)
            << "Load: " << colName << " " << typeid(ColmnarDB::Types::Point).name() << '\n';

        std::string table;
        std::string column;

        std::tie(table, column) = SplitColumnName(colName);

        const int32_t blockCount =
            database_->GetTables().at(table).GetColumns().at(column).get()->GetBlockCount();
        GpuSqlDispatcher::deviceCountLimit_ =
            std::min(Context::getInstance().getDeviceCount() - 1, blockCount - 1);
        if (blockIndex_ >= blockCount)
        {
            return InstructionStatus::OUT_OF_BLOCKS;
        }
        if (blockIndex_ >= blockCount - Context::getInstance().getDeviceCount())
        {
            isLastBlockOfDevice_ = true;
        }
        if (blockIndex_ == blockCount - 1)
        {
            isOverallLastBlock_ = true;
        }

        noLoad_ = false;

        if (loadNecessary_ == 0 || loadSize_ <= 0)
        {
            instructionPointer_ = jmpInstructionPosition_;
            return InstructionStatus::LOAD_SKIPPED;
        }

        auto col = dynamic_cast<const ColumnBase<ColmnarDB::Types::Point>*>(
            database_->GetTables().at(table).GetColumns().at(column).get());

        if (!usingJoin_)
        {
            auto block = dynamic_cast<BlockBase<ColmnarDB::Types::Point>*>(col->GetBlocksList()[blockIndex_]);

            std::vector<NativeGeoPoint> nativePoints;
            std::transform(block->GetData() + loadOffset_, block->GetData() + loadOffset_ + loadSize_, std::back_inserter(nativePoints), [](const ColmnarDB::Types::Point& point) -> NativeGeoPoint {
                return NativeGeoPoint{point.geopoint().latitude(), point.geopoint().longitude()};
            });

            auto cacheEntry = Context::getInstance().getCacheForCurrentDevice().getColumn<NativeGeoPoint>(
                database_->GetName(), colName, blockIndex_, nativePoints.size(), loadSize_, loadOffset_);
            if (!std::get<2>(cacheEntry))
            {
                GPUMemory::copyHostToDevice(std::get<0>(cacheEntry),
                                            reinterpret_cast<NativeGeoPoint*>(nativePoints.data()),
                                            nativePoints.size());
            }
            nullmask_t* nullMaskPtr = nullptr;
            if (block->GetNullBitmask())
            {
                if (allocatedPointers_.find(colName + NULL_SUFFIX) == allocatedPointers_.end())
                {
                    int32_t bitMaskCapacity = NullValues::GetNullBitMaskSize(loadSize_);
                    auto cacheMaskEntry =
                        Context::getInstance().getCacheForCurrentDevice().getColumn<nullmask_t>(
                        database_->GetName(), colName + NULL_SUFFIX, blockIndex_, bitMaskCapacity,
                        loadSize_, loadOffset_);
                    nullMaskPtr = std::get<0>(cacheMaskEntry);
                    if (!std::get<2>(cacheMaskEntry))
                    {
                        if (loadOffset_ > 0)
                        {
                            int32_t offsetBitMaskCapacity =
                                NullValues::GetNullBitMaskSize(loadSize_ + loadOffset_);
                            int32_t maxBitMaskCapacity = NullValues::GetNullBitMaskSize(block->GetSize());

                            offsetBitMaskCapacity = std::min(offsetBitMaskCapacity, maxBitMaskCapacity);

                            std::vector<nullmask_t> maskToOffset(block->GetNullBitmask(),
                                                              block->GetNullBitmask() + offsetBitMaskCapacity);
                            ShiftNullMaskLeft(maskToOffset, loadOffset_);
                            GPUMemory::copyHostToDevice(std::get<0>(cacheMaskEntry),
                                                        maskToOffset.data(), bitMaskCapacity);
                        }
                        else
                        {
                            GPUMemory::copyHostToDevice(std::get<0>(cacheMaskEntry),
                                                        block->GetNullBitmask(), bitMaskCapacity);
                        }
                    }
                    AddCachedRegister(colName + NULL_SUFFIX, std::get<0>(cacheMaskEntry), bitMaskCapacity);
                }
                else
                {
                    nullMaskPtr =
                        reinterpret_cast<nullmask_t*>(allocatedPointers_.at(colName + NULL_SUFFIX).GpuPtr);
                }
            }
            AddCachedRegister(colName, std::get<0>(cacheEntry), nativePoints.size(), nullMaskPtr);
            noLoad_ = false;
        }
        else
        {
            CudaLogBoost::getInstance(CudaLogBoost::debug) << "Loading joined block." << '\n';
            int32_t loadSize = joinIndices_->at(table)[blockIndex_].size();
            std::string joinCacheId = colName + "_join";
            for (auto& joinTable : *joinIndices_)
            {
                joinCacheId += "_" + joinTable.first;
            }

            std::vector<ColmnarDB::Types::Point> joinedPoints;
            nullmask_t* nullMaskPtr = nullptr;
            int32_t outDataSize;
            CPUJoinReorderer::reorderByJI<ColmnarDB::Types::Point>(joinedPoints, outDataSize, *col,
                                                                   blockIndex_, joinIndices_->at(table),
                                                                   database_->GetBlockSize());

            std::vector<NativeGeoPoint> nativePoints;
            std::transform(joinedPoints.data(), joinedPoints.data() + loadSize, std::back_inserter(nativePoints), [](const ColmnarDB::Types::Point& point) -> NativeGeoPoint {
                return NativeGeoPoint{point.geopoint().latitude(), point.geopoint().longitude()};
            });

            auto cacheEntry = Context::getInstance().getCacheForCurrentDevice().getColumn<NativeGeoPoint>(
                database_->GetName(), joinCacheId, blockIndex_, loadSize, loadSize_, loadOffset_);
            if (!std::get<2>(cacheEntry))
            {
                GPUMemory::copyHostToDevice(std::get<0>(cacheEntry),
                                            reinterpret_cast<NativeGeoPoint*>(nativePoints.data()),
                                            nativePoints.size());
            }

            if (col->GetIsNullable())
            {
                if (allocatedPointers_.find(colName + NULL_SUFFIX) == allocatedPointers_.end())
                {
                    int32_t bitMaskCapacity = NullValues::GetNullBitMaskSize(loadSize);
                    auto cacheMaskEntry =
                        Context::getInstance().getCacheForCurrentDevice().getColumn<nullmask_t>(
                        database_->GetName(), joinCacheId + NULL_SUFFIX, blockIndex_,
                        bitMaskCapacity, loadSize_, loadOffset_);
                    nullMaskPtr = std::get<0>(cacheMaskEntry);
                    if (!std::get<2>(cacheMaskEntry))
                    {
                        int32_t outMaskSize;
                        CPUJoinReorderer::reorderNullMaskByJIPushToGPU<ColmnarDB::Types::Point>(
                            std::get<0>(cacheMaskEntry), outMaskSize, *col, blockIndex_,
                            joinIndices_->at(table), database_->GetBlockSize());
                    }
                    AddCachedRegister(colName + NULL_SUFFIX, std::get<0>(cacheMaskEntry), bitMaskCapacity);
                }
                else
                {
                    nullMaskPtr =
                        reinterpret_cast<nullmask_t*>(allocatedPointers_.at(colName + NULL_SUFFIX).GpuPtr);
                }
            }

            AddCachedRegister(colName, std::get<0>(cacheEntry), loadSize, nullMaskPtr);
            noLoad_ = false;
        }
    }
    return InstructionStatus::CONTINUE;
}


template <>
GpuSqlDispatcher::InstructionStatus GpuSqlDispatcher::LoadCol<std::string>(std::string& colName)
{
    if (allocatedPointers_.find(colName + "_allChars") == allocatedPointers_.end() &&
        !colName.empty() && colName.front() != '$')
    {
        CudaLogBoost::getInstance(CudaLogBoost::debug)
            << "Load: " << colName << " " << typeid(std::string).name() << '\n';

        std::string table;
        std::string column;

        std::tie(table, column) = SplitColumnName(colName);

        const int32_t blockCount =
            database_->GetTables().at(table).GetColumns().at(column).get()->GetBlockCount();
        GpuSqlDispatcher::deviceCountLimit_ =
            std::min(Context::getInstance().getDeviceCount() - 1, blockCount - 1);
        if (blockIndex_ >= blockCount)
        {
            return InstructionStatus::OUT_OF_BLOCKS;
        }
        if (blockIndex_ >= blockCount - Context::getInstance().getDeviceCount())
        {
            isLastBlockOfDevice_ = true;
        }
        if (blockIndex_ == blockCount - 1)
        {
            isOverallLastBlock_ = true;
        }

        noLoad_ = false;

        if (loadNecessary_ == 0 || loadSize_ <= 0)
        {
            instructionPointer_ = jmpInstructionPosition_;
            return InstructionStatus::LOAD_SKIPPED;
        }

        auto col = dynamic_cast<const ColumnBase<std::string>*>(
            database_->GetTables().at(table).GetColumns().at(column).get());

        if (!usingJoin_)
        {
            auto block = dynamic_cast<BlockBase<std::string>*>(col->GetBlocksList()[blockIndex_]);
            nullmask_t* nullMaskPtr = nullptr;
            if (block->GetNullBitmask())
            {
                if (allocatedPointers_.find(colName + NULL_SUFFIX) == allocatedPointers_.end())
                {
                    int32_t bitMaskCapacity = NullValues::GetNullBitMaskSize(loadSize_);
                    nullMaskPtr = AllocateRegister<nullmask_t>(colName + NULL_SUFFIX, bitMaskCapacity);
                    if (loadOffset_ > 0)
                    {
                        int32_t offsetBitMaskCapacity = NullValues::GetNullBitMaskSize(loadSize_ + loadOffset_);
                        int32_t maxBitMaskCapacity = NullValues::GetNullBitMaskSize(block->GetSize());

                        offsetBitMaskCapacity = std::min(offsetBitMaskCapacity, maxBitMaskCapacity);

                        std::vector<nullmask_t> maskToOffset(block->GetNullBitmask(),
                                                          block->GetNullBitmask() + offsetBitMaskCapacity);
                        ShiftNullMaskLeft(maskToOffset, loadOffset_);
                        GPUMemory::copyHostToDevice(nullMaskPtr, maskToOffset.data(), bitMaskCapacity);
                    }
                    else
                    {
                        GPUMemory::copyHostToDevice(nullMaskPtr, block->GetNullBitmask(), bitMaskCapacity);
                    }
                }
                else
                {
                    nullMaskPtr =
                        reinterpret_cast<nullmask_t*>(allocatedPointers_.at(colName + NULL_SUFFIX).GpuPtr);
                }
            }
            InsertString(database_->GetName(), colName, block->GetData() + loadOffset_, loadSize_,
                         false, nullMaskPtr);
            noLoad_ = false;
        }
        else
        {
            CudaLogBoost::getInstance(CudaLogBoost::debug) << "Loading joined block." << '\n';
            int32_t loadSize = joinIndices_->at(table)[blockIndex_].size();
            std::string joinCacheId = colName + "_join";
            for (auto& joinTable : *joinIndices_)
            {
                joinCacheId += "_" + joinTable.first;
            }

            std::vector<std::string> joinedStrings;
            nullmask_t* nullMaskPtr = nullptr;

            int32_t outDataSize;
            CPUJoinReorderer::reorderByJI<std::string>(joinedStrings, outDataSize, *col, blockIndex_,
                                                       joinIndices_->at(table), database_->GetBlockSize());

            if (col->GetIsNullable())
            {
                if (allocatedPointers_.find(colName + NULL_SUFFIX) == allocatedPointers_.end())
                {
                    int32_t bitMaskCapacity = NullValues::GetNullBitMaskSize(loadSize);
                    auto cacheMaskEntry =
                        Context::getInstance().getCacheForCurrentDevice().getColumn<nullmask_t>(
                        database_->GetName(), joinCacheId + NULL_SUFFIX, blockIndex_,
                        bitMaskCapacity, loadSize_, loadOffset_);
                    nullMaskPtr = std::get<0>(cacheMaskEntry);
                    if (!std::get<2>(cacheMaskEntry))
                    {
                        int32_t outMaskSize;
                        CPUJoinReorderer::reorderNullMaskByJIPushToGPU<std::string>(
                            std::get<0>(cacheMaskEntry), outMaskSize, *col, blockIndex_,
                            joinIndices_->at(table), database_->GetBlockSize());
                    }
                }
                else
                {
                    nullMaskPtr =
                        reinterpret_cast<nullmask_t*>(allocatedPointers_.at(colName + NULL_SUFFIX).GpuPtr);
                }
            }

            InsertString(database_->GetName(), colName, joinedStrings.data(), loadSize, nullMaskPtr);
            noLoad_ = false;
        }
    }
    return InstructionStatus::CONTINUE;
}

template <>
GpuSqlDispatcher::InstructionStatus GpuSqlDispatcher::RetCol<ColmnarDB::Types::ComplexPolygon>()
{
    if (usingGroupBy_)
    {
        throw RetPolygonGroupByException();
    }
    else
    {
        auto col = arguments_.Read<std::string>();
        PayloadType payloadType = static_cast<PayloadType>(arguments_.Read<int32_t>());
        auto alias = arguments_.Read<std::string>();

        GpuSqlDispatcher::InstructionStatus loadFlag = LoadCol<ColmnarDB::Types::ComplexPolygon>(col);
        if (loadFlag != InstructionStatus::CONTINUE)
        {
            return loadFlag;
        }
        CudaLogBoost::getInstance(CudaLogBoost::debug)
            << "RetPolygonCol: " << col << ", thread: " << dispatcherThreadId_ << '\n';

        std::unique_ptr<std::string[]> outData(new std::string[database_->GetBlockSize()]);
        int32_t outSize;
        int32_t nullMaskPtrSize = 0;
        std::vector<nullmask_t> nullMaskVector = {};

        if (usingOrderBy_)
        {
            if (isOverallLastBlock_)
            {
                VariantArray<std::string>* reconstructedColumn = dynamic_cast<VariantArray<std::string>*>(
                    reconstructedOrderByColumnsMerged_.at(col).get());
                outData = std::move(reconstructedColumn->getDataRef());
                outSize = reconstructedColumn->GetSize();

                nullMaskPtrSize = NullValues::GetNullBitMaskSize(outSize);
                nullMaskVector =
                    std::vector<nullmask_t>(reconstructedOrderByColumnsNullMerged_.at(col).get(),
                                         reconstructedOrderByColumnsNullMerged_.at(col).get() + nullMaskPtrSize);
            }
            else
            {
                return InstructionStatus::CONTINUE;
            }
        }
        else
        {
            auto ACol = FindCompositeDataTypeAllocation<ColmnarDB::Types::ComplexPolygon>(col);


            if (ACol.GpuNullMaskPtr)
            {
                size_t bitMaskSize = NullValues::GetNullBitMaskSize(database_->GetBlockSize());
                std::unique_ptr<nullmask_t[]> nullMask(new nullmask_t[bitMaskSize]);
                GPUReconstruct::ReconstructPolyColToWKT(outData.get(), &outSize, ACol.GpuPtr,
                                                        reinterpret_cast<int8_t*>(filter_),
                                                        ACol.ElementCount, nullMask.get(),
                                                        reinterpret_cast<nullmask_t*>(ACol.GpuNullMaskPtr));
                nullMaskPtrSize = NullValues::GetNullBitMaskSize(outSize);
                nullMaskVector = std::vector<nullmask_t>(nullMask.get(), nullMask.get() + nullMaskPtrSize);
            }
            else
            {
                GPUReconstruct::ReconstructPolyColToWKT(outData.get(), &outSize, ACol.GpuPtr,
                                                        reinterpret_cast<int8_t*>(filter_), ACol.ElementCount);
            }
            CudaLogBoost::getInstance(CudaLogBoost::debug) << "dataSize: " << outSize << '\n';
        }

        if (outSize > 0)
        {
            ColmnarDB::NetworkClient::Message::QueryResponsePayload payload;
            InsertIntoPayload(payload, outData, outSize, payloadType);
            MergePayloadToSelfResponse(alias, col, payload, nullMaskVector);
        }
    }
    return InstructionStatus::CONTINUE;
}

template <>
GpuSqlDispatcher::InstructionStatus GpuSqlDispatcher::RetCol<ColmnarDB::Types::Point>()
{
    if (usingGroupBy_)
    {
        throw RetPointGroupByException();
    }
    else
    {
        auto colName = arguments_.Read<std::string>();
        PayloadType payloadType = static_cast<PayloadType>(arguments_.Read<int32_t>());
        auto alias = arguments_.Read<std::string>();

        GpuSqlDispatcher::InstructionStatus loadFlag = LoadCol<ColmnarDB::Types::Point>(colName);
        if (loadFlag != InstructionStatus::CONTINUE)
        {
            return loadFlag;
        }

        CudaLogBoost::getInstance(CudaLogBoost::debug)
            << "RetPointCol: " << colName << ", thread: " << dispatcherThreadId_ << '\n';

        std::unique_ptr<std::string[]> outData(new std::string[database_->GetBlockSize()]);
        int32_t outSize;
        int32_t nullMaskPtrSize = 0;
        std::vector<nullmask_t> nullMaskVector = {};
        // ToDo: Podmienene zapnut podla velkost buffera
        // GPUMemory::hostPin(outData.get(), database_->GetBlockSize());

        if (usingOrderBy_)
        {
            if (isOverallLastBlock_)
            {
                VariantArray<std::string>* reconstructedColumn = dynamic_cast<VariantArray<std::string>*>(
                    reconstructedOrderByColumnsMerged_.at(colName).get());
                outData = std::move(reconstructedColumn->getDataRef());
                outSize = reconstructedColumn->GetSize();

                nullMaskPtrSize = NullValues::GetNullBitMaskSize(outSize);
                nullMaskVector =
                    std::vector<nullmask_t>(reconstructedOrderByColumnsNullMerged_.at(colName).get(),
                                         reconstructedOrderByColumnsNullMerged_.at(colName).get() + nullMaskPtrSize);
            }
            else
            {
                return InstructionStatus::CONTINUE;
            }
        }
        else
        {
            PointerAllocation ACol = allocatedPointers_.at(colName);

            if (ACol.GpuNullMaskPtr)
            {
                size_t bitMaskSize = NullValues::GetNullBitMaskSize(database_->GetBlockSize());
                std::unique_ptr<nullmask_t[]> nullMask(new nullmask_t[bitMaskSize]);
                GPUReconstruct::ReconstructPointColToWKT(outData.get(), &outSize,
                                                         reinterpret_cast<NativeGeoPoint*>(ACol.GpuPtr),
                                                         reinterpret_cast<int8_t*>(filter_),
                                                         ACol.ElementCount, nullMask.get(),
                                                         reinterpret_cast<nullmask_t*>(ACol.GpuNullMaskPtr));
                nullMaskPtrSize = NullValues::GetNullBitMaskSize(outSize);
                nullMaskVector = std::vector<nullmask_t>(nullMask.get(), nullMask.get() + nullMaskPtrSize);
            }
            else
            {
                GPUReconstruct::ReconstructPointColToWKT(outData.get(), &outSize,
                                                         reinterpret_cast<NativeGeoPoint*>(ACol.GpuPtr),
                                                         reinterpret_cast<int8_t*>(filter_), ACol.ElementCount);
            }
            // GPUMemory::hostUnregister(outData.get());
            CudaLogBoost::getInstance(CudaLogBoost::debug) << "dataSize: " << outSize << '\n';
        }

        if (outSize > 0)
        {
            ColmnarDB::NetworkClient::Message::QueryResponsePayload payload;
            InsertIntoPayload(payload, outData, outSize, payloadType);
            MergePayloadToSelfResponse(alias, colName, payload, nullMaskVector);
        }
    }
    return InstructionStatus::CONTINUE;
}

template <>
GpuSqlDispatcher::InstructionStatus GpuSqlDispatcher::RetCol<std::string>()
{
    auto colName = arguments_.Read<std::string>();
    PayloadType payloadType = static_cast<PayloadType>(arguments_.Read<int32_t>());
    auto alias = arguments_.Read<std::string>();

    GpuSqlDispatcher::InstructionStatus loadFlag = LoadCol<std::string>(colName);
    if (loadFlag != InstructionStatus::CONTINUE)
    {
        return loadFlag;
    }

    CudaLogBoost::getInstance(CudaLogBoost::debug)
        << "RetStringCol: " << colName << ", thread: " << dispatcherThreadId_ << '\n';

    int32_t outSize;
    std::unique_ptr<std::string[]> outData;
    int32_t nullMaskPtrSize = 0;
    std::vector<nullmask_t> nullMaskVector = {};
    if (usingGroupBy_)
    {
        if (isOverallLastBlock_)
        {
            // Return key or value col (key if groupByColumns_ contains colName)
            auto col = FindCompositeDataTypeAllocation<std::string>(
                colName + (std::find_if(groupByColumns_.begin(), groupByColumns_.end(),
                                        StringDataTypeComp(colName)) != groupByColumns_.end() ?
                               KEYS_SUFFIX :
                               ""));
            outSize = col.ElementCount;

            if (usingOrderBy_)
            {
                CudaLogBoost::getInstance(CudaLogBoost::debug) << "Reordering result block." << '\n';

                GPUMemory::GPUString reorderedColumn;
                size_t inNullColSize = NullValues::GetNullBitMaskSize(outSize);
                cuda_ptr<nullmask_t> reorderedNullColumn(inNullColSize);

                PointerAllocation orderByIndices = allocatedPointers_.at("$orderByIndices");
                GPUOrderBy::ReOrderStringByIdx(reorderedColumn,
                                               reinterpret_cast<int32_t*>(orderByIndices.GpuPtr),
                                               col.GpuPtr, outSize);
                GPUOrderBy::ReOrderNullValuesByIdx(reorderedNullColumn.get(),
                                                   reinterpret_cast<int32_t*>(orderByIndices.GpuPtr),
                                                   reinterpret_cast<nullmask_t*>(col.GpuNullMaskPtr), outSize);

                GPUMemory::free(col.GpuPtr);
                GPUMemory::free(reinterpret_cast<int8_t*>(col.GpuNullMaskPtr));

                col.GpuPtr.stringIndices = reorderedColumn.stringIndices;
                col.GpuPtr.allChars = reorderedColumn.allChars;
                col.GpuNullMaskPtr = reinterpret_cast<std::uintptr_t>(reorderedNullColumn.release());
            }

            outData = std::unique_ptr<std::string[]>(new std::string[outSize]);
            if (col.GpuNullMaskPtr)
            {
                size_t bitMaskSize = NullValues::GetNullBitMaskSize(database_->GetBlockSize());
                std::unique_ptr<nullmask_t[]> nullMask =
                    std::unique_ptr<nullmask_t[]>(new nullmask_t[bitMaskSize]);
                GPUReconstruct::ReconstructStringCol(outData.get(), &outSize, col.GpuPtr, nullptr,
                                                     col.ElementCount, nullMask.get(),
                                                     reinterpret_cast<nullmask_t*>(col.GpuNullMaskPtr));
                nullMaskPtrSize = NullValues::GetNullBitMaskSize(outSize);
                nullMaskVector = std::vector<nullmask_t>(nullMask.get(), nullMask.get() + nullMaskPtrSize);
            }
            else
            {
                GPUReconstruct::ReconstructStringCol(outData.get(), &outSize, col.GpuPtr, nullptr, col.ElementCount);
            }
        }
        else
        {
            return InstructionStatus::CONTINUE;
        }
    }
    else
    {
        if (usingOrderBy_)
        {
            if (isOverallLastBlock_)
            {
                VariantArray<std::string>* reconstructedColumn = dynamic_cast<VariantArray<std::string>*>(
                    reconstructedOrderByColumnsMerged_.at(colName).get());
                outData = std::move(reconstructedColumn->getDataRef());
                outSize = reconstructedColumn->GetSize();

                nullMaskPtrSize = NullValues::GetNullBitMaskSize(outSize);
                nullMaskVector =
                    std::vector<nullmask_t>(reconstructedOrderByColumnsNullMerged_.at(colName).get(),
                                         reconstructedOrderByColumnsNullMerged_.at(colName).get() + nullMaskPtrSize);
          }
            else
            {
                return InstructionStatus::CONTINUE;
            }
        }
        else
        {
            auto col = FindCompositeDataTypeAllocation<std::string>(colName);
            outSize = col.ElementCount;
            outData = std::unique_ptr<std::string[]>(new std::string[outSize]);
            if (col.GpuNullMaskPtr)
            {
                size_t bitMaskSize = NullValues::GetNullBitMaskSize(database_->GetBlockSize());
                std::unique_ptr<nullmask_t[]> nullMask(new nullmask_t[bitMaskSize]);
                GPUReconstruct::ReconstructStringCol(outData.get(), &outSize, col.GpuPtr,
                                                     reinterpret_cast<int8_t*>(filter_),
                                                     col.ElementCount, nullMask.get(),
                                                     reinterpret_cast<nullmask_t*>(col.GpuNullMaskPtr));
                nullMaskPtrSize = NullValues::GetNullBitMaskSize(outSize);
                nullMaskVector = std::vector<nullmask_t>(nullMask.get(), nullMask.get() + nullMaskPtrSize);
            }
            else
            {
                GPUReconstruct::ReconstructStringCol(outData.get(), &outSize, col.GpuPtr,
                                                     reinterpret_cast<int8_t*>(filter_), col.ElementCount);
            }
        }
        CudaLogBoost::getInstance(CudaLogBoost::debug) << "dataSize: " << outSize << '\n';
    }

    if (outSize > 0)
    {
        ColmnarDB::NetworkClient::Message::QueryResponsePayload payload;
        InsertIntoPayload(payload, outData, outSize, payloadType);
        MergePayloadToSelfResponse(alias, colName, payload, nullMaskVector);
    }
    return InstructionStatus::CONTINUE;
}

template <>
GpuSqlDispatcher::InstructionStatus GpuSqlDispatcher::RetConst<std::string>()
{
    std::string cnst = arguments_.Read<std::string>();
    PayloadType payloadType = static_cast<PayloadType>(arguments_.Read<int32_t>());
    std::string alias = arguments_.Read<std::string>();

    CudaLogBoost::getInstance(CudaLogBoost::debug) << "RET: cnst" << typeid(std::string).name() << '\n';

    ColmnarDB::NetworkClient::Message::QueryResponsePayload payload;
    GpuSqlDispatcher::InstructionStatus loadFlag = LoadTableBlockInfo(loadedTableName_);
    if (loadFlag != InstructionStatus::CONTINUE)
    {
        return loadFlag;
    }

    // Compute count of copies of the const
    int64_t dataElementCount = GetBlockSize();
    if (filter_)
    {
        GPUReconstruct::Sum(dataElementCount, reinterpret_cast<int8_t*>(filter_), dataElementCount);
    }

    // Create array and merge to protobuf response
    std::unique_ptr<std::string[]> outData(new std::string[dataElementCount]);
    std::fill(outData.get(), outData.get() + dataElementCount, cnst);
    InsertIntoPayload(payload, outData, dataElementCount, payloadType);
    MergePayloadToSelfResponse(alias, cnst, payload, {});
    return InstructionStatus::CONTINUE;
}

template <>
GpuSqlDispatcher::InstructionStatus GpuSqlDispatcher::RetConst<ColmnarDB::Types::Point>()
{
    std::string cnst = arguments_.Read<std::string>();
    PayloadType payloadType = static_cast<PayloadType>(arguments_.Read<int32_t>());
    std::string alias = arguments_.Read<std::string>();

    CudaLogBoost::getInstance(CudaLogBoost::debug)
        << "RET: cnst" << typeid(ColmnarDB::Types::Point).name() << '\n';

    ColmnarDB::NetworkClient::Message::QueryResponsePayload payload;
    GpuSqlDispatcher::InstructionStatus loadFlag = LoadTableBlockInfo(loadedTableName_);
    if (loadFlag != InstructionStatus::CONTINUE)
    {
        return loadFlag;
    }

    // Compute count of copies of the const
    int64_t dataElementCount = GetBlockSize();
    if (filter_)
    {
        GPUReconstruct::Sum(dataElementCount, reinterpret_cast<int8_t*>(filter_), dataElementCount);
    }

    // Create array and merge to protobuf response
    std::unique_ptr<std::string[]> outData(new std::string[dataElementCount]);
    std::fill(outData.get(), outData.get() + dataElementCount, cnst);
    InsertIntoPayload(payload, outData, dataElementCount, payloadType);
    MergePayloadToSelfResponse(alias, cnst, payload, {});
    return InstructionStatus::CONTINUE;
}

template <>
GpuSqlDispatcher::InstructionStatus GpuSqlDispatcher::RetConst<ColmnarDB::Types::ComplexPolygon>()
{
    std::string cnst = arguments_.Read<std::string>();
    PayloadType payloadType = static_cast<PayloadType>(arguments_.Read<int32_t>());
    std::string alias = arguments_.Read<std::string>();

    CudaLogBoost::getInstance(CudaLogBoost::debug)
        << "RET: cnst" << typeid(ColmnarDB::Types::ComplexPolygon).name() << '\n';

    ColmnarDB::NetworkClient::Message::QueryResponsePayload payload;
    GpuSqlDispatcher::InstructionStatus loadFlag = LoadTableBlockInfo(loadedTableName_);
    if (loadFlag != InstructionStatus::CONTINUE)
    {
        return loadFlag;
    }

    // Compute count of copies of the const
    int64_t dataElementCount = GetBlockSize();
    if (filter_)
    {
        GPUReconstruct::Sum(dataElementCount, reinterpret_cast<int8_t*>(filter_), dataElementCount);
    }

    // Create array and merge to protobuf response
    std::unique_ptr<std::string[]> outData(new std::string[dataElementCount]);
    std::fill(outData.get(), outData.get() + dataElementCount, cnst);
    InsertIntoPayload(payload, outData, dataElementCount, payloadType);
    MergePayloadToSelfResponse(alias, cnst, payload, {});
    return InstructionStatus::CONTINUE;
}

GpuSqlDispatcher::InstructionStatus GpuSqlDispatcher::LockRegister()
{
    std::string reg = arguments_.Read<std::string>();
    CudaLogBoost::getInstance(CudaLogBoost::debug) << "Locked register: " << reg << '\n';
    registerLockList_.insert(reg);
    return InstructionStatus::CONTINUE;
}

GpuSqlDispatcher::InstructionStatus GpuSqlDispatcher::LoadTableBlockInfo(const std::string& tableName)
{
    CudaLogBoost::getInstance(CudaLogBoost::debug) << "TableInfo: " << tableName << '\n';

    const int32_t blockCount = GetBlockCount();
    GpuSqlDispatcher::deviceCountLimit_ =
        std::min(Context::getInstance().getDeviceCount() - 1, blockCount - 1);
    if (blockIndex_ >= blockCount)
    {
        return InstructionStatus::OUT_OF_BLOCKS;
    }
    if (blockIndex_ >= blockCount - Context::getInstance().getDeviceCount())
    {
        isLastBlockOfDevice_ = true;
    }
    if (blockIndex_ == blockCount - 1)
    {
        isOverallLastBlock_ = true;
    }

    noLoad_ = false;

    return InstructionStatus::CONTINUE;
}

size_t GpuSqlDispatcher::GetBlockSize(int32_t blockIndex)
{
    if (blockIndex == -1)
    {
        blockIndex = blockIndex_;
    }

    int64_t dataElementCount = 0;
    if (LoadTableBlockInfo(loadedTableName_) != InstructionStatus::CONTINUE)
    {
        return 0;
    }
    if (usingJoin_)
    {
        dataElementCount = joinIndices_->begin()->second[blockIndex].size();
    }
    else
    {
        dataElementCount =
            database_->GetTables().at(loadedTableName_).GetColumns().begin()->second->GetBlockSizeForIndex(blockIndex);
    }

    return dataElementCount;
}

int32_t GpuSqlDispatcher::GetBlockCount()
{
    return usingJoin_ ?
               joinIndices_->at(loadedTableName_).size() :
               database_->GetTables().at(loadedTableName_).GetColumns().begin()->second.get()->GetBlockCount();
}

GpuSqlDispatcher::InstructionStatus GpuSqlDispatcher::GetLoadSize()
{
    int64_t offset = arguments_.Read<int64_t>();
    int64_t limit = arguments_.Read<int64_t>();

    bool usingWhere = arguments_.Read<bool>();
    bool usingGroupBy = arguments_.Read<bool>();
    bool usingOrderBy = arguments_.Read<bool>();
    bool usingAggregation = arguments_.Read<bool>();
    bool usingJoin = arguments_.Read<bool>();
    bool usingLoad = arguments_.Read<bool>();

    if (usingWhere || usingGroupBy || usingOrderBy || usingAggregation || usingJoin || !usingLoad)
    {
        loadOffset_ = 0;
        loadSize_ = GetBlockSize();
    }

    else
    {
        CudaLogBoost::getInstance(CudaLogBoost::info)
            << "GetLoadSize Offset: " << offset << " Limit: " << limit << '\n';

        int64_t offsetBlockIdx = 0;
        int64_t remainingOffset = offset;
        loadOffset_ = 0;
        loadSize_ = 0;

        while (offsetBlockIdx < GetBlockCount() && remainingOffset >= 0)
        {
            remainingOffset -= GetBlockSize(offsetBlockIdx++);
        }
        offsetBlockIdx--;

        int64_t offsetLimitBlockIdx = 0;
        int64_t remainingLimitOffset = offset + limit;

        while (offsetLimitBlockIdx < GetBlockCount() && remainingLimitOffset >= 0)
        {
            remainingLimitOffset -= GetBlockSize(offsetLimitBlockIdx++);
        }
        offsetLimitBlockIdx--;

        if (blockIndex_ < offsetBlockIdx || blockIndex_ > offsetLimitBlockIdx)
        {
            loadSize_ = 0;
        }

        const int64_t currentBlockSize = static_cast<int64_t>(GetBlockSize());

        if (blockIndex_ == offsetBlockIdx)
        {
            int64_t offsetBlockDataSize = 0;
            for (int32_t i = 0; i < offsetBlockIdx + 1; i++)
            {
                offsetBlockDataSize += GetBlockSize(i);
            }

            loadSize_ = std::min(offsetBlockDataSize - offset, currentBlockSize);
            loadOffset_ = std::min(offsetBlockDataSize - loadSize_, currentBlockSize);
        }

        if (blockIndex_ == offsetLimitBlockIdx)
        {
            int64_t offsetLimitBlockDataSize = 0;
            for (int32_t i = 0; i < offsetLimitBlockIdx; i++)
            {
                offsetLimitBlockDataSize += GetBlockSize(i);
            }
            loadSize_ = std::min((offset + limit) - offsetLimitBlockDataSize, currentBlockSize) - loadOffset_;
        }

        if (blockIndex_ > offsetBlockIdx && blockIndex_ < offsetLimitBlockIdx)
        {
            loadSize_ = GetBlockSize();
        }

        CudaLogBoost::getInstance(CudaLogBoost::info) << "OffsetBlockIdx: " << offsetBlockIdx << '\n';
        CudaLogBoost::getInstance(CudaLogBoost::info) << "OffsetLimitBlockIdx: " << offsetLimitBlockIdx << '\n';
        CudaLogBoost::getInstance(CudaLogBoost::info) << "Block Load Size: " << loadSize_ << '\n';
        CudaLogBoost::getInstance(CudaLogBoost::info) << "Block Load Offset: " << loadOffset_ << '\n';
    }

    return InstructionStatus::CONTINUE;
}

void GpuSqlDispatcher::ShiftNullMaskLeft(std::vector<nullmask_t>& mask, int64_t shift)
{
    while (shift-- > 0)
    {
        uint64_t carryBit = 0;
        for (int32_t i = mask.size() - 1; i >= 0; i--)
        {
            uint64_t newCarryBit = mask[i] & 1;
            mask[i] >>= 1;
            uint64_t a = 9223372036854775807;
            mask[i] &= a;
            mask[i] |= (carryBit << 63);
            carryBit = newCarryBit;
        }
    }
}


template <>
void GpuSqlDispatcher::FillCompositeDataTypeRegister<std::string>(GpuSqlDispatcher::CompositeDataType<std::string> column,
                                                                  const std::string& reg,
                                                                  int32_t size,
                                                                  bool useCache,
                                                                  nullmask_t* nullMaskPtr)
{
    InsertRegister(reg + "_stringIndices",
                   PointerAllocation{reinterpret_cast<uintptr_t>(column.stringIndices), size,
                                     !useCache, reinterpret_cast<uintptr_t>(nullMaskPtr)});
    InsertRegister(reg + "_allChars", PointerAllocation{reinterpret_cast<uintptr_t>(column.allChars), size,
                                                        !useCache, reinterpret_cast<uintptr_t>(nullMaskPtr)});
}

template <>
void GpuSqlDispatcher::FillCompositeDataTypeRegister<ColmnarDB::Types::ComplexPolygon>(
    GpuSqlDispatcher::CompositeDataType<ColmnarDB::Types::ComplexPolygon> column,
    const std::string& reg,
    int32_t size,
    bool useCache,
    nullmask_t* nullMaskPtr)
{
    InsertRegister(reg + "_polyPoints",
                   PointerAllocation{reinterpret_cast<uintptr_t>(column.polyPoints), size,
                                     !useCache, reinterpret_cast<uintptr_t>(nullMaskPtr)});
    InsertRegister(reg + "_pointIdx", PointerAllocation{reinterpret_cast<uintptr_t>(column.pointIdx), size,
                                                        !useCache, reinterpret_cast<uintptr_t>(nullMaskPtr)});
    InsertRegister(reg + "_polyIdx", PointerAllocation{reinterpret_cast<uintptr_t>(column.polyIdx), size,
                                                       !useCache, reinterpret_cast<uintptr_t>(nullMaskPtr)});
}

template <>
GpuSqlDispatcher::CompositeDataType<std::string>
GpuSqlDispatcher::InsertConstCompositeDataType<std::string>(const std::string& str, size_t size)
{
    std::vector<std::string> strings(size, str);
    std::string name = "constString" + std::to_string(constStringCounter_);
    constStringCounter_++;
    return InsertString(database_->GetName(), name, strings.data(), size);
}

template <>
GpuSqlDispatcher::CompositeDataType<ColmnarDB::Types::ComplexPolygon>
GpuSqlDispatcher::InsertConstCompositeDataType<ColmnarDB::Types::ComplexPolygon>(const std::string& str, size_t size)
{
    std::string name = "constPolygon" + std::to_string(constPolygonCounter_);
    constPolygonCounter_++;
    return InsertComplexPolygon(database_->GetName(), name, {ComplexPolygonFactory::FromWkt(str)}, size);
}


GPUMemory::GPUPolygon
GpuSqlDispatcher::InsertComplexPolygon(const std::string& databaseName,
                                       const std::string& colName,
                                       const std::vector<ColmnarDB::Types::ComplexPolygon>& polygons,
                                       int32_t size,
                                       bool useCache,
                                       nullmask_t* nullMaskPtr)
{
    if (useCache)
    {
        if (Context::getInstance().getCacheForCurrentDevice().containsColumn(databaseName, colName + "_polyPoints",
                                                                             blockIndex_, loadSize_, loadOffset_) &&
            Context::getInstance().getCacheForCurrentDevice().containsColumn(databaseName, colName + "_pointIdx",
                                                                             blockIndex_, loadSize_, loadOffset_) &&
            Context::getInstance().getCacheForCurrentDevice().containsColumn(databaseName, colName + "_polyIdx",
                                                                             blockIndex_, loadSize_, loadOffset_))
        {
            GPUMemoryCache& cache = Context::getInstance().getCacheForCurrentDevice();
            GPUMemory::GPUPolygon polygon;
            polygon.polyPoints =
                std::get<0>(cache.getColumn<NativeGeoPoint>(databaseName, colName + "_polyPoints",
                                                            blockIndex_, size, loadSize_, loadOffset_));
            polygon.pointIdx =
                std::get<0>(cache.getColumn<int32_t>(databaseName, colName + "_pointIdx",
                                                     blockIndex_, size, loadSize_, loadOffset_));
            polygon.polyIdx = std::get<0>(cache.getColumn<int32_t>(databaseName, colName + "_polyIdx", blockIndex_,
                                                                   size, loadSize_, loadOffset_));

            FillCompositeDataTypeRegister<ColmnarDB::Types::ComplexPolygon>(polygon, colName, size,
                                                                            useCache, nullMaskPtr);

            return polygon;
        }
        else
        {
            GPUMemory::GPUPolygon polygon =
                ComplexPolygonFactory::PrepareGPUPolygon(polygons, databaseName, colName,
                                                         blockIndex_, loadSize_, loadOffset_);
            FillCompositeDataTypeRegister<ColmnarDB::Types::ComplexPolygon>(polygon, colName, size,
                                                                            useCache, nullMaskPtr);
            return polygon;
        }
    }
    else
    {
        GPUMemory::GPUPolygon polygon = ComplexPolygonFactory::PrepareGPUPolygon(polygons);
        FillCompositeDataTypeRegister<ColmnarDB::Types::ComplexPolygon>(polygon, colName, size,
                                                                        useCache, nullMaskPtr);
        return polygon;
    }
}

GPUMemory::GPUString GpuSqlDispatcher::InsertString(const std::string& databaseName,
                                                    const std::string& colName,
                                                    const std::string* strings,
                                                    const size_t stringCount,
                                                    bool useCache,
                                                    nullmask_t* nullMaskPtr)
{
    if (useCache)
    {
        if (Context::getInstance().getCacheForCurrentDevice().containsColumn(databaseName, colName + "_stringIndices",
                                                                             blockIndex_, loadSize_, loadOffset_) &&
            Context::getInstance().getCacheForCurrentDevice().containsColumn(databaseName, colName + "_allChars",
                                                                             blockIndex_, loadSize_, loadOffset_))
        {
            GPUMemoryCache& cache = Context::getInstance().getCacheForCurrentDevice();
            GPUMemory::GPUString gpuString;
            gpuString.stringIndices =
                std::get<0>(cache.getColumn<int64_t>(databaseName, colName + "_stringIndices",
                                                     blockIndex_, stringCount, loadSize_, loadOffset_));
            gpuString.allChars =
                std::get<0>(cache.getColumn<char>(databaseName, colName + "_allChars", blockIndex_,
                                                  stringCount, loadSize_, loadOffset_));
            FillCompositeDataTypeRegister<std::string>(gpuString, colName, stringCount, useCache, nullMaskPtr);
            return gpuString;
        }
        else
        {
            GPUMemory::GPUString gpuString =
                StringFactory::PrepareGPUString(strings, stringCount, databaseName, colName,
                                                blockIndex_, loadSize_, loadOffset_);
            FillCompositeDataTypeRegister<std::string>(gpuString, colName, stringCount, useCache, nullMaskPtr);
            return gpuString;
        }
    }
    else
    {
        GPUMemory::GPUString gpuString = StringFactory::PrepareGPUString(strings, stringCount);
        FillCompositeDataTypeRegister<std::string>(gpuString, colName, stringCount, useCache, nullMaskPtr);
        return gpuString;
    }
}