#include "hip/hip_runtime.h"
#include "GpuSqlDispatcherVMFunctions.h"
#include <array>
#include "../ParserExceptions.h"
#include "../../PointFactory.h"

std::array<GpuSqlDispatcher::DispatchFunction, DataType::DATA_TYPE_SIZE> GpuSqlDispatcher::retFunctions = { &GpuSqlDispatcher::retConst<int32_t>, &GpuSqlDispatcher::retConst<int64_t>, &GpuSqlDispatcher::retConst<float>, &GpuSqlDispatcher::retConst<double>, &GpuSqlDispatcher::retConst<ColmnarDB::Types::Point>, &GpuSqlDispatcher::retConst<ColmnarDB::Types::ComplexPolygon>, &GpuSqlDispatcher::retConst<std::string>, &GpuSqlDispatcher::invalidOperandTypesErrorHandlerConst<int8_t>, &GpuSqlDispatcher::retCol<int32_t>, &GpuSqlDispatcher::retCol<int64_t>, &GpuSqlDispatcher::retCol<float>, &GpuSqlDispatcher::retCol<double>, &GpuSqlDispatcher::retCol<ColmnarDB::Types::Point>, &GpuSqlDispatcher::retCol<ColmnarDB::Types::ComplexPolygon>, &GpuSqlDispatcher::retCol<std::string>, &GpuSqlDispatcher::invalidOperandTypesErrorHandlerCol<int8_t> };
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::filFunction = &GpuSqlDispatcher::fil;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::jmpFunction = &GpuSqlDispatcher::jmp;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::doneFunction = &GpuSqlDispatcher::done;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::showDatabasesFunction = &GpuSqlDispatcher::showDatabases;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::showTablesFunction = &GpuSqlDispatcher::showTables;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::showColumnsFunction = &GpuSqlDispatcher::showColumns;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::insertIntoDoneFunction = &GpuSqlDispatcher::insertIntoDone;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::createDatabaseFunction = &GpuSqlDispatcher::createDatabase;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::dropDatabaseFunction = &GpuSqlDispatcher::dropDatabase;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::createTableFunction = &GpuSqlDispatcher::createTable;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::dropTableFunction = &GpuSqlDispatcher::dropTable;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::alterTableFunction = &GpuSqlDispatcher::alterTable;
GpuSqlDispatcher::DispatchFunction GpuSqlDispatcher::createIndexFunction = &GpuSqlDispatcher::createIndex;

template <>
int32_t GpuSqlDispatcher::loadCol<ColmnarDB::Types::ComplexPolygon>(std::string& colName)
{
	if (allocatedPointers.find(colName) == allocatedPointers.end() && !colName.empty() && colName.front() != '$')
	{
		std::cout << "Load: " << colName << " " << typeid(ColmnarDB::Types::ComplexPolygon).name() << std::endl;

		std::string table;
		std::string column;

		std::tie(table, column) = splitColumnName(colName);

		const int32_t blockCount = database->GetTables().at(table).GetColumns().at(column).get()->GetBlockCount();
		GpuSqlDispatcher::groupByDoneLimit_ = std::min(Context::getInstance().getDeviceCount() - 1, blockCount - 1);
		if (blockIndex >= blockCount)
		{
			return 1;
		}
		if (blockIndex >= blockCount - Context::getInstance().getDeviceCount())
		{
			isLastBlockOfDevice = true;
		}
		if (blockIndex == blockCount - 1)
		{
			isOverallLastBlock = true;
		}

		auto col = dynamic_cast<const ColumnBase<ColmnarDB::Types::ComplexPolygon>*>(database->GetTables().at(table).GetColumns().at(column).get());


		if (!usingJoin)
		{
			auto block = dynamic_cast<BlockBase<ColmnarDB::Types::ComplexPolygon>*>(col->GetBlocksList()[blockIndex]);
			int8_t* nullMaskPtr = nullptr;
			if(block->GetNullBitmask())
			{
				int32_t bitMaskCapacity = ((block->GetSize() + sizeof(int8_t)*8 - 1) / (8*sizeof(int8_t)));
				nullMaskPtr = allocateRegister<int8_t>(colName + "_nullmask", bitMaskCapacity);
				GPUMemory::copyHostToDevice(nullMaskPtr, block->GetNullBitmask(), bitMaskCapacity);
			}
			insertComplexPolygon(database->GetName(), colName,
				std::vector<ColmnarDB::Types::ComplexPolygon>(block->GetData(),
					block->GetData() + block->GetSize()),
				block->GetSize(), false, nullMaskPtr);
			noLoad = false;
		}
		else
		{
			std::cout << "Loading joined block." << std::endl;
			int32_t loadSize = joinIndices->at(table)[blockIndex].size();
			std::string joinCacheId = colName + "_join";
			for (auto& joinTable : *joinIndices)
			{
				joinCacheId += "_" + joinTable.first;
			}

			std::vector<ColmnarDB::Types::ComplexPolygon> joinedPolygons;

			int32_t outDataSize;
			GPUJoin::reorderByJoinTableCPUKeep<ColmnarDB::Types::ComplexPolygon>(joinedPolygons, outDataSize, *col, blockIndex, joinIndices->at(table), database->GetBlockSize());
			
			insertComplexPolygon(database->GetName(), joinCacheId, joinedPolygons, loadSize);
			noLoad = false;
		}
	}
	return 0;
}

template <>
int32_t GpuSqlDispatcher::loadCol<ColmnarDB::Types::Point>(std::string& colName)
{
	if (allocatedPointers.find(colName) == allocatedPointers.end() && !colName.empty() && colName.front() != '$')
	{
		std::cout << "Load: " << colName << " " << typeid(ColmnarDB::Types::Point).name() << std::endl;

		std::string table;
		std::string column;

		std::tie(table, column) = splitColumnName(colName);

		const int32_t blockCount = database->GetTables().at(table).GetColumns().at(column).get()->GetBlockCount();
		GpuSqlDispatcher::groupByDoneLimit_ = std::min(Context::getInstance().getDeviceCount() - 1, blockCount - 1);
		if (blockIndex >= blockCount)
		{
			return 1;
		}
		if (blockIndex >= blockCount - Context::getInstance().getDeviceCount())
		{
			isLastBlockOfDevice = true;
		}
		if (blockIndex == blockCount - 1)
		{
			isOverallLastBlock = true;
		}

		auto col = dynamic_cast<const ColumnBase<ColmnarDB::Types::Point>*>(database->GetTables().at(table).GetColumns().at(column).get());
		
		if (!usingJoin)
		{
			auto block = dynamic_cast<BlockBase<ColmnarDB::Types::Point>*>(col->GetBlocksList()[blockIndex]);

			std::vector<NativeGeoPoint> nativePoints;
			std::transform(block->GetData(), block->GetData() + block->GetSize(), std::back_inserter(nativePoints), [](const ColmnarDB::Types::Point& point) -> NativeGeoPoint { return NativeGeoPoint{ point.geopoint().latitude(), point.geopoint().longitude() }; });

			auto cacheEntry =
				Context::getInstance().getCacheForCurrentDevice().getColumn<NativeGeoPoint>(database->GetName(), colName, blockIndex,
					nativePoints.size());
			if (!std::get<2>(cacheEntry))
			{
				GPUMemory::copyHostToDevice(std::get<0>(cacheEntry),
					reinterpret_cast<NativeGeoPoint*>(nativePoints.data()),
					nativePoints.size());
			}
			int8_t* nullMaskPtr = nullptr;
			if(block->GetNullBitmask())
			{
				int32_t bitMaskCapacity = ((block->GetSize() + sizeof(int8_t)*8 - 1) / (8*sizeof(int8_t)));
				auto cacheMaskEntry = Context::getInstance().getCacheForCurrentDevice().getColumn<int8_t>(
					database->GetName(), colName + "_nullmask", blockIndex, bitMaskCapacity);
				nullMaskPtr = std::get<0>(cacheMaskEntry);
				if (!std::get<2>(cacheMaskEntry))
				{
					GPUMemory::copyHostToDevice(std::get<0>(cacheMaskEntry), block->GetNullBitmask(), bitMaskCapacity);
				}
			}
			addCachedRegister(colName, std::get<0>(cacheEntry), nativePoints.size(), nullMaskPtr);
			noLoad = false;
		
		}
		else
		{
			std::cout << "Loading joined block." << std::endl;
			int32_t loadSize = joinIndices->at(table)[blockIndex].size();
			std::string joinCacheId = colName + "_join";
			for (auto& joinTable : *joinIndices)
			{
				joinCacheId += "_" + joinTable.first;
			}

			std::vector<ColmnarDB::Types::Point> joinedPoints;
			int32_t outDataSize;
			GPUJoin::reorderByJoinTableCPUKeep<ColmnarDB::Types::Point>(joinedPoints, outDataSize, *col, blockIndex, joinIndices->at(table), database->GetBlockSize());

			std::vector<NativeGeoPoint> nativePoints;
			std::transform(joinedPoints.data(), joinedPoints.data() + loadSize, std::back_inserter(nativePoints), [](const ColmnarDB::Types::Point& point) -> NativeGeoPoint { return NativeGeoPoint{ point.geopoint().latitude(), point.geopoint().longitude() }; });

			auto cacheEntry = Context::getInstance().getCacheForCurrentDevice().getColumn<NativeGeoPoint>(
				database->GetName(), joinCacheId, blockIndex, loadSize);
			if (!std::get<2>(cacheEntry))
			{
				GPUMemory::copyHostToDevice(std::get<0>(cacheEntry),
					reinterpret_cast<NativeGeoPoint*>(nativePoints.data()),
					nativePoints.size());
			}
			addCachedRegister(joinCacheId, std::get<0>(cacheEntry), loadSize);
			noLoad = false;
		}
	}
	return 0;
}


template <>
int32_t GpuSqlDispatcher::loadCol<std::string>(std::string& colName)
{
	if (allocatedPointers.find(colName) == allocatedPointers.end() && !colName.empty() && colName.front() != '$')
	{
		std::cout << "Load: " << colName << " " << typeid(std::string).name() << std::endl;

		std::string table;
		std::string column;

		std::tie(table, column) = splitColumnName(colName);

		const int32_t blockCount = database->GetTables().at(table).GetColumns().at(column).get()->GetBlockCount();
		GpuSqlDispatcher::groupByDoneLimit_ = std::min(Context::getInstance().getDeviceCount() - 1, blockCount - 1);
		if (blockIndex >= blockCount)
		{
			return 1;
		}
		if (blockIndex >= blockCount - Context::getInstance().getDeviceCount())
		{
			isLastBlockOfDevice = true;
		}
		if (blockIndex == blockCount - 1)
		{
			isOverallLastBlock = true;
		}

		auto col = dynamic_cast<const ColumnBase<std::string>*>(database->GetTables().at(table).GetColumns().at(column).get());

		if (!usingJoin)
		{
			auto block = dynamic_cast<BlockBase<std::string>*>(col->GetBlocksList()[blockIndex]);
			int8_t* nullMaskPtr = nullptr;
			if(block->GetNullBitmask())
			{
				int32_t bitMaskCapacity = ((block->GetSize() + sizeof(int8_t)*8 - 1) / (8*sizeof(int8_t)));
				nullMaskPtr = allocateRegister<int8_t>(colName + "_nullmask", bitMaskCapacity);
				GPUMemory::copyHostToDevice(nullMaskPtr, block->GetNullBitmask(), bitMaskCapacity);
			}
			insertString(database->GetName(), colName, std::vector<std::string>(block->GetData(), 
				block->GetData() + block->GetSize()),
				block->GetSize(), false, nullMaskPtr);
			noLoad = false;
		}
		else
		{
			std::cout << "Loading joined block." << std::endl;
			int32_t loadSize = joinIndices->at(table)[blockIndex].size();
			std::string joinCacheId = colName + "_join";
			for (auto& joinTable : *joinIndices)
			{
				joinCacheId += "_" + joinTable.first;
			}

			std::vector<std::string> joinedStrings;

			int32_t outDataSize;
			GPUJoin::reorderByJoinTableCPUKeep<std::string>(joinedStrings, outDataSize, *col, blockIndex, joinIndices->at(table), database->GetBlockSize());

			insertString(database->GetName(), joinCacheId, joinedStrings, loadSize);
			noLoad = false;
		}

	}
	return 0;
}

template <>
int32_t GpuSqlDispatcher::retCol<ColmnarDB::Types::ComplexPolygon>()
{
	if (usingGroupBy)
	{
		throw RetPolygonGroupByException();
	}
	else
	{
		auto col = arguments.read<std::string>();
		auto alias = arguments.read<std::string>();

		int32_t loadFlag = loadCol<ColmnarDB::Types::ComplexPolygon>(col);
		if (loadFlag)
		{
			return loadFlag;
		}
		std::cout << "RetPolygonCol: " << col << ", thread: " << dispatcherThreadId << std::endl;

		std::unique_ptr<std::string[]> outData(new std::string[database->GetBlockSize()]);
		std::tuple<GPUMemory::GPUPolygon, int32_t, int8_t*> ACol = findComplexPolygon(getAllocatedRegisterName(col));
		int32_t outSize;
		std::string nullMaskString = "";
		if(std::get<2>(ACol))
		{
			size_t bitMaskSize = (database->GetBlockSize() + sizeof(char)*8 - 1) / (sizeof(char)*8);
			std::unique_ptr<int8_t[]> nullMask = std::unique_ptr<int8_t[]>(new int8_t[bitMaskSize]);
			GPUReconstruct::ReconstructPolyColToWKT(outData.get(), &outSize,
			std::get<0>(ACol), reinterpret_cast<int8_t*>(filter_), std::get<1>(ACol),nullMask.get(), std::get<2>(ACol));
			bitMaskSize = (outSize + sizeof(char)*8 - 1) / (sizeof(char)*8);
			nullMaskString = std::string(reinterpret_cast<char*>(nullMask.get()), bitMaskSize);
		}
		else
		{
			GPUReconstruct::ReconstructPolyColToWKT(outData.get(), &outSize,
			std::get<0>(ACol), reinterpret_cast<int8_t*>(filter_), std::get<1>(ACol));
		}
		std::cout << "dataSize: " << outSize << std::endl;
		ColmnarDB::NetworkClient::Message::QueryResponsePayload payload;
		insertIntoPayload(payload, outData, outSize);
		MergePayloadToSelfResponse(alias, payload, nullMaskString);
	}
	return 0;
}

template<>
int32_t GpuSqlDispatcher::retCol<ColmnarDB::Types::Point>()
{
	if (usingGroupBy)
	{
		throw RetPointGroupByException();
	}
	else
	{
		auto colName = arguments.read<std::string>();
		auto alias = arguments.read<std::string>();

		int32_t loadFlag = loadCol<ColmnarDB::Types::Point>(colName);
		if (loadFlag)
		{
			return loadFlag;
		}

		std::cout << "RetPointCol: " << colName << ", thread: " << dispatcherThreadId << std::endl;

		std::unique_ptr<std::string[]> outData(new std::string[database->GetBlockSize()]);
		PointerAllocation ACol = allocatedPointers.at(getAllocatedRegisterName(colName));
		int32_t outSize;
		//ToDo: Podmienene zapnut podla velkost buffera
		//GPUMemory::hostPin(outData.get(), database->GetBlockSize());
		
		std::string nullMaskString = "";
		if(ACol.gpuNullMaskPtr)
		{
			size_t bitMaskSize = (database->GetBlockSize() + sizeof(char)*8 - 1) / (sizeof(char)*8);
			std::unique_ptr<int8_t[]> nullMask = std::unique_ptr<int8_t[]>(new int8_t[bitMaskSize]);
			GPUReconstruct::ReconstructPointColToWKT(outData.get(), &outSize,
				reinterpret_cast<NativeGeoPoint*>(ACol.gpuPtr), reinterpret_cast<int8_t*>(filter_), ACol.elementCount,
				nullMask.get(), reinterpret_cast<int8_t*>(ACol.gpuNullMaskPtr));
		    bitMaskSize = (outSize + sizeof(char)*8 - 1) / (sizeof(char)*8); 
			nullMaskString = std::string(reinterpret_cast<char*>(nullMask.get()), bitMaskSize);
		}
		else
		{
			GPUReconstruct::ReconstructPointColToWKT(outData.get(), &outSize,
				reinterpret_cast<NativeGeoPoint*>(ACol.gpuPtr), reinterpret_cast<int8_t*>(filter_), ACol.elementCount);
		}
		//GPUMemory::hostUnregister(outData.get());

		std::cout << "dataSize: " << outSize << std::endl;
		ColmnarDB::NetworkClient::Message::QueryResponsePayload payload;
		insertIntoPayload(payload, outData, outSize);
		MergePayloadToSelfResponse(alias, payload, nullMaskString);
	}
	return 0;
}

template <>
int32_t GpuSqlDispatcher::retCol<std::string>()
{
	auto colName = arguments.read<std::string>();
	auto alias = arguments.read<std::string>();

	int32_t loadFlag = loadCol<std::string>(colName);
	if (loadFlag)
	{
		return loadFlag;
	}

	std::cout << "RetStringCol: " << colName << ", thread: " << dispatcherThreadId << std::endl;
	
	int32_t outSize;
	std::unique_ptr<std::string[]> outData;
	std::string nullMaskString = "";
	if (usingGroupBy)
	{
		if (isOverallLastBlock)
		{
			// Return key or value col (key if groupByColumns contains colName)
			auto col = findStringColumn(getAllocatedRegisterName(colName) + (groupByColumns.find(colName) != groupByColumns.end() ? "_keys" : ""));
			outSize = std::get<1>(col);
			outData = std::make_unique<std::string[]>(outSize);
			if(std::get<2>(col))
			{
				size_t bitMaskSize = (database->GetBlockSize() + sizeof(char)*8 - 1) / (sizeof(char)*8);
				std::unique_ptr<int8_t[]> nullMask = std::unique_ptr<int8_t[]>(new int8_t[bitMaskSize]);
				GPUReconstruct::ReconstructStringCol(outData.get(), &outSize,
					std::get<0>(col), nullptr, std::get<1>(col), nullMask.get(), std::get<2>(col));
					bitMaskSize = (outSize + sizeof(char)*8 - 1) / (sizeof(char)*8);
				nullMaskString = std::string(reinterpret_cast<char*>(nullMask.get()), bitMaskSize);
			}
			else
			{
				GPUReconstruct::ReconstructStringCol(outData.get(), &outSize,
					std::get<0>(col), nullptr, std::get<1>(col));
			}
		}
		else
		{
			return 0;
		}
	}
	else
	{
		std::tuple<GPUMemory::GPUString, int32_t> col = findStringColumn(getAllocatedRegisterName(colName));
		outSize = std::get<1>(col);
		outData = std::make_unique<std::string[]>(outSize);
		if(std::get<2>(col))
		{
			size_t bitMaskSize = (database->GetBlockSize() + sizeof(char)*8 - 1) / (sizeof(char)*8);
			std::unique_ptr<int8_t[]> nullMask = std::unique_ptr<int8_t[]>(new int8_t[bitMaskSize]);
			GPUReconstruct::ReconstructStringCol(outData.get(), &outSize,
				std::get<0>(col), reinterpret_cast<int8_t*>(filter_), std::get<1>(col), nullMask.get(), std::get<2>(col));
				bitMaskSize = (outSize + sizeof(char)*8 - 1) / (sizeof(char)*8);
			nullMaskString = std::string(reinterpret_cast<char*>(nullMask.get()), bitMaskSize);
		}
		else
		{
			GPUReconstruct::ReconstructStringCol(outData.get(), &outSize,
				std::get<0>(col), reinterpret_cast<int8_t*>(filter_), std::get<1>(col));
		}
		std::cout << "dataSize: " << outSize << std::endl;
	}

	ColmnarDB::NetworkClient::Message::QueryResponsePayload payload;
	insertIntoPayload(payload, outData, outSize);
	MergePayloadToSelfResponse(alias, payload, nullMaskString);
	return 0;
}
