#include "hip/hip_runtime.h"
#include "GPUMergeJoin.cuh"

__global__ void kernel_label_input(int32_t *colBlockIndices, int32_t blockOffset, int32_t dataElementCount) 
{
	const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t stride = blockDim.x * gridDim.x;

	for (int32_t i = idx; i < dataElementCount; i += stride)
	{
		colBlockIndices[i] = blockOffset + i;
	}
}