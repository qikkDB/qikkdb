#include "hip/hip_runtime.h"
#include "GPUGroupByMultiKey.cuh"

__device__ int32_t GetHash(DataType* keyTypes,
                           const int32_t keysColCount,
                           void** inKeys,
                           nullmask_t** inKeysNullMask,
                           const int32_t i,
                           const int32_t hashCoef)
{
    uint32_t crc = 0xFFFFFFFF;

    for (int32_t t = 0; t < keysColCount; t++)
    {
        uint32_t hash;
        const bool null = (inKeysNullMask[t] != nullptr) &&
                          NullValues::GetConcreteBitFromBitmask(inKeysNullMask[t], i);
        if (null)
        {
            hash = 0;
        }
        else
        {
            // Compute one 32-bit (uint32_t) number from one value
            switch (keyTypes[t])
            {
            case DataType::COLUMN_INT:
                hash = reinterpret_cast<uint32_t*>(inKeys[t])[i];
                break;
            case DataType::COLUMN_LONG:
            {
                // use XOR of upper 32 bits and lower 32 bits
                const uint64_t allBits = reinterpret_cast<uint64_t*>(inKeys[t])[i];
                hash = static_cast<uint32_t>((allBits & 0xFFFFFFFFULL) ^ (allBits >> 32));
                break;
            }
            case DataType::COLUMN_FLOAT:
                // Use float bit representation
                hash = reinterpret_cast<uint32_t*>(inKeys[t])[i];
                break;
            case DataType::COLUMN_DOUBLE:
            {
                // Use XOR of upper 32 bits and lower 32 bits of double bit representation
                const uint64_t allBits = reinterpret_cast<uint64_t*>(inKeys[t])[i];
                hash = static_cast<uint32_t>((allBits & 0xFFFFFFFFULL) ^ (allBits >> 32));
                break;
            }
            case DataType::COLUMN_STRING:
            {
                GPUMemory::GPUString strCol = *reinterpret_cast<GPUMemory::GPUString*>(inKeys[t]);
                hash = GetHash(strCol.allChars + GetStringIndex(strCol.stringIndices, i),
                               GetStringLength(strCol.stringIndices, i));
            }
            break;
            case DataType::COLUMN_INT8_T:
                hash = static_cast<uint32_t>(reinterpret_cast<uint8_t*>(inKeys[t])[i]);
                break;
            default:
                hash = 0;
                break;
            }
        }
        for (int32_t i = 0; i < 4; i++)
        {
            crc = (CRC_32_TAB[((crc >> 24) ^ ((hash >> (i * 8)) ^ 0xFF)) & 0xFF] ^ (crc << 8));
        }
    }
    return (crc >> hashCoef) ^ (crc & ((1 << hashCoef) - 1));
}


__device__ bool AreEqualMultiKeys(DataType* keyTypes,
                                  const int32_t keysColCount,
                                  void** keysA,
                                  nullmask_t** keysANullMask,
                                  const int32_t indexA,
                                  void** keysB,
                                  nullmask_t** keysBNullMask,
                                  const int32_t indexB,
                                  const bool compressedBNullMask)
{
    for (int32_t t = 0; t < keysColCount; t++)
    {
        const bool nullA = (keysANullMask[t] != nullptr) &&
                           (NullValues::GetConcreteBitFromBitmask(keysANullMask[t], indexA));
        const bool nullB =
            (keysBNullMask[t] != nullptr) &&
            (compressedBNullMask ?
                 (NullValues::GetConcreteBitFromBitmask(keysBNullMask[t], indexB)) :
                 keysBNullMask[t][indexB]);
        switch (keyTypes[t])
        {
        case DataType::COLUMN_INT:
            if (nullA != nullB || (!nullA && reinterpret_cast<int32_t*>(keysA[t])[indexA] !=
                                                 reinterpret_cast<int32_t*>(keysB[t])[indexB]))
            {
                return false;
            }
            break;
        case DataType::COLUMN_LONG:
            if (nullA != nullB || (!nullA && reinterpret_cast<int64_t*>(keysA[t])[indexA] !=
                                                 reinterpret_cast<int64_t*>(keysB[t])[indexB]))
            {
                return false;
            }
            break;
        case DataType::COLUMN_FLOAT:
            if (nullA != nullB || (!nullA && reinterpret_cast<float*>(keysA[t])[indexA] !=
                                                 reinterpret_cast<float*>(keysB[t])[indexB]))
            {
                return false;
            }
            break;
        case DataType::COLUMN_DOUBLE:
            if (nullA != nullB || (!nullA && reinterpret_cast<double*>(keysA[t])[indexA] !=
                                                 reinterpret_cast<double*>(keysB[t])[indexB]))
            {
                return false;
            }
            break;
        case DataType::COLUMN_STRING:
        {
            GPUMemory::GPUString strColA = *reinterpret_cast<GPUMemory::GPUString*>(keysA[t]);
            GPUMemory::GPUString strColB = *reinterpret_cast<GPUMemory::GPUString*>(keysB[t]);
            if (nullA != nullB ||
                (!nullA && !AreEqualStrings(strColA.allChars + GetStringIndex(strColA.stringIndices, indexA),
                                            GetStringLength(strColA.stringIndices, indexA), strColB, indexB)))
            {
                return false;
            }
            break;
        }
        case DataType::COLUMN_INT8_T:
            if (nullA != nullB || (!nullA && reinterpret_cast<int8_t*>(keysA[t])[indexA] !=
                                                 reinterpret_cast<int8_t*>(keysB[t])[indexB]))
            {
                return false;
            }
            break;
        default:
            break;
        }
    }
    return true;
}


__device__ bool IsNewMultiKey(DataType* keyTypes,
                              const int32_t keysColCount,
                              void** inKeys,
                              nullmask_t** inKeysNullMask,
                              const int32_t i,
                              void** keysBuffer,
                              nullmask_t** keysNullBuffer,
                              int32_t* sourceIndices,
                              const int32_t index)
{
    return (sourceIndices[index] >= 0 &&
            !AreEqualMultiKeys(keyTypes, keysColCount, inKeys, inKeysNullMask, i, inKeys,
                               inKeysNullMask, sourceIndices[index], true)) ||
           (sourceIndices[index] == GBS_SOURCE_INDEX_KEY_IN_BUFFER &&
            !AreEqualMultiKeys(keyTypes, keysColCount, inKeys, inKeysNullMask, i, keysBuffer,
                               keysNullBuffer, index, false));
}


template <>
void ReconstructSingleKeyColKeep<std::string>(std::vector<void*>* outKeysVector,
                                              int32_t* outDataElementCount,
                                              int8_t* occupancyMaskPtr,
                                              void** keyCol,
                                              const int32_t elementCount)
{
    // Copy struct (we need to get pointer to struct at first)
    GPUMemory::GPUString* structPointer;
    GPUMemory::copyDeviceToHost(&structPointer, reinterpret_cast<GPUMemory::GPUString**>(keyCol), 1);
    GPUMemory::GPUString keyBufferSingleCol;
    GPUMemory::copyDeviceToHost(&keyBufferSingleCol, structPointer, 1);

    // Reconstruct string keys
    GPUMemory::GPUString* outKeysSingleCol = new GPUMemory::GPUString();
    GPUReconstruct::ReconstructStringColKeep(outKeysSingleCol, outDataElementCount,
                                             keyBufferSingleCol, occupancyMaskPtr, elementCount);

    outKeysVector->emplace_back(outKeysSingleCol);
}


template <>
void ReconstructSingleKeyCol<std::string>(std::vector<void*>* outKeysVector,
                                          int32_t* outDataElementCount,
                                          int8_t* occupancyMaskPtr,
                                          void** keyCol,
                                          int32_t elementCount)
{
    // Copy struct (we need to get pointer to struct at first)
    GPUMemory::GPUString* structPointer;
    GPUMemory::copyDeviceToHost(&structPointer, reinterpret_cast<GPUMemory::GPUString**>(keyCol), 1);
    GPUMemory::GPUString keyBufferSingleCol;
    GPUMemory::copyDeviceToHost(&keyBufferSingleCol, structPointer, 1);

    // Reconstruct string as raw
    std::vector<int32_t> stringLengths;
    std::vector<char> allChars;
    GPUReconstruct::ReconstructStringColRaw(stringLengths, allChars, outDataElementCount,
                                            keyBufferSingleCol, occupancyMaskPtr, elementCount);

    CPUString* outKeysSingleCol = new CPUString{stringLengths, allChars};
    outKeysVector->emplace_back(outKeysSingleCol);
}


void AllocKeysBuffer(void*** keysBuffer,
                     nullmask_t*** keysNullBuffer,
                     std::vector<DataType>& keyTypes,
                     int32_t rowCount,
                     std::vector<void*>* pointers,
                     std::vector<nullmask_t*>* pointersNullMask)
{
    GPUMemory::alloc(keysBuffer, keyTypes.size());
    GPUMemory::alloc(keysNullBuffer, keyTypes.size());
    for (int32_t i = 0; i < keyTypes.size(); i++)
    {
        switch (keyTypes[i])
        {
        case DataType::COLUMN_INT:
        {
            int32_t* gpuKeyCol;
            GPUMemory::alloc(&gpuKeyCol, rowCount);
            GPUMemory::copyHostToDevice(reinterpret_cast<int32_t**>(*keysBuffer + i), &gpuKeyCol, 1);
            if (pointers)
            {
                pointers->emplace_back(gpuKeyCol);
            }
            break;
        }
        case DataType::COLUMN_LONG:
        {
            int64_t* gpuKeyCol;
            GPUMemory::alloc(&gpuKeyCol, rowCount);
            GPUMemory::copyHostToDevice(reinterpret_cast<int64_t**>(*keysBuffer + i), &gpuKeyCol, 1);
            if (pointers)
            {
                pointers->emplace_back(gpuKeyCol);
            }
            break;
        }
        case DataType::COLUMN_FLOAT:
        {
            float* gpuKeyCol;
            GPUMemory::alloc(&gpuKeyCol, rowCount);
            GPUMemory::copyHostToDevice(reinterpret_cast<float**>(*keysBuffer + i), &gpuKeyCol, 1);
            if (pointers)
            {
                pointers->emplace_back(gpuKeyCol);
            }
            break;
        }
        case DataType::COLUMN_DOUBLE:
        {
            double* gpuKeyCol;
            GPUMemory::alloc(&gpuKeyCol, rowCount);
            GPUMemory::copyHostToDevice(reinterpret_cast<double**>(*keysBuffer + i), &gpuKeyCol, 1);
            if (pointers)
            {
                pointers->emplace_back(gpuKeyCol);
            }
            break;
        }
        case DataType::COLUMN_STRING:
        {
            GPUMemory::GPUString emptyStringCol{nullptr, nullptr};
            GPUMemory::GPUString* gpuKeyCol;
            GPUMemory::alloc(&gpuKeyCol, 1);
            GPUMemory::copyHostToDevice(gpuKeyCol, &emptyStringCol, 1);
            GPUMemory::copyHostToDevice(reinterpret_cast<GPUMemory::GPUString**>(*keysBuffer + i),
                                        &gpuKeyCol, 1);
            if (pointers)
            {
                pointers->emplace_back(gpuKeyCol);
            }
            break;
        }
        case DataType::COLUMN_INT8_T:
        {
            int8_t* gpuKeyCol;
            GPUMemory::alloc(&gpuKeyCol, rowCount);
            GPUMemory::copyHostToDevice(reinterpret_cast<int8_t**>(*keysBuffer + i), &gpuKeyCol, 1);
            if (pointers)
            {
                pointers->emplace_back(gpuKeyCol);
            }
            break;
        }
        default:
            CheckQueryEngineError(GPU_EXTENSION_ERROR, "Multi-key GROUP BY with keys of type " +
                                                           std::to_string(keyTypes[i]) + " is not supported");
            break;
        }
        nullmask_t* gpuKeyNullMask;
        GPUMemory::alloc(&gpuKeyNullMask, rowCount);
        GPUMemory::copyHostToDevice(*keysNullBuffer + i, &gpuKeyNullMask, 1);
        if (pointersNullMask)
        {
            pointersNullMask->emplace_back(gpuKeyNullMask);
        }
    }
}

void FreeKeysBuffer(void** keysBuffer, nullmask_t** keysNullBuffer, DataType* keyTypes, int32_t keysColCount)
{
    // Copy data types back from GPU
    std::vector<DataType> keyTypesHost;
    keyTypesHost.resize(keysColCount);
    GPUMemory::copyDeviceToHost(keyTypesHost.data(), keyTypes, keysColCount);

    for (int32_t i = 0; i < keysColCount; i++)
    {
        void* ptr;
        GPUMemory::copyDeviceToHost(&ptr, keysBuffer + i, 1); // copy single pointer
        if (ptr)
        {
            if (keyTypesHost[i] == DataType::COLUMN_STRING)
            {
                GPUMemory::GPUString str;
                GPUMemory::copyDeviceToHost(&str, reinterpret_cast<GPUMemory::GPUString*>(ptr), 1);
                GPUMemory::free(str);
            }
            GPUMemory::free(ptr);
        }
        nullmask_t* ptrNullBuffer;
        GPUMemory::copyDeviceToHost(&ptrNullBuffer, keysNullBuffer + i, 1); // copy single pointer
        if (ptrNullBuffer)
        {
            GPUMemory::free(ptrNullBuffer);
        }
    }
    GPUMemory::free(keysBuffer);
    GPUMemory::free(keysNullBuffer);
}

void FreeKeysVector(std::vector<void*> keysVector, std::vector<DataType> keyTypes)
{
    for (int32_t i = 0; i < keyTypes.size(); i++)
    {
        if (keysVector[i])
        {
            if (keyTypes[i] == DataType::COLUMN_STRING)
            {
                GPUMemory::GPUString* str = reinterpret_cast<GPUMemory::GPUString*>(keysVector[i]);
                GPUMemory::free(*str);
                delete str;
            }
            GPUMemory::free(keysVector[i]);
        }
    }
}


__global__ void kernel_collect_string_lengths(int32_t* stringLengths,
                                              int32_t* sourceIndices,
                                              GPUMemory::GPUString** inKeysSingleCol,
                                              GPUMemory::GPUString** keysBufferSingleCol,
                                              int32_t maxHashCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < maxHashCount; i += stride)
    {
        if (sourceIndices[i] >= 0) // string from input key array
        {
            stringLengths[i] = GetStringLength((*inKeysSingleCol)->stringIndices, sourceIndices[i]);
        }
        else if (sourceIndices[i] == GBS_SOURCE_INDEX_KEY_IN_BUFFER) // string stored in key buffer
        {
            stringLengths[i] = GetStringLength((*keysBufferSingleCol)->stringIndices, i);
        }
        else // GBS_SOURCE_INDEX_EMPTY_KEY - no string
        {
            stringLengths[i] = 0;
        }
    }
}


__global__ void kernel_collect_multi_keys(DataType* keyTypes,
                                          int32_t keysColCount,
                                          int32_t* sourceIndices,
                                          void** keysBuffer,
                                          nullmask_t** keysNullBuffer,
                                          GPUMemory::GPUString* stringSideBuffers,
                                          int32_t** stringLengthsBuffers,
                                          int32_t maxHashCount,
                                          void** inKeys,
                                          nullmask_t** inKeysNullMask)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < maxHashCount; i += stride)
    {
        if (sourceIndices[i] >= 0)
        {
            for (int32_t t = 0; t < keysColCount; t++)
            {
                switch (keyTypes[t])
                {
                case DataType::COLUMN_INT:
                    reinterpret_cast<int32_t*>(keysBuffer[t])[i] =
                        reinterpret_cast<int32_t*>(inKeys[t])[sourceIndices[i]];
                    break;
                case DataType::COLUMN_LONG:
                    reinterpret_cast<int64_t*>(keysBuffer[t])[i] =
                        reinterpret_cast<int64_t*>(inKeys[t])[sourceIndices[i]];
                    break;
                case DataType::COLUMN_FLOAT:
                    reinterpret_cast<float*>(keysBuffer[t])[i] =
                        reinterpret_cast<float*>(inKeys[t])[sourceIndices[i]];
                    break;
                case DataType::COLUMN_DOUBLE:
                    reinterpret_cast<double*>(keysBuffer[t])[i] =
                        reinterpret_cast<double*>(inKeys[t])[sourceIndices[i]];
                    break;
                case DataType::COLUMN_STRING:
                {
                    // Copy strings from inKeys according to sourceIndices
                    GPUMemory::GPUString& sideBufferStr = stringSideBuffers[t];
                    GPUMemory::GPUString& inKeysStr = *reinterpret_cast<GPUMemory::GPUString*>(inKeys[t]);
                    for (int32_t j = 0; j < stringLengthsBuffers[t][i]; j++)
                    {
                        sideBufferStr.allChars[GetStringIndex(sideBufferStr.stringIndices, i) + j] =
                            inKeysStr.allChars[GetStringIndex(inKeysStr.stringIndices, sourceIndices[i]) + j];
                    }
                    break;
                }
                case DataType::COLUMN_INT8_T:
                    reinterpret_cast<int8_t*>(keysBuffer[t])[i] =
                        reinterpret_cast<int8_t*>(inKeys[t])[sourceIndices[i]];
                    break;
                default:
                    break;
                }
                // If using keys null mask
                if (inKeysNullMask[t] != nullptr)
                {
                    keysNullBuffer[t][i] =
                        NullValues::GetConcreteBitFromBitmask(inKeysNullMask[t], sourceIndices[i]);
                }
                else // If not, set added key as not null
                {
                    keysNullBuffer[t][i] = false;
                }
            }
            sourceIndices[i] = GBS_SOURCE_INDEX_KEY_IN_BUFFER; // Mark as stored in keyBuffer
        }
        else if (sourceIndices[i] == GBS_SOURCE_INDEX_KEY_IN_BUFFER)
        {
            for (int32_t t = 0; t < keysColCount; t++)
            {
                if (keyTypes[t] == DataType::COLUMN_STRING)
                {
                    // Copy strings from keysBuffer
                    GPUMemory::GPUString& sideBufferStr = stringSideBuffers[t];
                    GPUMemory::GPUString& keysBufferStr =
                        *reinterpret_cast<GPUMemory::GPUString*>(keysBuffer[t]);
                    for (int32_t j = 0; j < stringLengthsBuffers[t][i]; j++)
                    {
                        sideBufferStr.allChars[GetStringIndex(sideBufferStr.stringIndices, i) + j] =
                            keysBufferStr.allChars[GetStringIndex(keysBufferStr.stringIndices, i) + j];
                    }
                }
            }
        }
    }
}
