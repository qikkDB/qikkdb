#include "hip/hip_runtime.h"
#include "GPUGroupByMultiKey.cuh"

__device__ int32_t GetHash(DataType* keyTypes, int32_t keysColCount, void** inKeys, int32_t i, const int32_t hashCoef)
{
    uint32_t crc = 0xFFFFFFFF;

    for (int32_t t = 0; t < keysColCount; t++)
    {
        uint32_t hash = 0;
        switch (keyTypes[t])
        {
        case DataType::COLUMN_INT:
            hash = reinterpret_cast<uint32_t*>(inKeys[t])[i];
            break;
        case DataType::COLUMN_LONG:
            hash = static_cast<uint32_t>(reinterpret_cast<int64_t*>(inKeys[t])[i]);
            break;
        case DataType::COLUMN_FLOAT:
            hash = static_cast<uint32_t>(reinterpret_cast<float*>(inKeys[t])[i]);
            break;
        case DataType::COLUMN_DOUBLE:
            hash = static_cast<uint32_t>(reinterpret_cast<double*>(inKeys[t])[i]);
            break;
        case DataType::COLUMN_STRING:
        {
            GPUMemory::GPUString strCol = *reinterpret_cast<GPUMemory::GPUString*>(inKeys[t]);
            hash = GetHash(strCol.allChars + GetStringIndex(strCol.stringIndices, i),
                           GetStringLength(strCol.stringIndices, i));
        }
        break;
        case DataType::COLUMN_INT8_T:
            hash = static_cast<uint32_t>(reinterpret_cast<int8_t*>(inKeys[t])[i]);
            break;
        default:
            hash = 0;
            break;
        }
        crc = (CRC_32_TAB[((crc >> 24) ^ hash) & 0xFF] ^ (crc << 8));
    }
    return (crc >> 16) ^ (crc & 0xFFFF);
}


__device__ bool
AreEqualMultiKeys(DataType* keyTypes, int32_t keysColCount, void** keysA, int32_t indexA, void** keysB, int32_t indexB)
{
    for (int32_t t = 0; t < keysColCount; t++)
    {
        switch (keyTypes[t])
        {
        case DataType::COLUMN_INT:
            if (reinterpret_cast<int32_t*>(keysA[t])[indexA] != reinterpret_cast<int32_t*>(keysB[t])[indexB])
            {
                return false;
            }
            break;
        case DataType::COLUMN_LONG:
            if (reinterpret_cast<int64_t*>(keysA[t])[indexA] != reinterpret_cast<int64_t*>(keysB[t])[indexB])
            {
                return false;
            }
            break;
        case DataType::COLUMN_FLOAT:
            if (reinterpret_cast<float*>(keysA[t])[indexA] != reinterpret_cast<float*>(keysB[t])[indexB])
            {
                return false;
            }
            break;
        case DataType::COLUMN_DOUBLE:
            if (reinterpret_cast<double*>(keysA[t])[indexA] != reinterpret_cast<double*>(keysB[t])[indexB])
            {
                return false;
            }
            break;
        case DataType::COLUMN_STRING:
        {
            GPUMemory::GPUString strColA = *reinterpret_cast<GPUMemory::GPUString*>(keysA[t]);
            GPUMemory::GPUString strColB = *reinterpret_cast<GPUMemory::GPUString*>(keysB[t]);
            if (!AreEqualStrings(strColA.allChars + GetStringIndex(strColA.stringIndices, indexA),
                                 GetStringLength(strColA.stringIndices, indexA), strColB, indexB))
            {
                return false;
            }
            break;
        }
        case DataType::COLUMN_INT8_T:
            if (reinterpret_cast<int8_t*>(keysA[t])[indexA] != reinterpret_cast<int8_t*>(keysB[t])[indexB])
            {
                return false;
            }
            break;
        default:
            break;
        }
    }
    return true;
}


__device__ bool
IsNewMultiKey(DataType* keyTypes, int32_t keysColCount, void** inKeys, int32_t i, void** keysBuffer, int32_t* sourceIndices, int32_t index)
{
    return (sourceIndices[index] >= 0 &&
            !AreEqualMultiKeys(keyTypes, keysColCount, inKeys, i, inKeys, sourceIndices[index])) ||
           (sourceIndices[index] == GBS_SOURCE_INDEX_KEY_IN_BUFFER &&
            !AreEqualMultiKeys(keyTypes, keysColCount, inKeys, i, keysBuffer, index));
}


template <>
void ReconstructSingleKeyColKeep<std::string>(std::vector<void*>* outKeysVector,
                                              int32_t* outDataElementCount,
                                              int8_t* occupancyMaskPtr,
                                              void** keyCol,
                                              int32_t elementCount)
{
    // Copy struct (we need to get pointer to struct at first)
    GPUMemory::GPUString* structPointer;
    GPUMemory::copyDeviceToHost(&structPointer, reinterpret_cast<GPUMemory::GPUString**>(keyCol), 1);
    GPUMemory::GPUString keyBufferSingleCol;
    GPUMemory::copyDeviceToHost(&keyBufferSingleCol, structPointer, 1);

    // Reconstruct string keys
    GPUMemory::GPUString* outKeysSingleCol = new GPUMemory::GPUString();
    GPUReconstruct::ReconstructStringColKeep(outKeysSingleCol, outDataElementCount,
                                             keyBufferSingleCol, occupancyMaskPtr, elementCount);

    outKeysVector->emplace_back(outKeysSingleCol);
}


template <>
void ReconstructSingleKeyCol<std::string>(std::vector<void*>* outKeysVector,
                                          int32_t* outDataElementCount,
                                          int8_t* occupancyMaskPtr,
                                          void** keyCol,
                                          int32_t elementCount)
{
    // Copy struct (we need to get pointer to struct at first)
    GPUMemory::GPUString* structPointer;
    GPUMemory::copyDeviceToHost(&structPointer, reinterpret_cast<GPUMemory::GPUString**>(keyCol), 1);
    GPUMemory::GPUString keyBufferSingleCol;
    GPUMemory::copyDeviceToHost(&keyBufferSingleCol, structPointer, 1);

    // Reconstruct string as raw
    std::vector<int32_t> stringLengths;
    std::vector<char> allChars;
    GPUReconstruct::ReconstructStringColRaw(stringLengths, allChars, outDataElementCount,
                                            keyBufferSingleCol, occupancyMaskPtr, elementCount);

    CPUString* outKeysSingleCol = new CPUString{stringLengths, allChars};
    outKeysVector->emplace_back(outKeysSingleCol);
}


void AllocKeysBuffer(void*** keysBuffer, std::vector<DataType> keyTypes, int32_t rowCount, std::vector<void*>* pointers)
{
    GPUMemory::alloc(keysBuffer, keyTypes.size());
    for (int32_t i = 0; i < keyTypes.size(); i++)
    {
        switch (keyTypes[i])
        {
        case DataType::COLUMN_INT:
        {
            int32_t* gpuKeyCol;
            GPUMemory::alloc(&gpuKeyCol, rowCount);
            GPUMemory::copyHostToDevice(reinterpret_cast<int32_t**>(*keysBuffer + i), &gpuKeyCol, 1);
            if (pointers)
            {
                pointers->emplace_back(gpuKeyCol);
            }
            break;
        }
        case DataType::COLUMN_LONG:
        {
            int64_t* gpuKeyCol;
            GPUMemory::alloc(&gpuKeyCol, rowCount);
            GPUMemory::copyHostToDevice(reinterpret_cast<int64_t**>(*keysBuffer + i), &gpuKeyCol, 1);
            if (pointers)
            {
                pointers->emplace_back(gpuKeyCol);
            }
            break;
        }
        case DataType::COLUMN_FLOAT:
        {
            float* gpuKeyCol;
            GPUMemory::alloc(&gpuKeyCol, rowCount);
            GPUMemory::copyHostToDevice(reinterpret_cast<float**>(*keysBuffer + i), &gpuKeyCol, 1);
            if (pointers)
            {
                pointers->emplace_back(gpuKeyCol);
            }
            break;
        }
        case DataType::COLUMN_DOUBLE:
        {
            double* gpuKeyCol;
            GPUMemory::alloc(&gpuKeyCol, rowCount);
            GPUMemory::copyHostToDevice(reinterpret_cast<double**>(*keysBuffer + i), &gpuKeyCol, 1);
            if (pointers)
            {
                pointers->emplace_back(gpuKeyCol);
            }
            break;
        }
        case DataType::COLUMN_STRING:
        {
            GPUMemory::GPUString emptyStringCol{nullptr, nullptr};
            GPUMemory::GPUString* gpuKeyCol;
            GPUMemory::alloc(&gpuKeyCol, 1);
            GPUMemory::copyHostToDevice(gpuKeyCol, &emptyStringCol, 1);
            GPUMemory::copyHostToDevice(reinterpret_cast<GPUMemory::GPUString**>(*keysBuffer + i),
                                        &gpuKeyCol, 1);
            if (pointers)
            {
                pointers->emplace_back(gpuKeyCol);
            }
            break;
        }
        case DataType::COLUMN_INT8_T:
        {
            int8_t* gpuKeyCol;
            GPUMemory::alloc(&gpuKeyCol, rowCount);
            GPUMemory::copyHostToDevice(reinterpret_cast<int8_t**>(*keysBuffer + i), &gpuKeyCol, 1);
            if (pointers)
            {
                pointers->emplace_back(gpuKeyCol);
            }
            break;
        }
        default:
            CheckQueryEngineError(GPU_EXTENSION_ERROR, "Multi-key GROUP BY with keys of type " +
                                                           std::to_string(keyTypes[i]) + " is not supported");
            break;
        }
    }
}

void FreeKeysBuffer(void** keysBuffer, DataType* keyTypes, int32_t keysColCount)
{
    // Copy data types back from GPU
    std::vector<DataType> keyTypesHost;
    keyTypesHost.resize(keysColCount);
    GPUMemory::copyDeviceToHost(keyTypesHost.data(), keyTypes, keysColCount);

    for (int32_t i = 0; i < keysColCount; i++)
    {
        void* ptr;
        GPUMemory::copyDeviceToHost(&ptr, keysBuffer + i, 1); // copy single pointer
        if (ptr)
        {
            if (keyTypesHost[i] == DataType::COLUMN_STRING)
            {
                GPUMemory::GPUString str;
                GPUMemory::copyDeviceToHost(&str, reinterpret_cast<GPUMemory::GPUString*>(ptr), 1);
                GPUMemory::free(str);
            }
            GPUMemory::free(ptr);
        }
    }
    GPUMemory::free(keysBuffer);
}

void FreeKeysVector(std::vector<void*> keysVector, std::vector<DataType> keyTypes)
{
    for (int32_t i = 0; i < keyTypes.size(); i++)
    {
        if (keysVector[i])
        {
            if (keyTypes[i] == DataType::COLUMN_STRING)
            {
                GPUMemory::GPUString* str = reinterpret_cast<GPUMemory::GPUString*>(keysVector[i]);
                GPUMemory::free(*str);
                delete str;
            }
            GPUMemory::free(keysVector[i]);
        }
    }
}


__global__ void kernel_collect_string_lengths(int32_t* stringLengths,
                                              int32_t* sourceIndices,
                                              GPUMemory::GPUString** inKeysSingleCol,
                                              GPUMemory::GPUString** keysBufferSingleCol,
                                              int32_t maxHashCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < maxHashCount; i += stride)
    {
        if (sourceIndices[i] >= 0) // string from input key array
        {
            stringLengths[i] = GetStringLength((*inKeysSingleCol)->stringIndices, sourceIndices[i]);
        }
        else if (sourceIndices[i] == GBS_SOURCE_INDEX_KEY_IN_BUFFER) // string stored in key buffer
        {
            stringLengths[i] = GetStringLength((*keysBufferSingleCol)->stringIndices, i);
        }
        else // GBS_SOURCE_INDEX_EMPTY_KEY - no string
        {
            stringLengths[i] = 0;
        }
    }
}


__global__ void kernel_collect_multi_keys(DataType* keyTypes,
                                          int32_t keysColCount,
                                          int32_t* sourceIndices,
                                          void** keysBuffer,
                                          GPUMemory::GPUString* stringSideBuffers,
                                          int32_t** stringLengthsBuffers,
                                          int32_t maxHashCount,
                                          void** inKeys)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < maxHashCount; i += stride)
    {
        if (sourceIndices[i] >= 0)
        {
            for (int32_t t = 0; t < keysColCount; t++)
            {
                switch (keyTypes[t])
                {
                case DataType::COLUMN_INT:
                    reinterpret_cast<int32_t*>(keysBuffer[t])[i] =
                        reinterpret_cast<int32_t*>(inKeys[t])[sourceIndices[i]];
                    break;
                case DataType::COLUMN_LONG:
                    reinterpret_cast<int64_t*>(keysBuffer[t])[i] =
                        reinterpret_cast<int64_t*>(inKeys[t])[sourceIndices[i]];
                    break;
                case DataType::COLUMN_FLOAT:
                    reinterpret_cast<float*>(keysBuffer[t])[i] =
                        reinterpret_cast<float*>(inKeys[t])[sourceIndices[i]];
                    break;
                case DataType::COLUMN_DOUBLE:
                    reinterpret_cast<double*>(keysBuffer[t])[i] =
                        reinterpret_cast<double*>(inKeys[t])[sourceIndices[i]];
                    break;
                case DataType::COLUMN_STRING:
                {
                    // Copy strings from inKeys according to sourceIndices
                    GPUMemory::GPUString& sideBufferStr = stringSideBuffers[t];
                    GPUMemory::GPUString& inKeysStr = *reinterpret_cast<GPUMemory::GPUString*>(inKeys[t]);
                    for (int32_t j = 0; j < stringLengthsBuffers[t][i]; j++)
                    {
                        sideBufferStr.allChars[GetStringIndex(sideBufferStr.stringIndices, i) + j] =
                            inKeysStr.allChars[GetStringIndex(inKeysStr.stringIndices, sourceIndices[i]) + j];
                    }
                    break;
                }
                case DataType::COLUMN_INT8_T:
                    reinterpret_cast<int8_t*>(keysBuffer[t])[i] =
                        reinterpret_cast<int8_t*>(inKeys[t])[sourceIndices[i]];
                    break;
                default:
                    break;
                }
            }
            sourceIndices[i] = GBS_SOURCE_INDEX_KEY_IN_BUFFER; // Mark as stored in keyBuffer
        }
        else if (sourceIndices[i] == GBS_SOURCE_INDEX_KEY_IN_BUFFER)
        {
            for (int32_t t = 0; t < keysColCount; t++)
            {
                if (keyTypes[t] == DataType::COLUMN_STRING)
                {
                    // Copy strings from keysBuffer
                    GPUMemory::GPUString& sideBufferStr = stringSideBuffers[t];
                    GPUMemory::GPUString& keysBufferStr =
                        *reinterpret_cast<GPUMemory::GPUString*>(keysBuffer[t]);
                    for (int32_t j = 0; j < stringLengthsBuffers[t][i]; j++)
                    {
                        sideBufferStr.allChars[GetStringIndex(sideBufferStr.stringIndices, i) + j] =
                            keysBufferStr.allChars[GetStringIndex(keysBufferStr.stringIndices, i) + j];
                    }
                }
            }
        }
    }
}
