#include "hip/hip_runtime.h"
#include "GPUPolygonClipping.cuh"

__device__ LLPolyVertex calc_intersect(NativeGeoPoint sA, NativeGeoPoint eA, NativeGeoPoint sB, NativeGeoPoint eB)
{
    float adx = eA.latitude - sA.latitude;
    float ady = eA.longitude - sA.longitude;
    float bdx = eB.latitude - sB.latitude;
    float bdy = eB.longitude - sB.longitude;

    float axb = adx * bdy - ady * bdx;

    if (axb == 0)
    {
        LLPolyVertex retFail = {{0, 0}, 0x0, -1.0, -1.0, -1, -1, -1};

        SetIsIntersection(retFail, true);
        SetIsValidIntersection(retFail, false);
        SetIsEntry(retFail, false);
        SetWasProcessed(retFail, false);

        return retFail;
    }

    float dx = sA.latitude - sB.latitude;
    float dy = sA.longitude - sB.longitude;

    float alongA = (bdx * dy - bdy * dx) / axb;
    float alongB = (adx * dy - ady * dx) / axb;

    bool intersectionValidity = (alongA > 0 && alongA < 1 && alongB > 0 && alongB < 1);

    LLPolyVertex ret = {{sA.latitude + alongA * adx, sA.longitude + alongA * ady},
                        0x0,
                        alongA,
                        alongB,
                        -1,
                        -1,
                        -1};

	SetHasIntersections(ret, intersectionValidity);
    SetIsIntersection(ret, true);
    SetIsValidIntersection(ret, intersectionValidity);
    SetIsEntry(ret, false);
    SetWasProcessed(ret, false);

    return ret;
}

__global__ void kernel_calc_ll_buffers_size(int32_t* llPolygonABufferSizes,
                                            int32_t* llPolygonBBufferSizes,
                                            int8_t* PolygonAIntersectionPresenceFlags,
                                            int8_t* PolygonBIntersectionPresenceFlags,
                                            GPUMemory::GPUPolygon polygonA,
                                            GPUMemory::GPUPolygon polygonB,
                                            bool isAConst,
                                            bool isBConst,
                                            int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        int32_t iAIdx = isAConst ? 0 : i;
        int32_t iBIdx = isBConst ? 0 : i;

        const int32_t polyIdxA = GPUMemory::PolyIdxAt(polygonA, iAIdx);
        const int32_t polyCountA = GPUMemory::PolyCountAt(polygonA, iAIdx);

        const int32_t polyIdxB = GPUMemory::PolyIdxAt(polygonB, iBIdx);
        const int32_t polyCountB = GPUMemory::PolyCountAt(polygonB, iBIdx);

        int32_t intersectCount = 0;
        for (int32_t a = polyIdxA; a < (polyIdxA + polyCountA); a++)
        {
            const int32_t pointIdxA = GPUMemory::PointIdxAt(polygonA, a);
            const int32_t pointCountA = GPUMemory::PointCountAt(polygonA, a);

            int8_t intersectionPresentInSubPolygonA = 0;

            for (int32_t b = polyIdxB; b < (polyIdxB + polyCountB); b++)
            {
                const int32_t pointIdxB = GPUMemory::PointIdxAt(polygonB, b);
                const int32_t pointCountB = GPUMemory::PointCountAt(polygonB, b);

                int8_t intersectionPresentInSubPolygonB = 0;

                // Calculate total intersections count
                for (int32_t pointA = pointIdxA; pointA < (pointIdxA + pointCountA); pointA++)
                {
                    for (int32_t pointB = pointIdxB; pointB < (pointIdxB + pointCountB); pointB++)
                    {
                        LLPolyVertex intersection =
                            calc_intersect(polygonA.polyPoints[pointA],
                                           polygonA.polyPoints[pointIdxA + (pointA - pointIdxA + 1) % pointCountA],
                                           polygonB.polyPoints[pointB],
                                           polygonB.polyPoints[pointIdxB + (pointB - pointIdxB + 1) % pointCountB]);

                        if (GetIsValidIntersection(intersection))
                        {
                            intersectionPresentInSubPolygonA = 1;
                            intersectionPresentInSubPolygonB = 1;

                            intersectCount++;
                        }
                    }
                }
                PolygonBIntersectionPresenceFlags[isBConst ? b + i * dataElementCount : b] |=
                    intersectionPresentInSubPolygonB;
            }
            PolygonAIntersectionPresenceFlags[isAConst ? a + i * dataElementCount : a] |=
                intersectionPresentInSubPolygonA;
        }

        // Get the complex polygon vertex counts n and k
        const int32_t n = GPUMemory::TotalPointCountAt(polygonA, iAIdx);
        const int32_t k = GPUMemory::TotalPointCountAt(polygonB, iBIdx);

        // Assign the calculated buffers size
        llPolygonABufferSizes[i] = n + intersectCount;
        llPolygonBBufferSizes[i] = k + intersectCount;
    }
}

__global__ void kernel_build_ll(LLPolyVertex* llPolygonBuffers,
                                GPUMemory::GPUPolygon polygon,
                                int32_t* llPolygonBufferSizesPrefixSum,
                                int8_t* PolygonIntersectionPresenceFlags,
                                bool isConst,
                                int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        int32_t iIdx = isConst ? 0 : i;

        int32_t llPolygonEndIdx = 0;

        const int32_t polyIdx = GPUMemory::PolyIdxAt(polygon, iIdx);
        const int32_t polyCount = GPUMemory::PolyCountAt(polygon, iIdx);

        // Transform polygon
        for (int32_t p = polyIdx; p < (polyIdx + polyCount); p++)
        {
            const int32_t pointIdx = GPUMemory::PointIdxAt(polygon, p);
            const int32_t pointCount = GPUMemory::PointCountAt(polygon, p);

            for (int32_t point = pointIdx; point < (pointIdx + pointCount); point++)
            {
                const int32_t localIdx = pointIdx - GPUMemory::PointIdxAt(polygon, polyIdx);

                // Set the linked list entry
                LLPolyVertex vertex = {
                    polygon.polyPoints[point],
                    0x0,
                    -1.0,
                    -1.0,
                    ((i == 0) ? 0 : llPolygonBufferSizesPrefixSum[i - 1]) + localIdx +
                        (point - pointIdx - 1 + pointCount) % pointCount,
                    ((i == 0) ? 0 : llPolygonBufferSizesPrefixSum[i - 1]) + localIdx +
                        (point - pointIdx + 1) % pointCount,
                    -1};

				SetHasIntersections(vertex, PolygonIntersectionPresenceFlags[p]);
                SetIsIntersection(vertex, false);
                SetIsValidIntersection(vertex, false);
                SetIsEntry(vertex, false);
                SetWasProcessed(vertex, false);

                llPolygonBuffers[((i == 0) ? 0 : llPolygonBufferSizesPrefixSum[i - 1]) + llPolygonEndIdx] = vertex;

                // Increment the local pointer to the end of the ll
                llPolygonEndIdx++;
            }
        }
    }
}

__global__ void kernel_add_and_crosslink_intersections_to_ll(LLPolyVertex* llPolygonABuffers,
                                                             LLPolyVertex* llPolygonBBuffers,
                                                             GPUMemory::GPUPolygon polygonA,
                                                             GPUMemory::GPUPolygon polygonB,
                                                             int32_t* llPolygonABufferSizesPrefixSum,
                                                             int32_t* llPolygonBBufferSizesPrefixSum,
                                                             bool isAConst,
                                                             bool isBConst,
                                                             int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        int32_t iAIdx = isAConst ? 0 : i;
        int32_t iBIdx = isBConst ? 0 : i;

        // "Pointers" to the element after the last valid element of the linked lists
        // They begin after the last non intersection e.g. poly vertex address
        int32_t llPolygonAEndIdx = GPUMemory::TotalPointCountAt(polygonA, iAIdx);
        int32_t llPolygonBEndIdx = GPUMemory::TotalPointCountAt(polygonB, iBIdx);

        const int32_t polyIdxA = GPUMemory::PolyIdxAt(polygonA, iAIdx);
        const int32_t polyCountA = GPUMemory::PolyCountAt(polygonA, iAIdx);

        const int32_t polyIdxB = GPUMemory::PolyIdxAt(polygonB, iBIdx);
        const int32_t polyCountB = GPUMemory::PolyCountAt(polygonB, iBIdx);

        for (int32_t a = polyIdxA; a < (polyIdxA + polyCountA); a++)
        {
            const int32_t pointIdxA = GPUMemory::PointIdxAt(polygonA, a);
            const int32_t pointCountA = GPUMemory::PointCountAt(polygonA, a);

            for (int32_t b = polyIdxB; b < (polyIdxB + polyCountB); b++)
            {
                const int32_t pointIdxB = GPUMemory::PointIdxAt(polygonB, b);
                const int32_t pointCountB = GPUMemory::PointCountAt(polygonB, b);

                // Calculate intersections and insert them into the ll
                for (int32_t pointA = pointIdxA; pointA < (pointIdxA + pointCountA); pointA++)
                {
                    for (int32_t pointB = pointIdxB; pointB < (pointIdxB + pointCountB); pointB++)
                    {
                        LLPolyVertex intersection =
                            calc_intersect(polygonA.polyPoints[pointA],
                                           polygonA.polyPoints[pointIdxA + (pointA - pointIdxA + 1) % pointCountA],
                                           polygonB.polyPoints[pointB],
                                           polygonB.polyPoints[pointIdxB + (pointB - pointIdxB + 1) % pointCountB]);

                        // If an intersection is valid, insert it into the linked lists and create a cross reference
                        if (GetIsValidIntersection(intersection))
                        {
                            int32_t llPolygonAEndIdxLocal =
                                ((i == 0) ? 0 : llPolygonABufferSizesPrefixSum[i - 1]) + llPolygonAEndIdx;
                            int32_t llPolygonBEndIdxLocal =
                                ((i == 0) ? 0 : llPolygonBBufferSizesPrefixSum[i - 1]) + llPolygonBEndIdx;

                            // Save the intersection data
                            llPolygonABuffers[llPolygonAEndIdxLocal] = intersection;
                            llPolygonBBuffers[llPolygonBEndIdxLocal] = intersection;

                            // Write the cross reference indices
                            llPolygonABuffers[llPolygonAEndIdxLocal].crossIdx = llPolygonBEndIdxLocal;
                            llPolygonBBuffers[llPolygonBEndIdxLocal].crossIdx = llPolygonAEndIdxLocal;

                            // "Rewire" the prev and next pointers in both linked lists
                            // so that the point is in it's correct place
                            // according to the parametric distance from the beginning of the line segment
                            //////////////////////////////////////////////////////////////////////////////
                            // First polygon - A
                            const int32_t localIdxA = pointIdxA - GPUMemory::PointIdxAt(polygonA, polyIdxA);

                            const int32_t begIdxA = ((i == 0) ? 0 : llPolygonABufferSizesPrefixSum[i - 1]) +
                                              localIdxA + (pointA - pointIdxA) % pointCountA;
                            const int32_t endIdxA = ((i == 0) ? 0 : llPolygonABufferSizesPrefixSum[i - 1]) +
                                              localIdxA + (pointA - pointIdxA + 1) % pointCountA;

                            int32_t nextIdxA = llPolygonABuffers[begIdxA].nextIdx;
                            while (nextIdxA != endIdxA && llPolygonABuffers[llPolygonAEndIdxLocal].distanceAlongA >
                                                              llPolygonABuffers[nextIdxA].distanceAlongA)
                            {
                                nextIdxA = llPolygonABuffers[nextIdxA].nextIdx;
                            }

                            // Rewire the pointers for the first polygon - A
                            llPolygonABuffers[llPolygonAEndIdxLocal].prevIdx =
                                llPolygonABuffers[nextIdxA].prevIdx;
                            llPolygonABuffers[llPolygonAEndIdxLocal].nextIdx = nextIdxA;

                            llPolygonABuffers[llPolygonABuffers[nextIdxA].prevIdx].nextIdx = llPolygonAEndIdxLocal;
                            llPolygonABuffers[nextIdxA].prevIdx = llPolygonAEndIdxLocal;
                            //////////////////////////////////////////////////////////////////////////////
                            // Second polygon - B
                            const int32_t localIdxB = pointIdxB - GPUMemory::PointIdxAt(polygonB, polyIdxB);

                            const int32_t begIdxB = ((i == 0) ? 0 : llPolygonBBufferSizesPrefixSum[i - 1]) +
                                              localIdxB + (pointB - pointIdxB) % pointCountB;
                            const int32_t endIdxB = ((i == 0) ? 0 : llPolygonBBufferSizesPrefixSum[i - 1]) +
                                              localIdxB + (pointB - pointIdxB + 1) % pointCountB;

                            int32_t nextIdxB = llPolygonBBuffers[begIdxB].nextIdx;
                            while (nextIdxB != endIdxB && llPolygonBBuffers[llPolygonBEndIdxLocal].distanceAlongB >
                                                              llPolygonBBuffers[nextIdxB].distanceAlongB)
                            {
                                nextIdxB = llPolygonBBuffers[nextIdxB].nextIdx;
                            }

                            // Rewire the pointers for the second polygon - B
                            llPolygonBBuffers[llPolygonBEndIdxLocal].prevIdx =
                                llPolygonBBuffers[nextIdxB].prevIdx;
                            llPolygonBBuffers[llPolygonBEndIdxLocal].nextIdx = nextIdxB;

                            llPolygonBBuffers[llPolygonBBuffers[nextIdxB].prevIdx].nextIdx = llPolygonBEndIdxLocal;
                            llPolygonBBuffers[nextIdxB].prevIdx = llPolygonBEndIdxLocal;
                            //////////////////////////////////////////////////////////////////////////////

                            // Increment the ll end pointers
                            llPolygonAEndIdx++;
                            llPolygonBEndIdx++;
                        }
                    }
                }
            }
        }
    }
}

__device__ bool is_point_in_complex_polygon_at(NativeGeoPoint geoPoint, GPUMemory::GPUPolygon polygon, int32_t idx)
{
    bool isPointInPolygon = false;

    const int32_t polyIdx = GPUMemory::PolyIdxAt(polygon, idx);
    const int32_t polyCount = GPUMemory::PolyCountAt(polygon, idx);

    for (int32_t p = polyIdx; p < (polyIdx + polyCount); p++)
    {
        const int32_t pointIdx = GPUMemory::PointIdxAt(polygon, p);
        const int32_t pointCount = GPUMemory::PointCountAt(polygon, p);

        // Dank raycasting magic as seen in GPUPolygonContains
        for (int32_t point = pointIdx; point < (pointIdx + pointCount); point++)
        {
            const int32_t pBeg = point;
            const int32_t pEnd = pointIdx + (point - pointIdx + 1) % pointCount;

            if (((polygon.polyPoints[pBeg].longitude > geoPoint.longitude) !=
                 (polygon.polyPoints[pEnd].longitude > geoPoint.longitude)) &&
                (geoPoint.latitude <
                 (polygon.polyPoints[pEnd].latitude - polygon.polyPoints[pBeg].latitude) *
                         (geoPoint.longitude - polygon.polyPoints[pBeg].longitude) /
                         (polygon.polyPoints[pEnd].longitude - polygon.polyPoints[pBeg].longitude) +
                     polygon.polyPoints[pBeg].latitude))
            {
                isPointInPolygon = !isPointInPolygon;
            }
        }
    }

    return isPointInPolygon;
}

__global__ void kernel_label_intersections(LLPolyVertex* llPolygonBuffers,
                                           GPUMemory::GPUPolygon polygonPrimary,
                                           GPUMemory::GPUPolygon polygonSecondary,
                                           int32_t* llPolygonBufferSizesPrefixSum,
                                           bool isPrimaryConst,
                                           bool isSecondaryConst,
                                           int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        const int32_t iPrimary = isPrimaryConst ? 0 : i;
        const int32_t iSecondary = isSecondaryConst ? 0 : i;

        const int32_t polyIdx = GPUMemory::PolyIdxAt(polygonPrimary, iPrimary);
        const int32_t polyCount = GPUMemory::PolyCountAt(polygonPrimary, iPrimary);

        for (int32_t p = polyIdx; p < (polyIdx + polyCount); p++)
        {
            const int32_t pointIdx = GPUMemory::PointIdxAt(polygonPrimary, p);
            const int32_t pointCount = GPUMemory::PointCountAt(polygonPrimary, p);

            // Iterate trough the linked list for the current sub polygon and label the intersections
            const int32_t localIdx = pointIdx - GPUMemory::PointIdxAt(polygonPrimary, polyIdx);

            const int32_t begIdx = ((i == 0) ? 0 : llPolygonBufferSizesPrefixSum[i - 1]) + localIdx;
            const int32_t endIdx =
                ((i == 0) ? 0 : llPolygonBufferSizesPrefixSum[i - 1]) + localIdx + pointCount - 1;

            // Check the inclusion of the first point in the other polygon
            bool isPointInPolygon =
                !is_point_in_complex_polygon_at(llPolygonBuffers[begIdx].vertex, polygonSecondary, iSecondary);

            int32_t nextIdx = begIdx;
            do
            {
                // If the given vertex is an intersection - assign the correct entry/exit label
                if (GetIsIntersection(llPolygonBuffers[nextIdx]))
                {
                    SetIsEntry(llPolygonBuffers[nextIdx], isPointInPolygon);
                    isPointInPolygon = !isPointInPolygon;
                }

                nextIdx = llPolygonBuffers[nextIdx].nextIdx;
            } while (nextIdx != begIdx);
        }
    }
}