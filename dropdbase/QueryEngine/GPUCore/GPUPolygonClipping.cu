#include "hip/hip_runtime.h"
#include "GPUPolygonClipping.cuh"

__device__ LLPolyVertex calc_intersect(NativeGeoPoint sA, NativeGeoPoint eA,
                                       NativeGeoPoint sB, NativeGeoPoint eB)
{
    float adx = eA.latitude - sA.latitude;
    float ady = eA.longitude - sA.longitude;
    float bdx = eB.latitude - sB.latitude;
    float bdy = eB.longitude - sB.longitude;

    float axb = adx * bdy - ady * bdx;

    if (axb == 0)
    {
        LLPolyVertex retFail = {{0, 0}, true, false, -1.0, -1.0, -1, -1, -1};
        return retFail;
    }

    float dx = sA.latitude - sB.latitude;
    float dy = sA.longitude - sB.longitude;

    float alongA = (bdx * dy - bdy * dx) / axb;
    float alongB = (adx * dy - ady * dx) / axb;

    bool intersectionValidity = (alongA > 0 && alongA < 1 && alongB > 0 && alongB < 1);

    LLPolyVertex ret = {
        {sA.latitude + alongA * adx, sA.longitude + alongA * ady},
        true,
        intersectionValidity,
        alongA,
        alongB,
        -1,
        -1,
        -1
    };

    return ret;
}

__global__ void kernel_calc_LL_buffers_size(int32_t *LLPolygonABufferSizes, 
                                            int32_t *LLPolygonBBufferSizes,
                                            GPUMemory::GPUPolygon polygonA,
                                            GPUMemory::GPUPolygon polygonB,
                                            int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        int32_t polyIdxA = GPUMemory::PolyIdxAt(polygonA, i);
        int32_t polyCountA = GPUMemory::PolyCountAt(polygonA, i);

        int32_t polyIdxB = GPUMemory::PolyIdxAt(polygonB, i);
        int32_t polyCountB = GPUMemory::PolyCountAt(polygonB, i);

        // The number of LL elements is the number of vertices of complex polygon A = n, 
        // complex polygon B = k and the number of intersections between them
        int32_t n = 0;
        int32_t k = 0;
        int32_t intersectCount = 0;
        for(int32_t a = polyIdxA; a < (polyIdxA + polyCountA); a++)
        {
            int32_t pointIdxA = GPUMemory::PointIdxAt(polygonA, a);
            int32_t pointCountA = GPUMemory::PointCountAt(polygonA, a);

            for(int32_t b = polyIdxB; b < (polyIdxB + polyCountB); b++)
            {
                int32_t pointIdxB = GPUMemory::PointIdxAt(polygonB, b);
                int32_t pointCountB = GPUMemory::PointCountAt(polygonB, b);

                // Calculate intersections count
                for(int32_t pointA = pointIdxA; pointA < (pointIdxA + pointCountA); pointA++)
                {
                    for(int32_t pointB = pointIdxB; pointB < (pointIdxB + pointCountB); pointB++)
                    {
                        LLPolyVertex result = calc_intersect(polygonA.polyPoints[pointA], 
                                                             polygonA.polyPoints[pointIdxA + (pointA + 1) % pointCountA],
                                                             polygonB.polyPoints[pointB],
                                                             polygonB.polyPoints[pointIdxB + (pointB + 1) % pointCountB]);
                        if(result.isValidIntersection)
                        {
                            intersectCount++;
                        }
                    }       
                }
            }   
        }

        // Get the complex polygon vertex counts n and k
        if(i == 0)
        {
            n = GPUMemory::PointIdxAt(polygonA, polyIdxA + polyCountA);
            k = GPUMemory::PointIdxAt(polygonB, polyIdxB + polyCountB);
        }
        else
        {
            int32_t polyIdxAPrev = GPUMemory::PolyIdxAt(polygonA, i - 1);
            int32_t polyCountAPrev = GPUMemory::PolyCountAt(polygonA, i - 1);
    
            int32_t polyIdxBPrev = GPUMemory::PolyIdxAt(polygonB, i - 1);
            int32_t polyCountBPrev = GPUMemory::PolyCountAt(polygonB, i - 1);

            n = GPUMemory::PointIdxAt(polygonA, polyIdxA + polyCountA) - GPUMemory::PointIdxAt(polygonA, polyIdxAPrev + polyCountAPrev);
            k = GPUMemory::PointIdxAt(polygonB, polyIdxB + polyCountB) - GPUMemory::PointIdxAt(polygonB, polyIdxBPrev + polyCountBPrev);
        }

        // Assign the calculated buffers size
        LLPolygonABufferSizes[i] = n + intersectCount;
        LLPolygonBBufferSizes[i] = k + intersectCount;
    }
}

__global__ void kernel_build_LL(LLPolyVertex *LLPolygonBuffers,
                                GPUMemory::GPUPolygon polygon,
                                int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        // "Pointer" to the element after the last valid element of the linked list
        int32_t LLPolygonEndIdx = 0;

        int32_t polyIdx = GPUMemory::PolyIdxAt(polygon, i);
        int32_t polyCount = GPUMemory::PolyCountAt(polygon, i);

        // Transform polygon
        for(int32_t p = polyIdx; p < (polyIdx + polyCount); p++)
        {
            int32_t pointIdx = GPUMemory::PointIdxAt(polygon, p);
            int32_t pointCount = GPUMemory::PointCountAt(polygon, p);

            for(int32_t point = pointIdx; point < (pointIdx + pointCount); point++)
            {
                // Set the linked list entry
                LLPolygonBuffers[point] = {
                    polygon.polyPoints[point], 
                    false, 
                    false, 
                    -1.0, 
                    -1.0, 
                    pointIdx + (point - 1 + pointCount) % pointCount,
                    pointIdx + (point + 1) % pointCount, 
                    -1
                };
            }  
        }
    }
}