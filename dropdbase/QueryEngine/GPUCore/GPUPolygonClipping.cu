#include "hip/hip_runtime.h"
#include "GPUPolygonClipping.cuh"

__device__ LLPolyVertex calc_intersect(NativeGeoPoint sA, NativeGeoPoint eA,
                                       NativeGeoPoint sB, NativeGeoPoint eB)
{
    float adx = eA.latitude - sA.latitude;
    float ady = eA.longitude - sA.longitude;
    float bdx = eB.latitude - sB.latitude;
    float bdy = eB.longitude - sB.longitude;

    float axb = adx * bdy - ady * bdx;

    if (axb == 0)
    {
        LLPolyVertex retFail = {0, 0, true, false, -1, -1, -1, -1, -1};
        return retFail;
    }

    float dx = sA.latitude - sB.latitude;
    float dy = sA.longitude - sB.longitude;

    float alongA = (bdx * dy - bdy * dx) / axb;
    float alongB = (adx * dy - ady * dx) / axb;

    bool intersectionValidity = (alongA > 0 && alongA < 1 && alongB > 0 && alongB < 1);

    LLPolyVertex ret = {
        sA.latitude + alongA * adx,
        sA.longitude + alongA * ady,
        true,
        intersectionValidity,
        alongA,
        alongB,
        -1,
        -1,
        -1
    };

    return ret;
}

__global__ void kernel_calc_intersection_counts(int32_t *intesection_counts, 
                                                GPUMemory::GPUPolygon polygonA,
                                                GPUMemory::GPUPolygon polygonB,
                                                int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        int32_t polyIdxA = GPUMemory::PolyIdxAt(polygonA, i);
        int32_t polyCountA = GPUMemory::PolyCountAt(polygonA, i);

        int32_t polyIdxB = GPUMemory::PolyIdxAt(polygonB, i);
        int32_t polyCountB = GPUMemory::PolyCountAt(polygonB, i);

        // Calcualte the intersection count
        int32_t intersectCount = 0;
        for(int32_t a = polyIdxA; a < (polyIdxA + polyCountA); a++)
        {
            int32_t pointIdxA = GPUMemory::PointIdxAt(polygonA, a);
            int32_t pointCountA = GPUMemory::PointCountAt(polygonA, a);
            for(int32_t b = polyIdxB; b < (polyIdxB + polyCountB); b++)
            {
                int32_t pointIdxB = GPUMemory::PointIdxAt(polygonB, b);
                int32_t pointCountB = GPUMemory::PointCountAt(polygonB, b);

                // Calculate intersections
                for(int32_t pointA = pointIdxA; pointA < (pointIdxA + pointCountA); pointA++)
                {
                    for(int32_t pointB = pointIdxB; pointB < (pointIdxB + pointCountB); pointB++)
                    {
                        LLPolyVertex result = calc_intersect(polygonA.polyPoints[pointA], 
                                                             polygonA.polyPoints[pointIdxA + (pointA + 1) % pointCountA],
                                                             polygonA.polyPoints[pointB],
                                                             polygonA.polyPoints[pointIdxB + (pointB + 1) % pointCountB]);
                        if(result.isValidIntersection)
                        {
                            intersectCount++;
                        }
                    }       
                }
            }   
        }
        intesection_counts[i] = intersectCount;
    }
}
