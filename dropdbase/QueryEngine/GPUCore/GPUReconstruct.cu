#include "hip/hip_runtime.h"
#include "GPUReconstruct.cuh"
#include "cuda_ptr.h"

// Polygon WKT format:
// POLYGON((179.9999 89.9999, 0.0000 0.0000, 179.9999 89.9999), (-179.9999 -89.9999, 52.1300 -27.0380, -179.9999 -89.9999))


__global__ void kernel_reconstruct_string_chars(GPUMemory::GPUString outStringCol,
	GPUMemory::GPUString inStringCol, int32_t * inStringLengths,
	int32_t *prefixSum, int8_t *inMask, int32_t stringCount)
{
	const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t stride = blockDim.x * gridDim.x;

	for (int32_t i = idx; i < stringCount; i += stride)
	{
		if (inMask[i] && (prefixSum[i] - 1) >= 0)
		{
			int64_t inIndex = (i == 0)? 0 : inStringCol.stringIndices[i - 1];
			int64_t outIndex = (prefixSum[i] - 1 == 0)? 0 : outStringCol.stringIndices[prefixSum[i] - 2];
			for (int32_t j = 0; j < inStringLengths[i]; j++)
			{
				outStringCol.allChars[outIndex + j] = inStringCol.allChars[inIndex + j];
			}
		}
	}
}

/// Helping function to calculate number of digints of integer part of float
__device__ int32_t GetNumberOfIntegerPartDigits(float number)
{
	return (floorf(fabsf(number)) > 3.0f ?
		static_cast<int32_t>(log10f(floorf(fabsf(number)))) : 0) + 1 + (number < 0 ? 1 : 0);
}

__global__ void kernel_predict_wkt_lengths(int32_t * outStringLengths, GPUMemory::GPUPolygon inPolygonCol, int32_t dataElementCount)
{
	const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t stride = blockDim.x * gridDim.x;

	for (int32_t i = idx; i < dataElementCount; i += stride)
	{
		// Count POLYGON word and parentheses ("POLYGON((), ())")
		int32_t charCounter = 11 + (4 * (GPUMemory::PolyCountAt(inPolygonCol, i) - 1));
		const int32_t subpolyStartIdx = GPUMemory::PolyIdxAt(inPolygonCol, i);
		const int32_t subpolyEndIdx = subpolyStartIdx + GPUMemory::PolyCountAt(inPolygonCol, i);
		for (int32_t j = subpolyStartIdx; j < subpolyEndIdx; j++)
		{
			const int32_t pointCount = GPUMemory::PointCountAt(inPolygonCol, j) - 2;
			const int32_t pointStartIdx = GPUMemory::PointIdxAt(inPolygonCol, j) + 1;
			const int32_t pointEndIdx = pointStartIdx + pointCount;

			// Count the decimal part and colons between points (".0000 .0000, .0000 .0000")
			charCounter += pointCount * (2 * WKT_DECIMAL_PLACES + 5) - 2;
			for (int32_t k = pointStartIdx; k < pointEndIdx; k++)
			{
				// Count the integer part ("150".0000, "-0".1000)
				charCounter += GetNumberOfIntegerPartDigits(inPolygonCol.polyPoints[k].latitude) +
					GetNumberOfIntegerPartDigits(inPolygonCol.polyPoints[k].longitude);
			}
		}
		outStringLengths[i] = charCounter;
	}
}

__global__ void kernel_generate_poly_submask(int8_t *outMask, int8_t *inMask, GPUMemory::GPUPolygon polygon, int32_t size)
{
	const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t stride = blockDim.x * gridDim.x;

	for (int32_t i = idx; i < size; i += stride)
	{
		for (int32_t j = 0; j < GPUMemory::PolyCountAt(polygon, i); j++)
		{
			outMask[GPUMemory::PolyIdxAt(polygon, i) + j] = inMask[i];
		}
	}
}

__global__ void kernel_generate_point_submask(int8_t *outMask, int8_t *inMask, GPUMemory::GPUPolygon polygon, int32_t size)
{
	const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t stride = blockDim.x * gridDim.x;

	for (int32_t i = idx; i < size; i += stride)
	{
		for (int32_t j = 0; j < GPUMemory::PointCountAt(polygon, i); j++)
		{
			outMask[GPUMemory::PointIdxAt(polygon, i) + j] = inMask[i];
		}
	}
}

/// Kernel for reconstructing polygon subPolygons
__global__ void kernel_reconstruct_polyCount_col(int32_t * outPolyCount, 
											GPUMemory::GPUPolygon polygon, 
											int32_t *prefixSum, 
											int8_t *inMask, 
											int32_t dataElementCount)
{
	const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t stride = blockDim.x * gridDim.x;

	for (int32_t i = idx; i < dataElementCount; i += stride)
	{
		if (inMask[i] && (prefixSum[i] - 1) >= 0)
		{
			outPolyCount[prefixSum[i] - 1] = GPUMemory::PolyCountAt(polygon, i);
		}
	}
}

/// Kernel for reconstructing polygon points
__global__ void kernel_reconstruct_pointCount_col(int32_t *outPointCount, 
												  GPUMemory::GPUPolygon polygon, 
												  int32_t *prefixSum, 
												  int8_t *inMask, 
												  int32_t dataElementCount)
{
	const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t stride = blockDim.x * gridDim.x;

	for (int32_t i = idx; i < dataElementCount; i += stride)
	{
		if (inMask[i] && (prefixSum[i] - 1) >= 0)
		{
			outPointCount[prefixSum[i] - 1] = GPUMemory::PointCountAt(polygon, i);
		}
	}
}

__global__ void kernel_predict_point_wkt_lengths(int32_t * outStringLengths, NativeGeoPoint* inPointCol, int32_t dataElementCount)
{
	const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t stride = blockDim.x * gridDim.x;

	for (int32_t i = idx; i < dataElementCount; i += stride)
	{
		// Count POINT word and parentheses ("POINT()")
		int32_t charCounter = 7;
		// Count the integer part ("150".0000 "-0".1000)
		charCounter += GetNumberOfIntegerPartDigits(inPointCol[i].latitude) + GetNumberOfIntegerPartDigits(inPointCol[i].longitude);
		// Count the decimal part, space and dots between points (".0000 .0000")
		charCounter += 2 * WKT_DECIMAL_PLACES + 3;
		outStringLengths[i] = charCounter;
	}
}


/// Helping function to "print" float to GPU char array
__device__ void FloatToString(char * allChars, int64_t &startIndex, float number)
{
	// Append sign
	if (number < 0)
	{
		allChars[startIndex] = '-';
		// (note that there is no addres move because we will count with negative sign later)
	}

	// Append integer part
	int32_t integerPart = static_cast<int32_t>(floorf(fabsf(number)));
	int32_t digits = GetNumberOfIntegerPartDigits(number);
	startIndex += digits;
	do
	{
		allChars[--startIndex] = ('0' + (integerPart % 10));
		integerPart /= 10;
	} while (integerPart > 0);		// Dynamic integer part places
	startIndex += digits - (number < 0 ? 1 : 0);

	// Append decimal part
	int32_t decimalPart = static_cast<int32_t>(roundf(fmodf(fabsf(number), 1.0f)*powf(10.0f, WKT_DECIMAL_PLACES)));
	allChars[startIndex++] = '.';
	startIndex += WKT_DECIMAL_PLACES;
	for (int32_t i = 0; i < WKT_DECIMAL_PLACES; i++)	// Fixed decimal places
	{
		allChars[--startIndex] = ('0' + (decimalPart % 10));
		decimalPart /= 10;
	}
	startIndex += WKT_DECIMAL_PLACES;
}


__global__ void kernel_convert_poly_to_wkt(GPUMemory::GPUString outWkt, GPUMemory::GPUPolygon inPolygonCol, int32_t dataElementCount)
{
	const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t stride = blockDim.x * gridDim.x;

	for (int32_t i = idx; i < dataElementCount; i += stride)	// via complex polygons
	{
		// "POLYGON("
		const int64_t stringStartIndex = (i == 0 ? 0 : outWkt.stringIndices[i - 1]);
		for (int32_t j = 0; j < 7; j++)
		{
			outWkt.allChars[stringStartIndex + j] = WKT_POLYGON[j];
		}
		int64_t charId = stringStartIndex + 7;
		outWkt.allChars[charId++] = '(';

		const int32_t subpolyStartIdx = GPUMemory::PolyIdxAt(inPolygonCol, i);
		const int32_t subpolyEndIdx = subpolyStartIdx + GPUMemory::PolyCountAt(inPolygonCol, i);

		for (int32_t j = subpolyStartIdx; j < subpolyEndIdx; j++)	// via sub-polygons
		{
			outWkt.allChars[charId++] = '(';
			const int32_t pointCount = GPUMemory::PointCountAt(inPolygonCol, j) - 2;
			const int32_t pointStartIdx = GPUMemory::PointIdxAt(inPolygonCol, j) + 1;
			const int32_t pointEndIdx = pointStartIdx + pointCount;

			for (int32_t k = pointStartIdx; k < pointEndIdx; k++)	// via points
			{
				FloatToString(outWkt.allChars, charId, inPolygonCol.polyPoints[k].latitude);
				outWkt.allChars[charId++] = ' ';
				FloatToString(outWkt.allChars, charId, inPolygonCol.polyPoints[k].longitude);

				if (k < pointEndIdx - 1)
				{
					outWkt.allChars[charId++] = ',';
					outWkt.allChars[charId++] = ' ';
				}
			}

			outWkt.allChars[charId++] = ')';
			if (j < subpolyEndIdx - 1)
			{
				outWkt.allChars[charId++] = ',';
				outWkt.allChars[charId++] = ' ';
			}
		}
		outWkt.allChars[charId++] = ')';
		
		// Lengths mis-match check
		if (charId != outWkt.stringIndices[i])
		{
			printf("Not match fin id! %d\n", outWkt.stringIndices[i] - charId);
		}
		else{
			printf("Match OK\n");
		}
		
	}
}

__global__ void kernel_convert_point_to_wkt(GPUMemory::GPUString outWkt, NativeGeoPoint* inPointCol, int32_t dataElementCount)
{
	const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t stride = blockDim.x * gridDim.x;

	for (int32_t i = idx; i < dataElementCount; i += stride)	// via points
	{
		// "POINT"
		const int64_t stringStartIndex = (i == 0 ? 0 : outWkt.stringIndices[i - 1]);
		for (int32_t j = 0; j < 5; j++)
		{
			outWkt.allChars[stringStartIndex + j] = WKT_POINT[j];
		}
		int64_t charId = stringStartIndex + 5;

		outWkt.allChars[charId++] = '(';
		FloatToString(outWkt.allChars, charId, inPointCol[i].latitude);
		outWkt.allChars[charId++] = ' ';
		FloatToString(outWkt.allChars, charId, inPointCol[i].longitude);
		outWkt.allChars[charId++] = ')';

		/*
		// Lengths mis-match check
		if (charId != outWkt.stringIndices[i])
		{
			printf("Not match fin id! %d\n", outWkt.stringIndices[i] - charId);
		}
		*/
	}
}

void GPUReconstruct::ReconstructStringColKeep(GPUMemory::GPUString *outStringCol, int32_t *outDataElementCount,
	GPUMemory::GPUString inStringCol, int8_t *inMask, int32_t inDataElementCount)
{
	Context& context = Context::getInstance();

	if (inMask)		// If mask is used (if inMask is not nullptr)
	{
		// Malloc a new buffer for the prefix sum vector
		cuda_ptr<int32_t> inPrefixSumPointer(inDataElementCount);
		PrefixSum(inPrefixSumPointer.get(), inMask, inDataElementCount);
		GPUMemory::copyDeviceToHost(outDataElementCount, inPrefixSumPointer.get() + inDataElementCount - 1, 1);

		if (*outDataElementCount > 0)	// Not empty result set
		{
			int64_t inTotalCharCount;
			GPUMemory::copyDeviceToHost(&inTotalCharCount, inStringCol.stringIndices + inDataElementCount - 1, 1);

			// Compute lenghts from indices (reversed inclusive prefix sum)
			cuda_ptr<int32_t> inLengths(inDataElementCount);
			kernel_lengths_from_indices << < context.calcGridDim(inDataElementCount), context.getBlockDim() >> >
				(inLengths.get(), inStringCol.stringIndices, inDataElementCount);

			// Reconstruct lenghts according to mask
			cuda_ptr<int32_t> outLengths(*outDataElementCount);
			kernel_reconstruct_col << < context.calcGridDim(inDataElementCount), context.getBlockDim() >> >
				(outLengths.get(), inLengths.get(), inPrefixSumPointer.get(), inMask, inDataElementCount);

			// Compute new indices as prefix sum of reconstructed lengths
			GPUMemory::alloc(&(outStringCol->stringIndices), *outDataElementCount);
			PrefixSum(outStringCol->stringIndices, outLengths.get(), *outDataElementCount);

			int64_t outTotalCharCount;
			GPUMemory::copyDeviceToHost(&outTotalCharCount, outStringCol->stringIndices + *outDataElementCount - 1, 1);
			GPUMemory::alloc(&(outStringCol->allChars), outTotalCharCount);

			// Reconstruct chars
			kernel_reconstruct_string_chars << < context.calcGridDim(inDataElementCount), context.getBlockDim() >> >
				(*outStringCol, inStringCol, inLengths.get(), inPrefixSumPointer.get(), inMask, inDataElementCount);
		}
		else	// Empty result set
		{
			outStringCol->allChars = nullptr;
			outStringCol->stringIndices = nullptr;
		}
	}
	else	// If mask is not used (is nullptr), just copy pointers from inCol to outCol
	{
		*outStringCol = inStringCol;
		*outDataElementCount = inDataElementCount;
	}

	// Get last error
	CheckCudaError(hipGetLastError());
}


void GPUReconstruct::ReconstructStringCol(std::string *outStringData, int32_t *outDataElementCount,
	GPUMemory::GPUString inStringCol, int8_t *inMask, int32_t inDataElementCount)
{
	GPUMemory::GPUString outStringCol;
	if (inMask)		// If mask is used (if inMask is not nullptr)
	{
		ReconstructStringColKeep(&outStringCol, outDataElementCount, inStringCol, inMask, inDataElementCount);
	}
	else	// If mask is not used
	{
		*outDataElementCount = inDataElementCount;
		outStringCol = inStringCol;
	}

	if (*outDataElementCount > 0)
	{
		// Copy string indices to host
		std::unique_ptr<int64_t[]> hostStringIndices = std::make_unique<int64_t[]>(*outDataElementCount);
		GPUMemory::copyDeviceToHost(hostStringIndices.get(), outStringCol.stringIndices, *outDataElementCount);
		int64_t fullCharCount = hostStringIndices[*outDataElementCount - 1];

		// Copy all chars to host
		std::unique_ptr<char[]> hostAllChars = std::make_unique<char[]>(fullCharCount);
		GPUMemory::copyDeviceToHost(hostAllChars.get(), outStringCol.allChars, fullCharCount);

		// Fill output string array
		for (int32_t i = 0; i < *outDataElementCount; i++)
		{
			size_t length = static_cast<size_t>(i == 0 ? hostStringIndices[0] :
				hostStringIndices[i] - hostStringIndices[i - 1]);
			outStringData[i] = std::string(hostAllChars.get() +
				(i == 0 ? 0 : hostStringIndices[i - 1]), length);
		}
		// Free GPUString because it is not going out
		GPUMemory::free(outStringCol);
	}
}


void GPUReconstruct::ReconstructStringColRaw(std::vector<int32_t>& keysStringLengths, std::vector<char>& keysAllChars,
	int32_t *outDataElementCount, GPUMemory::GPUString inStringCol, int8_t *inMask, int32_t inDataElementCount)
{
	Context& context = Context::getInstance();

	if (inMask)		// If mask is used (if inMask is not nullptr)
	{
		// Malloc a new buffer for the prefix sum vector
		cuda_ptr<int32_t> inPrefixSumPointer(inDataElementCount);
		PrefixSum(inPrefixSumPointer.get(), inMask, inDataElementCount);
		GPUMemory::copyDeviceToHost(outDataElementCount, inPrefixSumPointer.get() + inDataElementCount - 1, 1);

		if (*outDataElementCount > 0)	// Not empty result set
		{
			// Compute lenghts from indices (reversed inclusive prefix sum)
			cuda_ptr<int32_t> inLengths(inDataElementCount);
			kernel_lengths_from_indices << < context.calcGridDim(inDataElementCount), context.getBlockDim() >> >
				(inLengths.get(), inStringCol.stringIndices, inDataElementCount);

			// Reconstruct lenghts according to mask
			cuda_ptr<int32_t> outLengths(*outDataElementCount);
			kernel_reconstruct_col << < context.calcGridDim(inDataElementCount), context.getBlockDim() >> >
				(outLengths.get(), inLengths.get(), inPrefixSumPointer.get(), inMask, inDataElementCount);
			// Copy lengths to host
			keysStringLengths.resize(*outDataElementCount);
			GPUMemory::copyDeviceToHost(keysStringLengths.data(), outLengths.get(), *outDataElementCount);

			// Compute new indices as prefix sum of reconstructed lengths
			GPUMemory::GPUString outStringCol;
			GPUMemory::alloc(&(outStringCol.stringIndices), *outDataElementCount);
			PrefixSum(outStringCol.stringIndices, outLengths.get(), *outDataElementCount);

			int64_t outTotalCharCount;
			GPUMemory::copyDeviceToHost(&outTotalCharCount, outStringCol.stringIndices + *outDataElementCount - 1, 1);
			GPUMemory::alloc(&(outStringCol.allChars), outTotalCharCount);

			// Reconstruct chars
			kernel_reconstruct_string_chars << < context.calcGridDim(inDataElementCount), context.getBlockDim() >> >
				(outStringCol, inStringCol, inLengths.get(), inPrefixSumPointer.get(), inMask, inDataElementCount);
			// Copy chars to host
			keysAllChars.resize(outTotalCharCount);
			GPUMemory::copyDeviceToHost(keysAllChars.data(), outStringCol.allChars, outTotalCharCount);
			GPUMemory::free(outStringCol);
		}
	}
	else	// If mask is not used (is nullptr), just copy pointers from inCol to outCol
	{
		*outDataElementCount = inDataElementCount;

		// Compute lenghts from indices (reversed inclusive prefix sum)
		int64_t outTotalCharCount;
		GPUMemory::copyDeviceToHost(&outTotalCharCount, inStringCol.stringIndices + inDataElementCount - 1, 1);
		cuda_ptr<int32_t> lengths(inDataElementCount);
		kernel_lengths_from_indices << < context.calcGridDim(inDataElementCount), context.getBlockDim() >> >
			(lengths.get(), inStringCol.stringIndices, inDataElementCount);
		keysStringLengths.resize(inDataElementCount);
		GPUMemory::copyDeviceToHost(keysStringLengths.data(), lengths.get(), inDataElementCount);
		keysAllChars.resize(outTotalCharCount);
		GPUMemory::copyDeviceToHost(keysAllChars.data(), inStringCol.allChars, outTotalCharCount);
	}

	// Get last error
	CheckCudaError(hipGetLastError());
}


void GPUReconstruct::ConvertPolyColToWKTCol(GPUMemory::GPUString *outStringCol,
											GPUMemory::GPUPolygon inPolygonCol, 
											int32_t dataElementCount)
{
	Context& context = Context::getInstance();
	if (dataElementCount > 0)
	{
		// "Predict" (pre-calculate) string lengths
		cuda_ptr<int32_t>stringLengths(dataElementCount);
		kernel_predict_wkt_lengths << < context.calcGridDim(dataElementCount), context.getBlockDim() >> >
			(stringLengths.get(), inPolygonCol, dataElementCount);
		CheckCudaError(hipGetLastError());

		// Alloc and compute string indices as a prefix sum of the string lengths
		GPUMemory::alloc(&(outStringCol->stringIndices), dataElementCount);
		PrefixSum(outStringCol->stringIndices, stringLengths.get(), dataElementCount);

		// Get total char count and alloc array for all chars
		int64_t totalCharCount;
		GPUMemory::copyDeviceToHost(&totalCharCount, outStringCol->stringIndices + dataElementCount - 1, 1);
		GPUMemory::alloc(&(outStringCol->allChars), totalCharCount);

		// Finally convert polygons to WKTs
		kernel_convert_poly_to_wkt << < context.calcGridDim(dataElementCount), context.getBlockDim() >> >
			(*outStringCol, inPolygonCol, dataElementCount);
		CheckCudaError(hipGetLastError());
	}
	else
	{
		outStringCol->allChars = nullptr;
		outStringCol->stringIndices = nullptr;
	}
}

void GPUReconstruct::ConvertPointColToWKTCol(GPUMemory::GPUString *outStringCol,
	NativeGeoPoint* inPointCol, int32_t dataElementCount)
{
	Context& context = Context::getInstance();
	if (dataElementCount > 0)
	{
		// "Predict" (pre-calculate) string lengths
		cuda_ptr<int32_t>stringLengths(dataElementCount);
		kernel_predict_point_wkt_lengths << < context.calcGridDim(dataElementCount), context.getBlockDim() >> >
			(stringLengths.get(), inPointCol, dataElementCount);
		CheckCudaError(hipGetLastError());

		// Alloc and compute string indices as a prefix sum of the string lengths
		GPUMemory::alloc(&(outStringCol->stringIndices), dataElementCount);
		PrefixSum(outStringCol->stringIndices, stringLengths.get(), dataElementCount);

		// Get total char count and alloc array for all chars
		int64_t totalCharCount;
		GPUMemory::copyDeviceToHost(&totalCharCount, outStringCol->stringIndices + dataElementCount - 1, 1);
		GPUMemory::alloc(&(outStringCol->allChars), totalCharCount);

		// Finally convert points to WKTs
		kernel_convert_point_to_wkt << < context.calcGridDim(dataElementCount), context.getBlockDim() >> >
			(*outStringCol, inPointCol, dataElementCount);
		CheckCudaError(hipGetLastError());
	}
	else
	{
		outStringCol->allChars = nullptr;
		outStringCol->stringIndices = nullptr;
	}
}


void GPUReconstruct::ReconstructPolyColKeep(GPUMemory::GPUPolygon *outCol, 
											int32_t *outDataElementCount,
											GPUMemory::GPUPolygon inCol, 
											int8_t *inMask, 
											int32_t inDataElementCount)
{
	Context& context = Context::getInstance();

	if (inMask)		// If mask is used (if inMask is not nullptr)
	{
		// A buffer for the prefix sum vector
		cuda_ptr<int32_t> inPrefixSumPointer(inDataElementCount);
		PrefixSum(inPrefixSumPointer.get(), inMask, inDataElementCount);
		GPUMemory::copyDeviceToHost(outDataElementCount, inPrefixSumPointer.get() + inDataElementCount - 1, 1);

		if (*outDataElementCount > 0)	// Not empty result set
		{
			// Reconstruct each array independently
			int32_t inSubpolySize;
			int32_t inPointSize;

			GPUMemory::copyDeviceToHost(&inSubpolySize, inCol.polyIdx + inDataElementCount - 1, 1);
			GPUMemory::copyDeviceToHost(&inPointSize, inCol.pointIdx + inSubpolySize - 1 , 1);

			// Complex polygons (reconstruct polyCount and sum it to polyIdx)
			// Alloc a temp count buffer and the result index buffer
			cuda_ptr<int32_t> polyCount(*outDataElementCount);
			GPUMemory::alloc(&(outCol->polyIdx), *outDataElementCount);

			kernel_reconstruct_polyCount_col << < context.calcGridDim(inDataElementCount), context.getBlockDim() >> >
				(polyCount.get(), inCol, inPrefixSumPointer.get(), inMask, inDataElementCount);
			CheckCudaError(hipGetLastError());

			PrefixSum(outCol->polyIdx, polyCount.get(), *outDataElementCount);

			// Subpolygons (reconstruct pointCount and sum it to pointIdx)
			int32_t outSubpolySize;
			GPUMemory::copyDeviceToHost(&outSubpolySize, outCol->polyIdx + *outDataElementCount - 1, 1);

			cuda_ptr<int8_t> subpolyMask(inSubpolySize);
			kernel_generate_poly_submask << < context.calcGridDim(inDataElementCount), context.getBlockDim() >> >
				(subpolyMask.get(), inMask, inCol, inDataElementCount);
			CheckCudaError(hipGetLastError());

			cuda_ptr<int32_t> subpolyPrefixSumPointer(inSubpolySize);
			PrefixSum(subpolyPrefixSumPointer.get(), subpolyMask.get(), inSubpolySize);

			cuda_ptr<int32_t> pointCount(outSubpolySize);
			GPUMemory::alloc(&(outCol->pointIdx), outSubpolySize);

			kernel_reconstruct_pointCount_col << < context.calcGridDim(inSubpolySize), context.getBlockDim() >> >
				(pointCount.get(), inCol, subpolyPrefixSumPointer.get(), subpolyMask.get(), inSubpolySize);
			CheckCudaError(hipGetLastError());
			PrefixSum(outCol->pointIdx, pointCount.get(), outSubpolySize);

			// Points (reconstruct polyPoints)
			int32_t outPointSize;
			GPUMemory::copyDeviceToHost(&outPointSize, outCol->pointIdx + outSubpolySize - 1, 1);

			cuda_ptr<int8_t> pointMask(inPointSize);
			kernel_generate_point_submask << < context.calcGridDim(inSubpolySize), context.getBlockDim() >> >
				(pointMask.get(), subpolyMask.get(), inCol, inSubpolySize);
			CheckCudaError(hipGetLastError());

			cuda_ptr<int32_t> pointPrefixSumPointer(inPointSize);
			PrefixSum(pointPrefixSumPointer.get(), pointMask.get(), inPointSize);

			GPUMemory::alloc(&(outCol->polyPoints), outPointSize);
			kernel_reconstruct_col << < context.calcGridDim(inSubpolySize), context.getBlockDim() >> >
				(outCol->polyPoints, inCol.polyPoints, pointPrefixSumPointer.get(), pointMask.get(), inPointSize);
			CheckCudaError(hipGetLastError());
		}
		else	// Empty result set
		{
			outCol->polyPoints = nullptr;
			outCol->pointIdx = nullptr;
			outCol->polyIdx = nullptr;
		}
	}
	else	// If mask is not used (is nullptr), just copy pointers from inCol to outCol
	{
		*outCol = inCol;
		*outDataElementCount = inDataElementCount;
	}

	// Get last error
	CheckCudaError(hipGetLastError());
}


void GPUReconstruct::ReconstructPolyColToWKT(std::string *outStringData, int32_t *outDataElementCount,
	GPUMemory::GPUPolygon inPolygonCol, int8_t *inMask, int32_t inDataElementCount)
{
	GPUMemory::GPUPolygon reconstructedPolygonCol;
	ReconstructPolyColKeep(&reconstructedPolygonCol, outDataElementCount, inPolygonCol, inMask, inDataElementCount);
	GPUMemory::GPUString gpuWkt;
	ConvertPolyColToWKTCol(&gpuWkt, reconstructedPolygonCol, *outDataElementCount);
	GPUMemory::free(reconstructedPolygonCol);
	// Use reconstruct without mask - just to convert GPUString to CPU string array
	ReconstructStringCol(outStringData, outDataElementCount, gpuWkt, nullptr, *outDataElementCount);
	GPUMemory::free(gpuWkt);
}

void GPUReconstruct::ReconstructPointColToWKT(std::string * outStringData, int32_t * outDataElementCount, NativeGeoPoint* inPointCol, int8_t * inMask, int32_t inDataElementCount)
{
	NativeGeoPoint* reconstructedPointCol;
	reconstructColKeep<NativeGeoPoint>(&reconstructedPointCol, outDataElementCount, inPointCol, inMask, inDataElementCount);
	GPUMemory::GPUString gpuWkt;
	ConvertPointColToWKTCol(&gpuWkt, reconstructedPointCol, *outDataElementCount);
	GPUMemory::free(reconstructedPointCol);
	// Use reconstruct without mask - just to convert GPUString to CPU string array
	ReconstructStringCol(outStringData, outDataElementCount, gpuWkt, nullptr, *outDataElementCount);
	GPUMemory::free(gpuWkt);
}


template<>
void GPUReconstruct::reconstructCol<ColmnarDB::Types::Point>(ColmnarDB::Types::Point *outData,
	int32_t *outDataElementCount, ColmnarDB::Types::Point *ACol, int8_t *inMask, int32_t dataElementCount)
{
	// Not supported, just throw an error
	CheckQueryEngineError(QueryEngineErrorType::GPU_EXTENSION_ERROR,
		"ReconstructCol of Point not supported, use GenerateIndexes instead");
}

template<>
void GPUReconstruct::reconstructCol<ColmnarDB::Types::ComplexPolygon>(ColmnarDB::Types::ComplexPolygon *outData,
	int32_t *outDataElementCount, ColmnarDB::Types::ComplexPolygon *ACol, int8_t *inMask, int32_t dataElementCount)
{
	// Not supported, just throw an error
	CheckQueryEngineError(QueryEngineErrorType::GPU_EXTENSION_ERROR,
		"ReconstructCol of ComplexPolygon not supported, use GenerateIndexes instead");
}

template<>
void GPUReconstruct::reconstructColKeep<ColmnarDB::Types::Point>(ColmnarDB::Types::Point **outCol,
	int32_t *outDataElementCount, ColmnarDB::Types::Point *ACol, int8_t *inMask, int32_t dataElementCount)
{
	// Not supported, just throw an error
	CheckQueryEngineError(QueryEngineErrorType::GPU_EXTENSION_ERROR,
		"ReconstructColKeep of Point not supported, use GenerateIndexes instead");
}

template<>
void GPUReconstruct::reconstructColKeep<ColmnarDB::Types::ComplexPolygon>(ColmnarDB::Types::ComplexPolygon **outCol,
	int32_t *outDataElementCount, ColmnarDB::Types::ComplexPolygon *ACol, int8_t *inMask, int32_t dataElementCount)
{
	// Not supported, just throw an error
	CheckQueryEngineError(QueryEngineErrorType::GPU_EXTENSION_ERROR,
		"ReconstructColKeep of ComplexPolygon not supported, use GenerateIndexes instead");
}
