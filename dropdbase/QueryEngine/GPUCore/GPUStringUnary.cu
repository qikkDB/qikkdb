#include "hip/hip_runtime.h"
#include "GPUStringUnary.cuh"
#include "GPUReconstruct.cuh"
#include "cuda_ptr.h"


__global__ void kernel_reverse_string(GPUMemory::GPUString outCol, GPUMemory::GPUString inCol, int64_t stringCount)
{
	const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t stride = blockDim.x * gridDim.x;

	for (int32_t i = idx; i < stringCount; i += stride)
	{
		const int64_t firstCharIndex = (i == 0) ? 0 : inCol.stringIndices[i - 1];
		const int64_t length = inCol.stringIndices[i] - firstCharIndex;
		const int64_t lastCharIndex = inCol.stringIndices[i] - 1;

		for (int32_t j = 0; j < length; j++)
		{
			outCol.allChars[firstCharIndex + j] = inCol.allChars[lastCharIndex - j];
		}
	}
}


void GPUStringUnary::ColLen(int32_t * outCol, GPUMemory::GPUString inCol, int32_t dataElementCount)
{
	Context& context = Context::getInstance();
	kernel_lengths_from_indices << < context.calcGridDim(dataElementCount), context.getBlockDim() >> >
		(outCol, inCol.stringIndices, dataElementCount);
	CheckCudaError(hipGetLastError());
}
