#include "hip/hip_runtime.h"
#include "GPUOrderBy.cuh"
#include "GPUStringUnary.cuh"

// Fill the index buffers with default indices
__global__ void kernel_fill_indices(int32_t* indices, int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        indices[i] = i;
    }
}

__global__ void kernel_reorder_chars_by_idx(GPUMemory::GPUString outCol,
                                            int32_t* inIndices,
                                            GPUMemory::GPUString inCol,
                                            int64_t* outStringIndices,
                                            int32_t* outStringLengths,
                                            int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        int32_t outColIdx = GetStringIndex(outStringIndices, i);
        int32_t inColIdx = GetStringIndex(inCol.stringIndices, inIndices[i]);
        for (int32_t j = 0; j < outStringLengths[i]; j++)
        {
            outCol.allChars[outColIdx + j] = inCol.allChars[inColIdx + j];
        }
        outCol.stringIndices[i] = outStringIndices[i];
    }
}

__global__ void kernel_reorder_points_by_idx(GPUMemory::GPUPolygon outCol,
                                             int32_t* inIndices,
                                             GPUMemory::GPUPolygon inCol,
                                             int32_t* outPolygonIndices,
                                             int32_t* outPolygonLengths,
                                             int32_t* outPointIndices,
                                             int32_t* outPointLengths,
                                             int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        int32_t outPolygonIdx = outPolygonIndices[i];
        int32_t inPolygonIdx = inCol.polyIdx[inIndices[i]];
        for (int32_t j = 0; j < outPolygonLengths[i]; j++)
        {
            int32_t outPointIdx = outPointIndices[outPolygonIdx];
            int32_t inPointIdx = inCol.pointIdx[inPolygonIdx];
            for (int32_t k = 0; k < outPointLengths[outPolygonIdx]; k++)
            {
                outCol.polyPoints[outPointIdx + k] = inCol.polyPoints[inPointIdx + k];
            }
            outCol.pointIdx[outPolygonIdx + j] = inCol.pointIdx[inPolygonIdx + j];
            outCol.pointCount[outPolygonIdx + j] = inCol.pointCount[inPolygonIdx + j];
        }
        outCol.polyIdx[i] = outPolygonIndices[i];
        outCol.polyCount[i] = outPolygonLengths[i];
    }
}

__global__ void kernel_reorder_poly_lengths_by_cp_idx_and_cp_lenghts(int32_t* outPointLengths,
                                                                     int32_t* inOrderIndices,
                                                                     int32_t* inPointLenghts,
                                                                     int32_t* inPolygonIndices,
                                                                     int32_t* outPolygonIndices,
                                                                     int32_t* outPolygonLengths,
                                                                     int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        int32_t outPointIdx = outPolygonIndices[i];
        int32_t inPointIdx = inPolygonIndices[inOrderIndices[i]];
        for (int32_t j = 0; j < outPolygonLengths[i]; j++)
        {
            outPointLengths[outPointIdx + j] = inPointLenghts[inPointIdx + j];
        }
    }
}

// Reorder a null column by a given index column
__global__ void
kernel_reorder_null_values_by_idx(int32_t* outNullBitMask, int32_t* inIndices, int8_t* inNullBitMask, int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        int32_t nullBit =
            (inNullBitMask[inIndices[i] / (sizeof(int8_t) * 8)] >> (inIndices[i] % (sizeof(int8_t) * 8))) & 1;
        nullBit <<= (i % (sizeof(int32_t) * 8));
        atomicOr(outNullBitMask + (i / (sizeof(int32_t) * 8)), nullBit);
    }
}

GPUOrderBy::GPUOrderBy(int32_t dataElementCount)
{
    GPUMemory::alloc(&indices1, dataElementCount);
    GPUMemory::alloc(&indices2, dataElementCount);

    // Initialize the index buffer
    kernel_fill_indices<<<Context::getInstance().calcGridDim(dataElementCount),
                          Context::getInstance().getBlockDim()>>>(indices1, dataElementCount);
}

GPUOrderBy::~GPUOrderBy()
{
    GPUMemory::free(indices1);
    GPUMemory::free(indices2);
}

void GPUOrderBy::ReOrderNullValuesByIdx(int8_t* outNullBitMask, int32_t* indices, int8_t* inNullBitMask, int32_t dataElementCount)
{
    if (inNullBitMask != nullptr)
    {
        // Zero the out mask
        GPUMemory::fillArray(outNullBitMask, static_cast<int8_t>(0), dataElementCount);

        // Reorder the bits
        kernel_reorder_null_values_by_idx<<<Context::getInstance().calcGridDim(dataElementCount),
                                            Context::getInstance().getBlockDim()>>>(
            reinterpret_cast<int32_t*>(outNullBitMask), indices, inNullBitMask, dataElementCount);
    }
}

void GPUOrderBy::ReOrderStringByIdx(GPUMemory::GPUString& outCol, int32_t* inIndices, GPUMemory::GPUString inCol, int32_t dataElementCount)
{
    Context& context = Context::getInstance();

    if (dataElementCount > 0)
    {

        cuda_ptr<int32_t> inStringLengths(dataElementCount);
        kernel_lengths_from_indices<int32_t, int64_t>
            <<<context.calcGridDim(dataElementCount), context.getBlockDim()>>>(inStringLengths.get(),
                                                                               inCol.stringIndices,
                                                                               dataElementCount);
        cuda_ptr<int32_t> outStringLengths(dataElementCount);
        kernel_reorder_by_idx<<<context.calcGridDim(dataElementCount), context.getBlockDim()>>>(
            outStringLengths.get(), inIndices, inStringLengths.get(), dataElementCount);
        cuda_ptr<int64_t> outStringIndices(dataElementCount);
        GPUReconstruct::PrefixSum(outStringIndices.get(), outStringLengths.get(), dataElementCount);
        GPUMemory::alloc(&outCol.stringIndices, dataElementCount);

        int64_t totalCharCount;
        GPUMemory::copyDeviceToHost(&totalCharCount, &inCol.stringIndices[dataElementCount - 1], 1);
        GPUMemory::alloc(&outCol.allChars, totalCharCount);

        kernel_reorder_chars_by_idx<<<context.calcGridDim(dataElementCount), context.getBlockDim()>>>(
            outCol, inIndices, inCol, outStringIndices.get(), outStringLengths.get(), dataElementCount);
    }
    else
    {
        outCol.stringIndices = nullptr;
        outCol.allChars = nullptr;
    }
}

void GPUOrderBy::ReOrderPolygonByIdx(GPUMemory::GPUPolygon& outCol,
                                     int32_t* inIndices,
                                     GPUMemory::GPUPolygon inCol,
                                     int32_t dataElementCount)
{
    Context& context = Context::getInstance();

    if (dataElementCount > 0)
    {
        GPUMemory::PrintGpuBuffer("In polygon indices: ", inCol.polyIdx, dataElementCount);
        GPUMemory::PrintGpuBuffer("In polygon count: ", inCol.polyCount, dataElementCount);

        cuda_ptr<int32_t> outPolygonLengths(dataElementCount);
        kernel_reorder_by_idx<<<context.calcGridDim(dataElementCount), context.getBlockDim()>>>(
            outPolygonLengths.get(), inIndices, inCol.polyCount, dataElementCount);
        CheckCudaError(hipGetLastError());

        GPUMemory::PrintGpuBuffer("Reordered polygon lengths: ", outPolygonLengths.get(), dataElementCount);

        cuda_ptr<int32_t> outPolygonIndices(dataElementCount);
        GPUReconstruct::PrefixSumExclusive(outPolygonIndices.get(), outPolygonLengths.get(), dataElementCount);

        GPUMemory::PrintGpuBuffer("Reordered polygon indices: ", outPolygonIndices.get(), dataElementCount);

        cuda_ptr<int32_t> outPolygonCounts(dataElementCount);
        GPUReconstruct::PrefixSum(outPolygonCounts.get(), outPolygonLengths.get(), dataElementCount);

        int32_t totalPolygonCount;
        GPUMemory::copyDeviceToHost(&totalPolygonCount, &outPolygonCounts.get()[dataElementCount - 1], 1);

        std::cout << "Polygon count: " << totalPolygonCount << std::endl;

        GPUMemory::PrintGpuBuffer("In point lenghts: ", inCol.pointCount, totalPolygonCount);

        cuda_ptr<int32_t> outPointLengths(totalPolygonCount);
        kernel_reorder_poly_lengths_by_cp_idx_and_cp_lenghts<<<context.calcGridDim(dataElementCount),
                                                               context.getBlockDim()>>>(
            outPointLengths.get(), inIndices, inCol.pointCount, inCol.polyIdx,
            outPolygonIndices.get(), outPolygonLengths.get(), dataElementCount);
        CheckCudaError(hipGetLastError());

        GPUMemory::PrintGpuBuffer("Reordered point lenghts: ", outPointLengths.get(), totalPolygonCount);

        cuda_ptr<int32_t> outPointIndices(totalPolygonCount);
        GPUReconstruct::PrefixSumExclusive(outPointIndices.get(), outPointLengths.get(), totalPolygonCount);

        GPUMemory::PrintGpuBuffer("Reordered point indices: ", outPointIndices.get(), totalPolygonCount);

        cuda_ptr<int32_t> outPointsCounts(totalPolygonCount);
        GPUReconstruct::PrefixSum(outPointsCounts.get(), outPointLengths.get(), totalPolygonCount);

        int32_t totalPointCount;
        GPUMemory::copyDeviceToHost(&totalPointCount, &outPointsCounts.get()[totalPolygonCount - 1], 1);

        std::cout << "Point count: " << totalPointCount << std::endl;

        GPUMemory::alloc(&outCol.polyCount, dataElementCount);
        GPUMemory::alloc(&outCol.polyIdx, dataElementCount);
        GPUMemory::alloc(&outCol.pointCount, totalPolygonCount);
        GPUMemory::alloc(&outCol.pointIdx, totalPolygonCount);
        GPUMemory::alloc(&outCol.polyPoints, totalPointCount);

        kernel_reorder_points_by_idx<<<context.calcGridDim(dataElementCount), context.getBlockDim()>>>(
            outCol, inIndices, inCol, outPolygonIndices.get(), outPolygonLengths.get(),
            outPointIndices.get(), outPointLengths.get(), dataElementCount);
        CheckCudaError(hipGetLastError());
    }
    else
    {
        outCol.polyPoints = nullptr;
        outCol.pointIdx = nullptr;
        outCol.pointCount = nullptr;
        outCol.polyIdx = nullptr;
        outCol.polyCount = nullptr;
    }
}
