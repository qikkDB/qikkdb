#include "hip/hip_runtime.h"
#include "GPUOrderBy.cuh"
#include "GPUStringUnary.cuh"

// Fill the index buffers with default indices
__global__ void kernel_fill_indices(int32_t* indices, int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        indices[i] = i;
    }
}

__device__ int64_t GetPolygonIndex(int32_t* indices, const int32_t i)
{
    return (i == 0) ? 0 : indices[i - 1];
}

__global__ void kernel_reorder_chars_by_idx(GPUMemory::GPUString outCol,
                                            int32_t* inIndices,
                                            GPUMemory::GPUString inCol,
                                            int64_t* outStringIndices,
                                            int32_t* outStringLengths,
                                            int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        int32_t outColIdx = GetStringIndex(outStringIndices, i);
        int32_t inColIdx = GetStringIndex(inCol.stringIndices, inIndices[i]);
        for (int32_t j = 0; j < outStringLengths[i]; j++)
        {
            outCol.allChars[outColIdx + j] = inCol.allChars[inColIdx + j];
        }
        outCol.stringIndices[i] = outStringIndices[i];
    }
}

__global__ void kernel_reorder_points_by_idx(GPUMemory::GPUPolygon outCol,
                                             int32_t* inIndices,
                                             GPUMemory::GPUPolygon inCol,
                                             int32_t* outPolygonIndices,
                                             int32_t* outPolygonLengths,
                                             int32_t* outPointIndices,
                                             int32_t* outPointLengths,
                                             int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        int32_t outPolygonIdx = GetPolygonIndex(outPolygonIndices, i);
        int32_t inPolygonIdx = GetPolygonIndex(inCol.polyIdx, inIndices[i]);
        for (int32_t j = 0; j < outPolygonLengths[i]; j++)
        {
            int32_t outPointIdx = GetPolygonIndex(outPointIndices, outPolygonIdx);
            int32_t inPointIdx = GetPolygonIndex(inCol.pointIdx, inPolygonIdx);
            for (int32_t k = 0; k < outPointLengths[outPolygonIdx]; k++)
            {
                outCol.polyPoints[outPointIdx + k] = inCol.polyPoints[inPointIdx + k];
            }
            outCol.pointIdx[outPolygonIdx + j] = outPointIndices[inPolygonIdx + j];
            outCol.pointCount[outPolygonIdx + j] = outPointLengths[inPolygonIdx + j];
        }
        outCol.polyIdx[i] = outPolygonIndices[i];
        outCol.polyCount[i] = outPolygonLengths[i];
    }
}

__global__ void kernel_reorder_poly_lengths_by_cp_idx_and_cp_lenghts(int32_t* outPointLengths,
                                                                     int32_t* inOrderIndices,
                                                                     int32_t* inPointLenghts,
                                                                     int32_t* inPolygonIndices,
                                                                     int32_t* outPolygonIndices,
                                                                     int32_t* outPolygonLengths,
                                                                     int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        int32_t outPointIdx = GetPolygonIndex(outPolygonIndices, i);
        int32_t inPointIdx = GetPolygonIndex(inPolygonIndices, inOrderIndices[i]);
        for (int32_t j = 0; j < outPolygonLengths[i]; j++)
        {
            outPointLengths[outPointIdx + j] = inPointLenghts[inPointIdx + j];
        }
    }
}

// Reorder a null column by a given index column
__global__ void
kernel_reorder_null_values_by_idx(int32_t* outNullBitMask, int32_t* inIndices, int8_t* inNullBitMask, int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < dataElementCount; i += stride)
    {
        int32_t nullBit =
            (inNullBitMask[inIndices[i] / (sizeof(int8_t) * 8)] >> (inIndices[i] % (sizeof(int8_t) * 8))) & 1;
        nullBit <<= (i % (sizeof(int32_t) * 8));
        atomicOr(outNullBitMask + (i / (sizeof(int32_t) * 8)), nullBit);
    }
}

GPUOrderBy::GPUOrderBy(int32_t dataElementCount)
{
    GPUMemory::alloc(&indices1, dataElementCount);
    GPUMemory::alloc(&indices2, dataElementCount);

    // Initialize the index buffer
    kernel_fill_indices<<<Context::getInstance().calcGridDim(dataElementCount),
                          Context::getInstance().getBlockDim()>>>(indices1, dataElementCount);
}

GPUOrderBy::~GPUOrderBy()
{
    GPUMemory::free(indices1);
    GPUMemory::free(indices2);
}

void GPUOrderBy::ReOrderNullValuesByIdx(int8_t* outNullBitMask, int32_t* indices, int8_t* inNullBitMask, int32_t dataElementCount)
{
    if (inNullBitMask != nullptr)
    {
        // Zero the out mask
        GPUMemory::fillArray(outNullBitMask, static_cast<int8_t>(0), dataElementCount);

        // Reorder the bits
        kernel_reorder_null_values_by_idx<<<Context::getInstance().calcGridDim(dataElementCount),
                                            Context::getInstance().getBlockDim()>>>(
            reinterpret_cast<int32_t*>(outNullBitMask), indices, inNullBitMask, dataElementCount);
    }
}

void GPUOrderBy::ReOrderStringByIdx(GPUMemory::GPUString& outCol, int32_t* inIndices, GPUMemory::GPUString inCol, int32_t dataElementCount)
{
    Context& context = Context::getInstance();

    if (dataElementCount > 0)
    {

        cuda_ptr<int32_t> inStringLengths(dataElementCount);
        kernel_lengths_from_indices<int32_t, int64_t>
            <<<context.calcGridDim(dataElementCount), context.getBlockDim()>>>(inStringLengths.get(),
                                                                               inCol.stringIndices,
                                                                               dataElementCount);
        cuda_ptr<int32_t> outStringLengths(dataElementCount);
        kernel_reorder_by_idx<<<context.calcGridDim(dataElementCount), context.getBlockDim()>>>(
            outStringLengths.get(), inIndices, inStringLengths.get(), dataElementCount);
        cuda_ptr<int64_t> outStringIndices(dataElementCount);
        GPUReconstruct::PrefixSum(outStringIndices.get(), outStringLengths.get(), dataElementCount);
        GPUMemory::alloc(&outCol.stringIndices, dataElementCount);

        int64_t totalCharCount;
        GPUMemory::copyDeviceToHost(&totalCharCount, &inCol.stringIndices[dataElementCount - 1], 1);
        GPUMemory::alloc(&outCol.allChars, totalCharCount);

        kernel_reorder_chars_by_idx<<<context.calcGridDim(dataElementCount), context.getBlockDim()>>>(
            outCol, inIndices, inCol, outStringIndices.get(), outStringLengths.get(), dataElementCount);
    }
    else
    {
        outCol.stringIndices = nullptr;
        outCol.allChars = nullptr;
    }
}

void GPUOrderBy::ReOrderPolygonByIdx(GPUMemory::GPUPolygon& outCol,
                                     int32_t* inIndices,
                                     GPUMemory::GPUPolygon inCol,
                                     int32_t dataElementCount)
{
    Context& context = Context::getInstance();

    if (dataElementCount > 0)
    {

        cuda_ptr<int32_t> inPolygonLengths(dataElementCount);
        kernel_lengths_from_indices<int32_t, int32_t>
            <<<context.calcGridDim(dataElementCount), context.getBlockDim()>>>(inPolygonLengths.get(),
                                                                               inCol.polyIdx, dataElementCount);

        cuda_ptr<int32_t> outPolygonLengths(dataElementCount);
        kernel_reorder_by_idx<<<context.calcGridDim(dataElementCount), context.getBlockDim()>>>(
            outPolygonLengths.get(), inIndices, inPolygonLengths.get(), dataElementCount);

        cuda_ptr<int32_t> outPolygonIndices(dataElementCount);
        GPUReconstruct::PrefixSum(outPolygonIndices.get(), outPolygonLengths.get(), dataElementCount);

        int32_t totalPolygonCount;
        GPUMemory::copyDeviceToHost(&totalPolygonCount, &inCol.polyIdx[dataElementCount - 1], 1);

        cuda_ptr<int32_t> inPointLengths(totalPolygonCount);
        kernel_lengths_from_indices<int32_t, int32_t>
            <<<context.calcGridDim(totalPolygonCount), context.getBlockDim()>>>(inPointLengths.get(),
                                                                                inCol.pointIdx,
                                                                                totalPolygonCount);

        cuda_ptr<int32_t> outPointLengths(totalPolygonCount);
        kernel_reorder_poly_lengths_by_cp_idx_and_cp_lenghts<<<context.calcGridDim(dataElementCount),
                                                               context.getBlockDim()>>>(
            outPointLengths.get(), inIndices, inPointLengths.get(), inCol.polyIdx,
            outPolygonIndices.get(), outPolygonLengths.get(), dataElementCount);

        cuda_ptr<int32_t> outPointIndices(totalPolygonCount);
        GPUReconstruct::PrefixSum(outPointIndices.get(), outPointLengths.get(), totalPolygonCount);

        int32_t totalPointCount;
        GPUMemory::copyDeviceToHost(&totalPointCount, &outPointIndices.get()[totalPolygonCount - 1], 1);

        GPUMemory::alloc(&outCol.polyCount, dataElementCount);
        GPUMemory::alloc(&outCol.polyIdx, dataElementCount);
        GPUMemory::alloc(&outCol.pointCount, totalPolygonCount);
        GPUMemory::alloc(&outCol.pointIdx, totalPolygonCount);
        GPUMemory::alloc(&outCol.polyPoints, totalPointCount);

        kernel_reorder_points_by_idx<<<context.calcGridDim(dataElementCount), context.getBlockDim()>>>(
            outCol, inIndices, inCol, outPolygonIndices.get(), outPolygonLengths.get(),
            outPointIndices.get(), outPointLengths.get(), dataElementCount);
    }
    else
    {
        outCol.polyPoints = nullptr;
        outCol.pointIdx = nullptr;
        outCol.pointCount = nullptr;
        outCol.polyIdx = nullptr;
        outCol.polyCount = nullptr;
    }
}
