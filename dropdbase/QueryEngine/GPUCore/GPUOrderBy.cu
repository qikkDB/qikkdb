#include "hip/hip_runtime.h"
#include "GPUOrderBy.cuh"

// Fill the index buffers with default indices
__global__ void kernel_fill_indices(int32_t* indices, int32_t dataElementCount)
{
	const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t stride = blockDim.x * gridDim.x;

	for (int32_t i = idx; i < dataElementCount; i += stride)
	{
		indices[i] = i;
	}
}

// Reorder a null column by a given index column
__global__ void kernel_reorder_null_values_by_idx(int32_t* outNullBitMask, int32_t* inIndices, int8_t* inNullBitMask, int32_t dataElementCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
	const int32_t stride = blockDim.x * gridDim.x;

	for (int32_t i = idx; i < dataElementCount; i += stride)
	{
		int32_t nullBit = (inNullBitMask[inIndices[i] / (sizeof(int8_t) * 8)] >> (inIndices[i] % (sizeof(int8_t) * 8))) & 1;
		nullBit <<= (i % (sizeof(int32_t) * 8));
		atomicOr(outNullBitMask + (i / (sizeof(int32_t) * 8)), nullBit);
    }
}

GPUOrderBy::GPUOrderBy(int32_t dataElementCount)
{
	GPUMemory::alloc(&indices1, dataElementCount);
	GPUMemory::alloc(&indices2, dataElementCount);

	// Initialize the index buffer
	kernel_fill_indices << < Context::getInstance().calcGridDim(dataElementCount),
		Context::getInstance().getBlockDim() >> >
		(indices1, dataElementCount);
}

GPUOrderBy::~GPUOrderBy()
{
	GPUMemory::free(indices1);
	GPUMemory::free(indices2);
}

void GPUOrderBy::ReOrderNullValuesByIdx(int8_t* outNullBitMask, int32_t* indices, int8_t* inNullBitMask, int32_t dataElementCount)
{
	if(inNullBitMask != nullptr)
	{
		// Zero the out mask
		GPUMemory::fillArray(outNullBitMask, static_cast<int8_t>(0), dataElementCount);

		// Reorder the bits
		kernel_reorder_null_values_by_idx<<< Context::getInstance().calcGridDim(dataElementCount), 
											 Context::getInstance().getBlockDim() >>> (reinterpret_cast<int32_t*>(outNullBitMask), 
																					indices, 
																					inNullBitMask,
																					dataElementCount);
	}
}
