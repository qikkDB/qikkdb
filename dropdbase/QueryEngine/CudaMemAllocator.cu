#include "hip/hip_runtime.h"
#include "CudaMemAllocator.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cstdio>
#include <stdexcept>

CudaMemAllocator::CudaMemAllocator(int32_t deviceID) :
	deviceID_(deviceID)
{
	int32_t oldDevice;
	hipGetDevice(&oldDevice);
	hipSetDevice(deviceID);

	size_t free, total;
	hipMemGetInfo(&free, &total);
	//printf("Device %d: %s Total: %zu Free: %zu\n", deviceID_, props.name, total, free);
	if (hipMalloc(&cudaBufferStart_, free - RESERVED_MEMORY) != hipSuccess)
	{
		throw std::invalid_argument("Failed to alloc GPU buffer");
	}
	chainedBlocks_.push_back({ false, blocksBySize_.end(), free - RESERVED_MEMORY, cudaBufferStart_ });
	(*chainedBlocks_.begin()).sizeOrderIt = blocksBySize_.emplace(std::make_pair(free - RESERVED_MEMORY, chainedBlocks_.begin()));
#ifdef DEBUG_ALLOC
	logOut = fopen("E:\\alloc.log", "a");
	fprintf(logOut, "CudaMemAllocator %d\n", deviceID);
	fprintf(logOut, "Available blocks: %zu\n", chainedBlocks_.size());
	for (auto & ptrs : chainedBlocks_)
	{
		fprintf(logOut, "%zu bytes at %p\n", ptrs.blockSize, ptrs.ptr);
	}
#endif // DEBUG_ALLOC
	hipSetDevice(oldDevice);
}

CudaMemAllocator::~CudaMemAllocator()
{
	if (cudaBufferStart_ != nullptr)
	{
		int oldDevice;
		hipGetDevice(&oldDevice);
		hipSetDevice(deviceID_);
		hipFree(cudaBufferStart_);
		hipSetDevice(oldDevice);
	}
#ifdef DEBUG_ALLOC
	if (logOut != nullptr)
	{
		fprintf(logOut, "~CudaMemAllocator %d\n", deviceID_);
		fclose(logOut);
	}
#endif // DEBUG_ALLOC
}

int8_t * CudaMemAllocator::allocate(std::ptrdiff_t numBytes)
{
	//Minimal allocation unit is 512bytes, same as hipMalloc. Thurst relies on this internally.
	std::size_t alignedSize = numBytes % 512 == 0 ? numBytes : numBytes + (512 - numBytes % 512);
	auto it = blocksBySize_.lower_bound(alignedSize);
	if (it == blocksBySize_.end())
	{
		throw std::out_of_range("Out of GPU memory");
	}
	auto blockInfoIt = (*it).second;
	(*blockInfoIt).allocated = true;
	if ((*it).first > alignedSize)
	{
		SplitBlock(it, alignedSize);
	}
	blocksBySize_.erase(it);
	allocatedBlocks_.emplace(std::make_pair((*blockInfoIt).ptr, blockInfoIt));
#ifdef DEBUG_ALLOC
	fprintf(logOut,"%d CudaMemAllocator::allocate %p %zu\n", deviceID_, (*blockInfoIt).ptr, alignedSize);
	fflush(logOut);
#endif
	return static_cast<int8_t*>((*blockInfoIt).ptr);
}

void CudaMemAllocator::deallocate(int8_t * ptr, size_t numBytes)
{
#ifdef DEBUG_ALLOC
	fprintf(logOut, "%d CudaMemAllocator::deallocate ptr %p\n", deviceID_, ptr);
	fflush(logOut);
#endif
	auto allocListIt = allocatedBlocks_.find(ptr);
	if (allocListIt == allocatedBlocks_.end())
	{
		return;
	}
	auto listIt = (*allocListIt).second;
	allocatedBlocks_.erase(allocListIt);
	if (listIt != chainedBlocks_.begin())
	{
		auto prevIt = (listIt);
		prevIt--;
		if (!(*prevIt).allocated)
		{
			blocksBySize_.erase((*prevIt).sizeOrderIt);
			(*prevIt).blockSize += (*listIt).blockSize;
			chainedBlocks_.erase(listIt);
			listIt = prevIt;
		}
	}
	auto nextIt = listIt;
	nextIt++;
	if (nextIt != chainedBlocks_.end() && !(*nextIt).allocated)
	{
		blocksBySize_.erase((*nextIt).sizeOrderIt);
		(*listIt).blockSize += (*nextIt).blockSize;
		chainedBlocks_.erase(nextIt);
	}
	(*listIt).allocated = false;
	(*listIt).sizeOrderIt = blocksBySize_.emplace(std::make_pair((*listIt).blockSize, listIt));
#ifdef DEBUG_ALLOC
	fprintf(logOut, "CudaMemAllocator::deallocate final ptr %p %zu\n", (*listIt).ptr, (*listIt).blockSize);
	fflush(logOut);
#endif
}

void CudaMemAllocator::SplitBlock(std::multimap<size_t, std::list<BlockInfo>::iterator>::iterator blockIterator, size_t requestedSize)
{
	auto blockInfoIt = (*blockIterator).second;
	size_t oldSize = (*blockIterator).first;
	void* newFreePtr = static_cast<int8_t*>((*blockInfoIt).ptr) + requestedSize;
	auto nextBlockInfo = blockInfoIt;
	nextBlockInfo++;
	auto listIt = chainedBlocks_.insert(nextBlockInfo, { false, blocksBySize_.end(), oldSize - requestedSize, newFreePtr });
	(*listIt).sizeOrderIt = blocksBySize_.emplace(std::make_pair(oldSize - requestedSize, listIt));
	(*blockInfoIt).blockSize = requestedSize;
}

void CudaMemAllocator::Clear()
{
#ifdef DEBUG_ALLOC
	fprintf(logOut, "---------------\nAllocation statistics for GPU %d:\n",deviceID_);
	fprintf(logOut, "Leaked pointers: %zu\n", allocatedBlocks_.size());
	for (auto & ptrs : allocatedBlocks_)
	{
		fprintf(logOut, "%zu bytes at %p\n", (*ptrs.second).blockSize, ptrs.first);
	}
	fprintf(logOut, "---------------\n");
	fprintf(logOut, "Available blocks: %zu\n", chainedBlocks_.size());
	for (auto & ptrs : chainedBlocks_)
	{
		fprintf(logOut, "%zu bytes at %p\n", ptrs.blockSize, ptrs.ptr);
	}
	fprintf(logOut, "---------------\n");
	fflush(logOut);
#endif
}