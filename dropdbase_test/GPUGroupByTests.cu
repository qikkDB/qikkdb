#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdlib>
#include <memory>

#include "../dropdbase/QueryEngine/Context.h"
#include "../dropdbase/QueryEngine/GPUCore/GPUGroupByString.cuh"
#include "../dropdbase/QueryEngine/GPUCore/GPUGroupByMultiKey.cuh"
#include "../dropdbase/QueryEngine/GPUCore/AggregationFunctions.cuh"
#include "../dropdbase/QueryEngine/GPUCore/GPUMemory.cuh"
#include "../dropdbase/QueryEngine/GPUCore/cuda_ptr.h"
#include "../dropdbase/DataType.h"
#include "../dropdbase/StringFactory.h"
#include "gtest/gtest.h"

template <typename AGG>
void TestGroupByString(std::vector<std::vector<std::string>> keys,
                       std::vector<std::vector<int32_t>> values,
                       std::unordered_map<std::string, int32_t> correctPairs)
{
    constexpr int32_t hashTableSize = 8;
    GPUGroupBy<AGG, int32_t, std::string, int32_t> groupBy(hashTableSize);
    for (int32_t b = 0; b < keys.size(); b++) // per "block"
    {
        // std::cout << "BLOCK " << b << ":" << std::endl;
        int32_t dataElementCount = min(keys[b].size(), values[b].size());
        GPUMemory::GPUString gpuInKeys = StringFactory::PrepareGPUString(keys[b]);
        cuda_ptr<int32_t> gpuInValues(dataElementCount);
        GPUMemory::copyHostToDevice(gpuInValues.get(), values[b].data(), dataElementCount);

        groupBy.groupBy(gpuInKeys, gpuInValues.get(), dataElementCount);
        GPUMemory::free(gpuInKeys);
        /*
        // DEBUG prints
        int32_t sourceIds[hashTableSize];
        int32_t stringLens[hashTableSize];
        GPUMemory::copyDeviceToHost(sourceIds, groupBy.sourceIndices_, hashTableSize);
        GPUMemory::copyDeviceToHost(stringLens, groupBy.stringLengths_, hashTableSize);
        for (int32_t i = 0; i < hashTableSize; i++)
        {
            std::cout << stringLens[i] << "  " << sourceIds[i];
            if (sourceIds[i] >= 0)
            {
                std::cout << " (" << keys[b][sourceIds[i]] << ")";
            }
            std::cout << std::endl;
        }
        */
    }
    GPUMemory::GPUString resultKeysGpu;
    int32_t* resultValuesGpu;
    int32_t resultCount;
    groupBy.getResults(&resultKeysGpu, &resultValuesGpu, &resultCount);
    std::unique_ptr<std::string[]> resultKeys = std::make_unique<std::string[]>(hashTableSize);
    std::unique_ptr<int32_t[]> resultValues = std::make_unique<int32_t[]>(hashTableSize);
    GPUReconstruct::ReconstructStringCol(resultKeys.get(), &resultCount, resultKeysGpu, nullptr, resultCount);
    GPUMemory::copyDeviceToHost(resultValues.get(), resultValuesGpu, resultCount);

    ASSERT_EQ(correctPairs.size(), resultCount) << " wrong number of keys";
    for (int32_t i = 0; i < resultCount; i++)
    {
        ASSERT_FALSE(correctPairs.find(resultKeys[i]) == correctPairs.end())
            << " key \"" << resultKeys[i] << "\"";
        ASSERT_EQ(correctPairs[resultKeys[i]], resultValues[i]) << " at key \"" << resultKeys[i] << "\"";
    }
    GPUMemory::free(resultKeysGpu);
    GPUMemory::free(resultValuesGpu);
}


template <typename AGG>
void TestGroupByMultiKey(std::vector<DataType> keyTypes,
                         std::vector<std::vector<void*>> keys,
                         std::vector<std::vector<int32_t>> values,
                         std::vector<void*> correctKeys,
                         std::vector<int32_t> correctValues)
{
    constexpr int32_t hashTableSize = 8;
    GPUGroupBy<AGG, int32_t, std::vector<void*>, int32_t> groupBy(hashTableSize, keyTypes);
    int32_t keysColCount = keyTypes.size();
    for (int32_t b = 0; b < keys.size(); b++) // per "block"
    {
        // std::cout << "BLOCK " << b << ":" << std::endl;
        int32_t dataElementCount = values[b].size();
        std::vector<void*> gpuInKeys;
        for (int32_t t = 0; t < keysColCount; t++)
        {
            switch (keyTypes[t])
            {
            case DataType::COLUMN_INT:
            {
                int32_t* inKeysSingleCol;
                GPUMemory::alloc(&inKeysSingleCol, dataElementCount);
                GPUMemory::copyHostToDevice(inKeysSingleCol, reinterpret_cast<int32_t*>(keys[b][t]), dataElementCount);
                gpuInKeys.emplace_back(inKeysSingleCol);
                break;
            }
            case DataType::COLUMN_LONG:
            {
                int64_t* inKeysSingleCol;
                GPUMemory::alloc(&inKeysSingleCol, dataElementCount);
                GPUMemory::copyHostToDevice(inKeysSingleCol, reinterpret_cast<int64_t*>(keys[b][t]), dataElementCount);
                gpuInKeys.emplace_back(inKeysSingleCol);
                break;
            }
            case DataType::COLUMN_FLOAT:
            {
                float* inKeysSingleCol;
                GPUMemory::alloc(&inKeysSingleCol, dataElementCount);
                GPUMemory::copyHostToDevice(inKeysSingleCol, reinterpret_cast<float*>(keys[b][t]), dataElementCount);
                gpuInKeys.emplace_back(inKeysSingleCol);
                break;
            }
            case DataType::COLUMN_DOUBLE:
            {
                double* inKeysSingleCol;
                GPUMemory::alloc(&inKeysSingleCol, dataElementCount);
                GPUMemory::copyHostToDevice(inKeysSingleCol, reinterpret_cast<double*>(keys[b][t]), dataElementCount);
                gpuInKeys.emplace_back(inKeysSingleCol);
                break;
            }
            case DataType::COLUMN_STRING:
            {
                GPUMemory::GPUString * inKeysSingleCol;
                GPUMemory::alloc(&inKeysSingleCol, 1);
                std::string * cpuStrArray = reinterpret_cast<std::string*>(keys[b][t]);
                std::vector<std::string> cpuString(cpuStrArray, cpuStrArray + dataElementCount);
                GPUMemory::GPUString cpuStructInKeys = StringFactory::PrepareGPUString(cpuString);
                GPUMemory::copyHostToDevice(inKeysSingleCol, &cpuStructInKeys, 1);
                gpuInKeys.emplace_back(inKeysSingleCol);
                break;
            }
            case DataType::COLUMN_INT8_T:
            {
                int8_t* inKeysSingleCol;
                GPUMemory::alloc(&inKeysSingleCol, dataElementCount);
                GPUMemory::copyHostToDevice(inKeysSingleCol, reinterpret_cast<int8_t*>(keys[b][t]), dataElementCount);
                gpuInKeys.emplace_back(inKeysSingleCol);
                break;
            }
            default:
                break;
            }
        }
        cuda_ptr<int32_t> gpuInValues(dataElementCount);
        GPUMemory::copyHostToDevice(gpuInValues.get(), values[b].data(), dataElementCount);

        groupBy.groupBy(gpuInKeys, gpuInValues.get(), dataElementCount);
        for (int32_t t = 0; t < keysColCount; t++)
        {
            if (keyTypes[t] == DataType::COLUMN_STRING)
            {
                GPUMemory::GPUString cpuStruct;
                GPUMemory::copyDeviceToHost(&cpuStruct, reinterpret_cast<GPUMemory::GPUString*>(gpuInKeys[t]), 1);
                GPUMemory::free(cpuStruct);
            }
            GPUMemory::free(gpuInKeys[t]);
        }
    }
    std::vector<void*> gpuResultKeys;
    int32_t* resultValuesGpu;
    int32_t resultCount;
    groupBy.getResults(&gpuResultKeys, &resultValuesGpu, &resultCount);
    std::vector<void*> cpuResultKeys;
    for (int32_t t = 0; t < keysColCount; t++)
    {
        switch (keyTypes[t])
        {
        case DataType::COLUMN_INT:
        {
            int32_t* outKeysSingleCol = new int32_t[resultCount];
            GPUMemory::copyDeviceToHost(outKeysSingleCol, reinterpret_cast<int32_t*>(gpuResultKeys[t]), resultCount);
            cpuResultKeys.emplace_back(outKeysSingleCol);
            break;
        }
        case DataType::COLUMN_LONG:
        {
            int64_t* outKeysSingleCol = new int64_t[resultCount];
            GPUMemory::copyDeviceToHost(outKeysSingleCol, reinterpret_cast<int64_t*>(gpuResultKeys[t]), resultCount);
            cpuResultKeys.emplace_back(outKeysSingleCol);
            break;
        }
        case DataType::COLUMN_FLOAT:
        {
            float* outKeysSingleCol = new float[resultCount];
            GPUMemory::copyDeviceToHost(outKeysSingleCol, reinterpret_cast<float*>(gpuResultKeys[t]), resultCount);
            cpuResultKeys.emplace_back(outKeysSingleCol);
            break;
        }
        case DataType::COLUMN_DOUBLE:
        {
            double* outKeysSingleCol = new double[resultCount];
            GPUMemory::copyDeviceToHost(outKeysSingleCol, reinterpret_cast<double*>(gpuResultKeys[t]), resultCount);
            cpuResultKeys.emplace_back(outKeysSingleCol);
            break;
        }
        case DataType::COLUMN_STRING:
        {
            std::string * outKeysSingleCol = new std::string[resultCount];
            GPUMemory::GPUString cpuStruct;
            GPUMemory::copyDeviceToHost(&cpuStruct, reinterpret_cast<GPUMemory::GPUString*>(gpuResultKeys[t]), 1);
            GPUReconstruct::ReconstructStringCol(outKeysSingleCol, &resultCount,
                cpuStruct, nullptr, resultCount);
            cpuResultKeys.emplace_back(outKeysSingleCol);
            break;
        }
        case DataType::COLUMN_INT8_T:
        {
            int8_t* outKeysSingleCol = new int8_t[resultCount];
            GPUMemory::copyDeviceToHost(outKeysSingleCol, reinterpret_cast<int8_t*>(gpuResultKeys[t]), resultCount);
            cpuResultKeys.emplace_back(outKeysSingleCol);
            break;
        }
        default:
            break;
        }
    }

    std::unique_ptr<int32_t[]> resultValues = std::make_unique<int32_t[]>(resultCount);
    GPUMemory::copyDeviceToHost(resultValues.get(), resultValuesGpu, resultCount);

    for (int32_t t = 0; t < keysColCount; t++)
    {
        if (keyTypes[t] == DataType::COLUMN_STRING)
        {
            GPUMemory::GPUString cpuStruct;
            GPUMemory::copyDeviceToHost(&cpuStruct, reinterpret_cast<GPUMemory::GPUString*>(gpuResultKeys[t]), 1);
            GPUMemory::free(cpuStruct);
        }
        GPUMemory::free(gpuResultKeys[t]);
    }
    GPUMemory::free(resultValuesGpu);

    ASSERT_EQ(correctValues.size(), resultCount) << " wrong number of keys";
    for (int32_t i = 0; i < resultCount; i++)
    {
        int32_t rowId = -1;
        for (int32_t j = 0; j < resultCount; j++)
        {
            bool equals = true;
            for (int32_t t = 0; t < keysColCount; t++)
            {
                switch (keyTypes[t])
                {
                case DataType::COLUMN_INT:
                    equals &= (reinterpret_cast<int32_t*>(correctKeys[t])[j] == reinterpret_cast<int32_t*>(cpuResultKeys[t])[i]);
                    break;
                case DataType::COLUMN_LONG:
                    equals &= (reinterpret_cast<int64_t*>(correctKeys[t])[j] == reinterpret_cast<int64_t*>(cpuResultKeys[t])[i]);
                    break;
                case DataType::COLUMN_FLOAT:
                    equals &= (reinterpret_cast<float*>(correctKeys[t])[j] == reinterpret_cast<float*>(cpuResultKeys[t])[i]);
                    break;
                case DataType::COLUMN_DOUBLE:
                    equals &= (reinterpret_cast<double*>(correctKeys[t])[j] == reinterpret_cast<double*>(cpuResultKeys[t])[i]);
                    break;
                case DataType::COLUMN_STRING:
                    equals &= (reinterpret_cast<std::string*>(correctKeys[t])[j] == reinterpret_cast<std::string*>(cpuResultKeys[t])[i]);
                    break;
                case DataType::COLUMN_INT8_T:
                    equals &= (reinterpret_cast<int8_t*>(correctKeys[t])[j] == reinterpret_cast<int8_t*>(cpuResultKeys[t])[i]);
                    break;
                default:
                    break;
                }
            }
            if (equals)
            {
                rowId = j;
                break;
            }
        }
        ASSERT_NE(rowId, -1) << " incorrect key";
        ASSERT_EQ(correctValues[rowId], resultValues[i]) << " at correct result row " << rowId;
    }

    for (int32_t t = 0; t < keysColCount; t++)
    {
        delete[] cpuResultKeys[t];
    }
}


TEST(GPUGroupByTests, StringUnique)
{
    TestGroupByString<AggregationFunctions::sum>({{"Apple", "Abcd", "XYZ"}}, {{1, 2, -1}},
                                                 {{"Apple", 1}, {"Abcd", 2}, {"XYZ", -1}});
}

TEST(GPUGroupByTests, StringSimple)
{
    TestGroupByString<AggregationFunctions::sum>({{"Apple", "Abcd", "XYZ", "Abcd", "ZYX", "XYZ"}},
                                                 {{1, 2, -1, 3, 7, 5}},
                                                 {{"Apple", 1}, {"Abcd", 5}, {"XYZ", 4}, {"ZYX", 7}});
}

TEST(GPUGroupByTests, StringMultiBlockSimple)
{
    TestGroupByString<AggregationFunctions::sum>({{"Apple", "Abcd"}, {"XYZ", "Abcd"}, {"ZYX", "XYZ"}, {"Apple", "Apple"}},
                                                 {{1, 1}, {1, 2}, {1, 2}, {2, 4}},
                                                 {{"Apple", 7}, {"Abcd", 3}, {"XYZ", 3}, {"ZYX", 1}});
}

TEST(GPUGroupByTests, StringMultiBlockMediumSum)
{
    TestGroupByString<AggregationFunctions::sum>(
        {{"Apple", "Abcd", "Apple", "XYZ"}, {"Banana", "XYZ", "Abcd", "0"}, {"XYZ", "XYZ"}},
        {{1, 2, 3, 4}, {5, 6, 7, 10}, {13, 15}},
        {{"Apple", 4}, {"Abcd", 9}, {"Banana", 5}, {"XYZ", 38}, {"0", 10}});
}

TEST(GPUGroupByTests, StringMultiBlockMediumMin)
{
    TestGroupByString<AggregationFunctions::min>(
        {{"Apple", "Abcd", "Apple", "XYZ"}, {"Banana", "XYZ", "Abcd", "0"}, {"XYZ", "XYZ"}},
        {{1, 2, 3, 4}, {5, 6, 7, 10}, {13, 15}},
        {{"Apple", 1}, {"Abcd", 2}, {"Banana", 5}, {"XYZ", 4}, {"0", 10}});
}

TEST(GPUGroupByTests, StringMultiBlockMediumMax)
{
    TestGroupByString<AggregationFunctions::max>(
        {{"Apple", "Abcd", "Apple", "XYZ"}, {"Banana", "XYZ", "Abcd", "0"}, {"XYZ", "XYZ"}},
        {{1, 2, 3, 4}, {5, 6, 7, 10}, {13, 15}},
        {{"Apple", 3}, {"Abcd", 7}, {"Banana", 5}, {"XYZ", 15}, {"0", 10}});
}


TEST(GPUGroupByTests, MultiKeyUnique)
{
    int32_t colA[] = {1, 1, 1, 1, 2, 3, 4, 0};
    int32_t colB[] = {1, 2, 3, 4, 1, 1, 1, 0};
    int32_t correctKeysA[] = {1, 1, 1, 1, 2, 3, 4, 0};
    int32_t correctKeysB[] = {1, 2, 3, 4, 1, 1, 1, 0};
    TestGroupByMultiKey<AggregationFunctions::sum>(
        { DataType::COLUMN_INT, DataType::COLUMN_INT },
        { { colA, colB } },
        { { 1, 1, 1, 1, 1, 1, 1, 1 } },
        { correctKeysA, correctKeysB },
        { 1, 1, 1, 1, 1, 1, 1, 1 }
    );
}

TEST(GPUGroupByTests, MultiKeySimple)
{
    int32_t colA[] = {1, 1, 1, 2, 1, 1, 2, 2};
    int32_t colB[] = {1, 2, 3, 4, 1, 1, -1, -1};
    int32_t correctKeysA[] = {1, 1, 1, 2, 2};
    int32_t correctKeysB[] = {1, 2, 3, 4, -1};
    TestGroupByMultiKey<AggregationFunctions::sum>(
        { DataType::COLUMN_INT, DataType::COLUMN_INT },
        { { colA, colB } },
        { { 1, 1, 1, 1, 1, 1, 1, 1 } },
        { correctKeysA, correctKeysB },
        { 3, 1, 1, 1, 2 }
    );
}

TEST(GPUGroupByTests, MultiKeyStringSimple)
{
    int32_t colA[] = { 5, 2, 2, 2, 2, 5, 1, 7 };
    int32_t colB[] = { 1, 1, 1, 1, 1, 1, 2, 0 };
    std::string colC[] = { "Apple", "Nut", "Nut", "Apple", "XYZ", "Apple", "Apple", "Nut" };
    int32_t correctKeysA[] = { 2, 2, 1, 7, 5, 2 };
    int32_t correctKeysB[] = { 1, 1, 2, 0, 1, 1 };
    std::string correctKeysC[] = { "Apple", "XYZ", "Apple", "Nut", "Apple", "Nut" };
    TestGroupByMultiKey<AggregationFunctions::sum>(
        { DataType::COLUMN_INT, DataType::COLUMN_INT, DataType::COLUMN_STRING },
        { { colA, colB, colC } },
        { { 1, 1, 1, 1, 1, 1, 1, 1 } },
        { correctKeysA, correctKeysB, correctKeysC },
        { 1, 1, 1, 1, 2, 2 }
    );
}
