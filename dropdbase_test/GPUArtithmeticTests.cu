#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <memory>
#include <random>

#include "../dropdbase/QueryEngine/Context.h"
#include "../dropdbase/QueryEngine/GPUCore/GPUArithmetic.cuh"
#include "../dropdbase/QueryEngine/GPUCore/GPUMemory.cuh"
#include "../dropdbase/QueryEngine/GPUCore/cuda_ptr.h"
#include "../dropdbase/QueryEngine/NullConstants.cuh"
#include "gtest/gtest.h"


// Count of the testing data elements:
const int32_t DATA_ELEMENT_COUNT = 1 << 18;

template <typename T>
void testColColArithmetic()
{
    // CPU data:
    std::unique_ptr<T[]> inputDataA = std::make_unique<T[]>(DATA_ELEMENT_COUNT);
    std::unique_ptr<T[]> inputDataB = std::make_unique<T[]>(DATA_ELEMENT_COUNT);
    std::unique_ptr<T[]> outputData = std::make_unique<T[]>(DATA_ELEMENT_COUNT);

    // Fill input data buffers:
    std::default_random_engine generator;
    if (std::is_integral<T>::value)
    {
        std::uniform_int_distribution<int32_t> distributionInt(-1024, 1024);
        for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
        {
            inputDataA[i] = distributionInt(generator);
            inputDataB[i] = distributionInt(generator);
            if (inputDataB[i] == 0)
            {
                inputDataB[i] = 1;
            }
        }
    }
    else
    {
        std::uniform_real_distribution<float> distributionFloat(-1024.0f, 1024.0f);
        for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
        {
            inputDataA[i] = distributionFloat(generator);
            inputDataB[i] = distributionFloat(generator);
            if (inputDataB[i] == 0)
            {
                inputDataB[i] = 1;
            }
        }
    }


    // Create CUDA buffers:
    T* inputBufferA;
    T* inputBufferB;
    T* outputBuffer;

    // Alloc buffers in GPU memory:
    GPUMemory::alloc(&inputBufferA, DATA_ELEMENT_COUNT);
    GPUMemory::alloc(&inputBufferB, DATA_ELEMENT_COUNT);
    GPUMemory::alloc(&outputBuffer, DATA_ELEMENT_COUNT);

    // Copy the contents of the buffers to the GPU
    GPUMemory::copyHostToDevice(inputBufferA, inputDataA.get(), DATA_ELEMENT_COUNT);
    GPUMemory::copyHostToDevice(inputBufferB, inputDataB.get(), DATA_ELEMENT_COUNT);

    //////////////////////////////////////////////////////////////////////////////////////
    // Run kernels, copy back values and compare them

    // Add
    GPUArithmetic::Arithmetic<ArithmeticOperations::add>(outputBuffer, inputBufferA, inputBufferB, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        if (std::is_integral<T>::value)
        {
            ASSERT_EQ(outputData[i], inputDataA[i] + inputDataB[i]);
        }
        else
        {
            ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] + inputDataB[i]);
        }
    }

    // Sub
    GPUArithmetic::Arithmetic<ArithmeticOperations::sub>(outputBuffer, inputBufferA, inputBufferB, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        if (std::is_integral<T>::value)
        {
            ASSERT_EQ(outputData[i], inputDataA[i] - inputDataB[i]);
        }
        else
        {
            ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] - inputDataB[i]);
        }
    }

    // Mul
    GPUArithmetic::Arithmetic<ArithmeticOperations::mul>(outputBuffer, inputBufferA, inputBufferB, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        if (std::is_integral<T>::value)
        {
            ASSERT_EQ(outputData[i], inputDataA[i] * inputDataB[i]);
        }
        else
        {
            ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] * inputDataB[i]);
        }
    }

    // Div
    GPUArithmetic::Arithmetic<ArithmeticOperations::div>(outputBuffer, inputBufferA, inputBufferB, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        if (std::is_integral<T>::value)
        {
            if (inputDataB[i] != 0)
            {
                ASSERT_EQ(outputData[i], static_cast<T>(inputDataA[i] / inputDataB[i]));
            }
        }
        else
        {
            if (inputDataB[i] != 0)
            {
                ASSERT_FLOAT_EQ(outputData[i], static_cast<T>(inputDataA[i] / inputDataB[i]));
            }
        }
    }

    // Modulus
    if (std::is_integral<T>::value)
    {
        GPUArithmetic::Arithmetic<ArithmeticOperations::mod>(outputBuffer, inputBufferA,
                                                             inputBufferB, DATA_ELEMENT_COUNT);
        GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
        for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
        {
            if (inputDataB[i] != 0)
            {
                ASSERT_EQ(outputData[i], inputDataA[i] % inputDataB[i]);
            }
        }
    }

    //////////////////////////////////////////////////////////////////////////////////////

    // Free buffers in GPU memory:
    GPUMemory::free(inputBufferA);
    GPUMemory::free(inputBufferB);
    GPUMemory::free(outputBuffer);
}

template <>
void testColColArithmetic<float>()
{
    // CPU data:
    std::unique_ptr<float[]> inputDataA = std::make_unique<float[]>(DATA_ELEMENT_COUNT);
    std::unique_ptr<float[]> inputDataB = std::make_unique<float[]>(DATA_ELEMENT_COUNT);
    std::unique_ptr<float[]> outputData = std::make_unique<float[]>(DATA_ELEMENT_COUNT);

    // Fill input data buffers:
    std::default_random_engine generator;
    std::uniform_real_distribution<float> distributionFloat(-1024.0f, 1024.0f);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        inputDataA[i] = distributionFloat(generator);
        inputDataB[i] = distributionFloat(generator);
        if (inputDataB[i] == 0)
        {
            inputDataB[i] = 1;
        }
    }

    // Create CUDA buffers:
    float* inputBufferA;
    float* inputBufferB;
    float* outputBuffer;

    // Alloc buffers in GPU memory:
    GPUMemory::alloc(&inputBufferA, DATA_ELEMENT_COUNT);
    GPUMemory::alloc(&inputBufferB, DATA_ELEMENT_COUNT);
    GPUMemory::alloc(&outputBuffer, DATA_ELEMENT_COUNT);

    // Copy the contents of the buffers to the GPU
    GPUMemory::copyHostToDevice(inputBufferA, inputDataA.get(), DATA_ELEMENT_COUNT);
    GPUMemory::copyHostToDevice(inputBufferB, inputDataB.get(), DATA_ELEMENT_COUNT);

    //////////////////////////////////////////////////////////////////////////////////////
    // Run kernels, copy back values and compare them

    // Add
    GPUArithmetic::Arithmetic<ArithmeticOperations::add>(outputBuffer, inputBufferA, inputBufferB, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] + inputDataB[i]);
    }

    // Sub
    GPUArithmetic::Arithmetic<ArithmeticOperations::sub>(outputBuffer, inputBufferA, inputBufferB, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] - inputDataB[i]);
    }

    // Mul
    GPUArithmetic::Arithmetic<ArithmeticOperations::mul>(outputBuffer, inputBufferA, inputBufferB, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] * inputDataB[i]);
    }

    // Div
    GPUArithmetic::Arithmetic<ArithmeticOperations::div>(outputBuffer, inputBufferA, inputBufferB, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        if (inputDataB[i] != 0)
        {
            ASSERT_FLOAT_EQ(outputData[i], static_cast<float>(inputDataA[i] / inputDataB[i]));
        }
    }

    //////////////////////////////////////////////////////////////////////////////////////

    // Free buffers in GPU memory:
    GPUMemory::free(inputBufferA);
    GPUMemory::free(inputBufferB);
    GPUMemory::free(outputBuffer);
}

TEST(GPUArithmeticTests, ArithmeticsColCol)
{
    // Initialize CUDA context:
    Context::getInstance();

    testColColArithmetic<int32_t>();
    testColColArithmetic<int64_t>();
    testColColArithmetic<float>();
}

////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename T>
void testColConstArithmetic()
{
    // CPU data:
    std::unique_ptr<T[]> inputDataA = std::make_unique<T[]>(DATA_ELEMENT_COUNT);
    T inputDataBConst;
    std::unique_ptr<T[]> outputData = std::make_unique<T[]>(DATA_ELEMENT_COUNT);

    // Fill input data buffers:
    std::default_random_engine generator;
    if (std::is_integral<T>::value)
    {
        std::uniform_int_distribution<int32_t> distributionInt(-1024, 1024);
        for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
        {
            inputDataA[i] = distributionInt(generator);
        }
        inputDataBConst = distributionInt(generator);
        if (inputDataBConst == 0)
        {
            inputDataBConst = 1;
        }
    }
    else
    {
        std::uniform_real_distribution<float> distributionFloat(-1024.0f, 1024.0f);
        for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
        {
            inputDataA[i] = distributionFloat(generator);
        }
        inputDataBConst = distributionFloat(generator);
        if (inputDataBConst == 0)
        {
            inputDataBConst = 1;
        }
    }


    // Create CUDA buffers:
    T* inputBufferA;
    T* outputBuffer;

    // Alloc buffers in GPU memory:
    GPUMemory::alloc(&inputBufferA, DATA_ELEMENT_COUNT);
    GPUMemory::alloc(&outputBuffer, DATA_ELEMENT_COUNT);

    // Copy the contents of the buffers to the GPU
    GPUMemory::copyHostToDevice(inputBufferA, inputDataA.get(), DATA_ELEMENT_COUNT);

    //////////////////////////////////////////////////////////////////////////////////////
    // Run kernels, copy back values and compare them

    // Add
    GPUArithmetic::Arithmetic<ArithmeticOperations::add>(outputBuffer, inputBufferA,
                                                         inputDataBConst, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        if (std::is_integral<T>::value)
        {
            ASSERT_EQ(outputData[i], inputDataA[i] + inputDataBConst);
        }
        else
        {
            ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] + inputDataBConst);
        }
    }

    // Sub
    GPUArithmetic::Arithmetic<ArithmeticOperations::sub>(outputBuffer, inputBufferA,
                                                         inputDataBConst, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        if (std::is_integral<T>::value)
        {
            ASSERT_EQ(outputData[i], inputDataA[i] - inputDataBConst);
        }
        else
        {
            ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] - inputDataBConst);
        }
    }

    // Mul
    GPUArithmetic::Arithmetic<ArithmeticOperations::mul>(outputBuffer, inputBufferA,
                                                         inputDataBConst, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        if (std::is_integral<T>::value)
        {
            ASSERT_EQ(outputData[i], inputDataA[i] * inputDataBConst);
        }
        else
        {
            ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] * inputDataBConst);
        }
    }

    // Div
    GPUArithmetic::Arithmetic<ArithmeticOperations::div>(outputBuffer, inputBufferA,
                                                         inputDataBConst, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        if (std::is_integral<T>::value)
        {
            if (inputDataBConst != 0)
            {
                ASSERT_EQ(outputData[i], static_cast<T>(inputDataA[i] / inputDataBConst));
            }
        }
        else
        {
            if (inputDataBConst != 0)
            {
                ASSERT_FLOAT_EQ(outputData[i], static_cast<T>(inputDataA[i] / inputDataBConst));
            }
        }
    }

    // Modulus
    if (std::is_integral<T>::value)
    {
        GPUArithmetic::Arithmetic<ArithmeticOperations::mod>(outputBuffer, inputBufferA,
                                                             inputDataBConst, DATA_ELEMENT_COUNT);
        GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
        for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
        {
            if (inputDataBConst != 0)
            {
                ASSERT_EQ(outputData[i], inputDataA[i] % inputDataBConst);
            }
        }
    }

    //////////////////////////////////////////////////////////////////////////////////////

    // Free buffers in GPU memory:
    GPUMemory::free(inputBufferA);
    GPUMemory::free(outputBuffer);
}

template <>
void testColConstArithmetic<float>()
{
    // CPU data:
    std::unique_ptr<float[]> inputDataA = std::make_unique<float[]>(DATA_ELEMENT_COUNT);
    float inputDataBConst;
    std::unique_ptr<float[]> outputData = std::make_unique<float[]>(DATA_ELEMENT_COUNT);

    // Fill input data buffers:
    std::default_random_engine generator;
    if (std::is_integral<float>::value)
    {
        std::uniform_int_distribution<int32_t> distributionInt(-1024, 1024);
        for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
        {
            inputDataA[i] = distributionInt(generator);
        }
        inputDataBConst = distributionInt(generator);
    }
    else
    {
        std::uniform_real_distribution<float> distributionFloat(-1024.0f, 1024.0f);
        for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
        {
            inputDataA[i] = distributionFloat(generator);
        }
        inputDataBConst = distributionFloat(generator);
    }


    // Create CUDA buffers:
    float* inputBufferA;
    float* outputBuffer;

    // Alloc buffers in GPU memory:
    GPUMemory::alloc(&inputBufferA, DATA_ELEMENT_COUNT);
    GPUMemory::alloc(&outputBuffer, DATA_ELEMENT_COUNT);

    // Copy the contents of the buffers to the GPU
    GPUMemory::copyHostToDevice(inputBufferA, inputDataA.get(), DATA_ELEMENT_COUNT);

    //////////////////////////////////////////////////////////////////////////////////////
    // Run kernels, copy back values and compare them

    // Add
    GPUArithmetic::Arithmetic<ArithmeticOperations::add>(outputBuffer, inputBufferA,
                                                         inputDataBConst, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] + inputDataBConst);
    }

    // Sub
    GPUArithmetic::Arithmetic<ArithmeticOperations::sub>(outputBuffer, inputBufferA,
                                                         inputDataBConst, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] - inputDataBConst);
    }

    // Mul
    GPUArithmetic::Arithmetic<ArithmeticOperations::mul>(outputBuffer, inputBufferA,
                                                         inputDataBConst, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] * inputDataBConst);
    }

    // Div
    GPUArithmetic::Arithmetic<ArithmeticOperations::div>(outputBuffer, inputBufferA,
                                                         inputDataBConst, DATA_ELEMENT_COUNT);
    GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
    for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
    {
        if (inputDataBConst != 0)
        {
            ASSERT_FLOAT_EQ(outputData[i], static_cast<float>(inputDataA[i] / inputDataBConst));
        }
    }

    //////////////////////////////////////////////////////////////////////////////////////

    // Free buffers in GPU memory:
    GPUMemory::free(inputBufferA);
    GPUMemory::free(outputBuffer);
}

TEST(GPUArithmeticTests, ArithmeticsColConst)
{
    // Initialize CUDA context:
    Context::getInstance();

    testColConstArithmetic<int32_t>();
    testColConstArithmetic<int64_t>();
    testColConstArithmetic<float>();
}

////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename OP, typename T>
void testErrors(std::vector<T> inputDataA, std::vector<T> inputDataB, std::vector<T> correctResult, QueryEngineErrorType errorType)
{
    int32_t size = std::min(inputDataA.size(), inputDataB.size());
    ASSERT_EQ(correctResult.size(), size) << "Input data and result sizes are not equal";
    std::unique_ptr<T[]> hostResult = std::make_unique<T[]>(size);
    cuda_ptr<T> deviceDataA(size);
    cuda_ptr<T> deviceDataB(size);
    cuda_ptr<T> deviceResult(size);
    GPUMemory::copyHostToDevice(deviceDataA.get(), inputDataA.data(), size);
    GPUMemory::copyHostToDevice(deviceDataB.get(), inputDataB.data(), size);

    EXPECT_THROW(
        {
            try
            {
                GPUArithmetic::Arithmetic<OP>(deviceResult.get(), deviceDataA.get(), deviceDataB.get(), size);
            }
            catch (query_engine_error& err)
            {
                EXPECT_EQ(err.GetQueryEngineError(), errorType)
                    << "Query Engine Error that shouldn't be thrown";
                throw;
            }
        },
        query_engine_error);

    GPUMemory::copyDeviceToHost(hostResult.get(), deviceResult.get(), size);
    for (int i = 0; i < size; i++)
    {
        ASSERT_EQ(hostResult[i], correctResult[i]) << "Value at [" << i << "] does not match";
    }
}

TEST(GPUArithmeticTests, DivisionByZero)
{
    // Initialize CUDA context:
    Context::getInstance();

    testErrors<ArithmeticOperations::div, int32_t>({1, 8, 1, -1, 0, -1}, {1, 4, 0, 0, 0, 1},
                                                   {1, 2, GetNullConstant<int32_t>(), GetNullConstant<int32_t>(),
                                                    GetNullConstant<int32_t>(), -1},
                                                   QueryEngineErrorType::GPU_DIVISION_BY_ZERO_ERROR);
}

TEST(GPUArithmeticTests, AddOverflow)
{
    // Initialize CUDA context:
    Context::getInstance();

    testErrors<ArithmeticOperations::add, int32_t>({1, 2147483646, -2147483646, -1, 0, -1}, {1, 10, -10, -1},
                                                   {2, GetNullConstant<int32_t>(),
                                                    GetNullConstant<int32_t>(), -2},
                                                   QueryEngineErrorType::GPU_INTEGER_OVERFLOW_ERROR);
}

TEST(GPUArithmeticTests, MulOverflow)
{
    // Initialize CUDA context:
    Context::getInstance();

    testErrors<ArithmeticOperations::mul, int32_t>({1, 65536, -65536, -1, 0, -1}, {1, 65536, 65536, -1},
                                                   {1, GetNullConstant<int32_t>(),
                                                    GetNullConstant<int32_t>(), 1},
                                                   QueryEngineErrorType::GPU_INTEGER_OVERFLOW_ERROR);
}
