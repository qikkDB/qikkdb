#include "hip/hip_runtime.h"
#include <memory>
#include <cstdlib>
#include <cstdio>
#include <random>

#include "gtest/gtest.h"
#include "../dropdbase/QueryEngine/Context.h"
#include "../dropdbase/QueryEngine/GPUCore/GPUMemory.cuh"
#include "../dropdbase/QueryEngine/GPUCore/GPUFilter.cuh"

// Initialize random generators with a seed
const int32_t SEED = 42;

// Count of the testing data elements:
const int32_t DATA_ELEMENT_COUNT = 1 << 18;


template<typename T>
void testColCol()
{
	// CPU data:
	std::unique_ptr<T[]> inputDataA = std::make_unique<T[]>(DATA_ELEMENT_COUNT);
	std::unique_ptr<T[]> inputDataB = std::make_unique<T[]>(DATA_ELEMENT_COUNT);
	std::unique_ptr<int8_t[]> outputData = std::make_unique<int8_t[]>(DATA_ELEMENT_COUNT);

	// Fill input data buffers:
	std::default_random_engine generator;
	if (std::is_integral<T>::value)
	{
		std::uniform_int_distribution<int32_t> distributionInt(std::numeric_limits<int32_t>::lowest(), std::numeric_limits<int32_t>::max());
		for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
		{
			inputDataA[i] = distributionInt(generator);
			inputDataB[i] = distributionInt(generator);
		}
	}
	else
	{
		std::uniform_real_distribution<float> distributionFloat(std::numeric_limits<float>::lowest(), std::numeric_limits<float>::max());
		for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
		{
			inputDataA[i] = distributionFloat(generator);
			inputDataB[i] = distributionFloat(generator);
		}
	}


	// Create CUDA buffers:
	T *inputBufferA;
	T *inputBufferB;
	int8_t *outputBuffer;

	// Alloc buffers in GPU memory:
	GPUMemory::alloc(&inputBufferA, DATA_ELEMENT_COUNT);
	GPUMemory::alloc(&inputBufferB, DATA_ELEMENT_COUNT);
	GPUMemory::alloc(&outputBuffer, DATA_ELEMENT_COUNT);

	// Copy the contents of the buffers to the GPU
	GPUMemory::copyHostToDevice(inputBufferA, inputDataA.get(), DATA_ELEMENT_COUNT);
	GPUMemory::copyHostToDevice(inputBufferB, inputDataB.get(), DATA_ELEMENT_COUNT);

	//////////////////////////////////////////////////////////////////////////////////////
	// Run kernels, copy back values and compare them

	// Greater than
	GPUFilter::colCol<FilterConditions::greater>(outputBuffer, inputBufferA, inputBufferB, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataA[i] > inputDataB[i]);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] > inputDataB[i]);
		}
	}

	// Greater than equal
	GPUFilter::colCol<FilterConditions::greaterEqual>(outputBuffer, inputBufferA, inputBufferB, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataA[i] >= inputDataB[i]);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] >= inputDataB[i]);
		}
	}

	// Less than
	GPUFilter::colCol<FilterConditions::less>(outputBuffer, inputBufferA, inputBufferB, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataA[i] < inputDataB[i]);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] < inputDataB[i]);
		}
	}

	// Less than equal
	GPUFilter::colCol<FilterConditions::lessEqual>(outputBuffer, inputBufferA, inputBufferB, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataA[i] <= inputDataB[i]);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] <= inputDataB[i]);
		}
	}

	// Equal
	GPUFilter::colCol<FilterConditions::equal>(outputBuffer, inputBufferA, inputBufferB, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataA[i] == inputDataB[i]);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] == inputDataB[i]);
		}
	}

	// Non equal
	GPUFilter::colCol<FilterConditions::notEqual>(outputBuffer, inputBufferA, inputBufferB, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataA[i] != inputDataB[i]);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] != inputDataB[i]);
		}
	}

	//////////////////////////////////////////////////////////////////////////////////////

	// Free buffers in GPU memory:
	GPUMemory::free(inputBufferA);
	GPUMemory::free(inputBufferB);
	GPUMemory::free(outputBuffer);
}


TEST(GPUFilterTests, FiltersColCol)
{
	// Initialize CUDA context:
	Context::getInstance();

	testColCol<int32_t>();
	testColCol<int64_t>();
	testColCol<float>();
}

//////////////////////////////////////////////////////////////////////////////////////

template<typename T>
void testColConst()
{
	// CPU data:
	std::unique_ptr<T[]> inputDataA = std::make_unique<T[]>(DATA_ELEMENT_COUNT);
	T inputDataBConstant;
	std::unique_ptr<int8_t[]> outputData = std::make_unique<int8_t[]>(DATA_ELEMENT_COUNT);

	// Fill input data buffers:
	std::default_random_engine generator;
	if (std::is_integral<T>::value)
	{
		std::uniform_int_distribution<int32_t> distributionInt(std::numeric_limits<int32_t>::lowest(), std::numeric_limits<int32_t>::max());
		for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
		{
			inputDataA[i] = distributionInt(generator);
		}
		inputDataBConstant = distributionInt(generator);
	}
	else
	{
		std::uniform_real_distribution<float> distributionFloat(std::numeric_limits<float>::lowest(), std::numeric_limits<float>::max());
		for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
		{
			inputDataA[i] = distributionFloat(generator);
		}
		inputDataBConstant = distributionFloat(generator);
	}


	// Create CUDA buffers:
	T *inputBufferA;
	int8_t *outputBuffer;

	// Alloc buffers in GPU memory:
	GPUMemory::alloc(&inputBufferA, DATA_ELEMENT_COUNT);
	GPUMemory::alloc(&outputBuffer, DATA_ELEMENT_COUNT);

	// Copy the contents of the buffers to the GPU
	GPUMemory::copyHostToDevice(inputBufferA, inputDataA.get(), DATA_ELEMENT_COUNT);

	//////////////////////////////////////////////////////////////////////////////////////
	// Run kernels, copy back values and compare them

	// Greater than
	GPUFilter::colConst<FilterConditions::greater>(outputBuffer, inputBufferA, inputDataBConstant, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataA[i] > inputDataBConstant);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] > inputDataBConstant);
		}
	}

	// Greater than equal
	GPUFilter::colConst<FilterConditions::greaterEqual>(outputBuffer, inputBufferA, inputDataBConstant, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataA[i] >= inputDataBConstant);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] >= inputDataBConstant);
		}
	}

	// Less than
	GPUFilter::colConst<FilterConditions::less>(outputBuffer, inputBufferA, inputDataBConstant, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataA[i] < inputDataBConstant);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] < inputDataBConstant);
		}
	}

	// Less than equal
	GPUFilter::colConst<FilterConditions::lessEqual>(outputBuffer, inputBufferA, inputDataBConstant, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataA[i] <= inputDataBConstant);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] <= inputDataBConstant);
		}
	}

	// Equal
	GPUFilter::colConst<FilterConditions::equal>(outputBuffer, inputBufferA, inputDataBConstant, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataA[i] == inputDataBConstant);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] == inputDataBConstant);
		}
	}

	// Non equal
	GPUFilter::colConst<FilterConditions::notEqual>(outputBuffer, inputBufferA, inputDataBConstant, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataA[i] != inputDataBConstant);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] != inputDataBConstant);
		}
	}

	//////////////////////////////////////////////////////////////////////////////////////

	// Free buffers in GPU memory:
	GPUMemory::free(inputBufferA);
	GPUMemory::free(outputBuffer);
}

TEST(GPUFilterTests, FiltersColConst)
{
	// Initialize CUDA context:
	Context::getInstance();

	testColConst<int32_t>();
	testColConst<int64_t>();
	testColConst<float>();
}

//////////////////////////////////////////////////////////////////////////////////////

template<typename T>
void testConstCol()
{
	// CPU data:
	std::unique_ptr<T[]> inputDataA = std::make_unique<T[]>(DATA_ELEMENT_COUNT);
	T inputDataBConstant;
	std::unique_ptr<int8_t[]> outputData = std::make_unique<int8_t[]>(DATA_ELEMENT_COUNT);

	// Fill input data buffers:
	std::default_random_engine generator;
	if (std::is_integral<T>::value)
	{
		std::uniform_int_distribution<int32_t> distributionInt(std::numeric_limits<int32_t>::lowest(), std::numeric_limits<int32_t>::max());
		for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
		{
			inputDataA[i] = distributionInt(generator);
		}
		inputDataBConstant = distributionInt(generator);
	}
	else
	{
		std::uniform_real_distribution<float> distributionFloat(std::numeric_limits<float>::lowest(), std::numeric_limits<float>::max());
		for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
		{
			inputDataA[i] = distributionFloat(generator);
		}
		inputDataBConstant = distributionFloat(generator);
	}


	// Create CUDA buffers:
	T *inputBufferA;
	int8_t *outputBuffer;

	// Alloc buffers in GPU memory:
	GPUMemory::alloc(&inputBufferA, DATA_ELEMENT_COUNT);
	GPUMemory::alloc(&outputBuffer, DATA_ELEMENT_COUNT);

	// Copy the contents of the buffers to the GPU
	GPUMemory::copyHostToDevice(inputBufferA, inputDataA.get(), DATA_ELEMENT_COUNT);

	//////////////////////////////////////////////////////////////////////////////////////
	// Run kernels, copy back values and compare them

	// Greater than
	GPUFilter::constCol<FilterConditions::greater>(outputBuffer, inputDataBConstant, inputBufferA, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataBConstant > inputDataA[i]);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataBConstant > inputDataA[i]);
		}
	}

	// Greater than equal
	GPUFilter::constCol<FilterConditions::greaterEqual>(outputBuffer, inputDataBConstant, inputBufferA, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataBConstant >= inputDataA[i]);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataBConstant >= inputDataA[i]);
		}
	}

	// Less than
	GPUFilter::constCol<FilterConditions::less>(outputBuffer, inputDataBConstant, inputBufferA, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataBConstant < inputDataA[i]);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataBConstant < inputDataA[i]);
		}
	}

	// Less than equal
	GPUFilter::constCol<FilterConditions::lessEqual>(outputBuffer, inputDataBConstant, inputBufferA, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataBConstant <= inputDataA[i]);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataBConstant <= inputDataA[i]);
		}
	}

	// Equal
	GPUFilter::constCol<FilterConditions::equal>(outputBuffer, inputDataBConstant, inputBufferA, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataA[i] == inputDataBConstant);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] == inputDataBConstant);
		}
	}

	// Non equal
	GPUFilter::constCol<FilterConditions::notEqual>(outputBuffer, inputDataBConstant, inputBufferA, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(outputData.get(), outputBuffer, DATA_ELEMENT_COUNT);
	for (int i = 0; i < DATA_ELEMENT_COUNT; i++)
	{
		if (std::is_integral<T>::value)
		{
			ASSERT_EQ(outputData[i], inputDataA[i] != inputDataBConstant);
		}
		else
		{
			ASSERT_FLOAT_EQ(outputData[i], inputDataA[i] != inputDataBConstant);
		}
	}

	//////////////////////////////////////////////////////////////////////////////////////

	// Free buffers in GPU memory:
	GPUMemory::free(inputBufferA);
	GPUMemory::free(outputBuffer);
}

TEST(GPUFilterTests, FiltersConstCol)
{
	// Initialize CUDA context:
	Context::getInstance();

	testConstCol<int32_t>();
	testConstCol<int64_t>();
	testConstCol<float>();
}


//////////////////////////////////////////////////////////////////////////////////////

template<typename T>
void testConstConst()
{
	// CPU data:
	T inputDataAConstant;
	T inputDataBConstant;
	int8_t outputData;

	// Fill input data buffers:
	std::default_random_engine generator;
	if (std::is_integral<T>::value)
	{
		std::uniform_int_distribution<int32_t> distributionInt(std::numeric_limits<int32_t>::lowest(), std::numeric_limits<int32_t>::max());

		inputDataAConstant = distributionInt(generator);
		inputDataBConstant = distributionInt(generator);
	}
	else
	{
		std::uniform_real_distribution<float> distributionFloat(std::numeric_limits<float>::lowest(), std::numeric_limits<float>::max());

		inputDataAConstant = distributionFloat(generator);
		inputDataBConstant = distributionFloat(generator);
	}

	// Create CUDA buffers:
	int8_t *outputBuffer;

	// Alloc buffers in GPU memory:
	GPUMemory::alloc(&outputBuffer, DATA_ELEMENT_COUNT);

	//////////////////////////////////////////////////////////////////////////////////////
	// Run kernels, copy back values and compare them

	// Greater than
	GPUFilter::constConst<FilterConditions::greater>(outputBuffer, inputDataAConstant, inputDataBConstant, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(&outputData, outputBuffer, DATA_ELEMENT_COUNT);

	if (std::is_integral<T>::value)
	{
		ASSERT_EQ(outputData, inputDataAConstant > inputDataBConstant);
	}
	else
	{
		ASSERT_FLOAT_EQ(outputData, inputDataAConstant > inputDataBConstant);
	}


	// Greater than equal
	GPUFilter::constConst<FilterConditions::greaterEqual>(outputBuffer, inputDataAConstant, inputDataBConstant, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(&outputData, outputBuffer, DATA_ELEMENT_COUNT);

	if (std::is_integral<T>::value)
	{
		ASSERT_EQ(outputData, inputDataAConstant >= inputDataBConstant);
	}
	else
	{
		ASSERT_FLOAT_EQ(outputData, inputDataAConstant >= inputDataBConstant);
	}


	// Less than
	GPUFilter::constConst<FilterConditions::less>(outputBuffer, inputDataAConstant, inputDataBConstant, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(&outputData, outputBuffer, DATA_ELEMENT_COUNT);

	if (std::is_integral<T>::value)
	{
		ASSERT_EQ(outputData, inputDataAConstant < inputDataBConstant);
	}
	else
	{
		ASSERT_FLOAT_EQ(outputData, inputDataAConstant < inputDataBConstant);
	}


	// Less than equal
	GPUFilter::constConst<FilterConditions::lessEqual>(outputBuffer, inputDataAConstant, inputDataBConstant, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(&outputData, outputBuffer, DATA_ELEMENT_COUNT);

	if (std::is_integral<T>::value)
	{
		ASSERT_EQ(outputData, inputDataAConstant <= inputDataBConstant);
	}
	else
	{
		ASSERT_FLOAT_EQ(outputData, inputDataAConstant <= inputDataBConstant);
	}


	// Equal
	GPUFilter::constConst<FilterConditions::equal>(outputBuffer, inputDataAConstant, inputDataBConstant, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(&outputData, outputBuffer, DATA_ELEMENT_COUNT);

	if (std::is_integral<T>::value)
	{
		ASSERT_EQ(outputData, inputDataAConstant == inputDataBConstant);
	}
	else
	{
		ASSERT_FLOAT_EQ(outputData, inputDataAConstant == inputDataBConstant);
	}


	// Non equal
	GPUFilter::constConst<FilterConditions::notEqual>(outputBuffer, inputDataAConstant, inputDataBConstant, DATA_ELEMENT_COUNT);
	GPUMemory::copyDeviceToHost(&outputData, outputBuffer, DATA_ELEMENT_COUNT);

	if (std::is_integral<T>::value)
	{
		ASSERT_EQ(outputData, inputDataAConstant != inputDataBConstant);
	}
	else
	{
		ASSERT_FLOAT_EQ(outputData, inputDataAConstant != inputDataBConstant);
	}


	//////////////////////////////////////////////////////////////////////////////////////

	// Free buffers in GPU memory:
	GPUMemory::free(outputBuffer);
}


TEST(GPUFilterTests, FiltersConstConst)
{
	// Initialize CUDA context:
	Context::getInstance();

	testConstConst<int32_t>();
	testConstConst<int64_t>();
	testConstConst<float>();
}