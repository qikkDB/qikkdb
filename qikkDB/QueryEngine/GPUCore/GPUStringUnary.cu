#include "hip/hip_runtime.h"
#include "GPUStringUnary.cuh"

__device__ int64_t GetStringIndex(int64_t* indices, const int64_t i)
{
    return (i == 0) ? 0 : indices[i - 1];
}

__device__ int32_t GetStringLength(int64_t* indices, const int64_t i)
{
    return static_cast<int32_t>(indices[i] - GetStringIndex(indices, i));
}

__global__ void kernel_reverse_string(GPUMemory::GPUString outCol, GPUMemory::GPUString inCol, int32_t stringCount)
{
    const int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < stringCount; i += stride)
    {
        const int64_t firstCharIndex = GetStringIndex(inCol.stringIndices, i);
        const int64_t length = inCol.stringIndices[i] - firstCharIndex;
        const int64_t lastCharIndex = inCol.stringIndices[i] - 1;

        for (int32_t j = 0; j < length; j++)
        {
            outCol.allChars[firstCharIndex + j] = inCol.allChars[lastCharIndex - j];
        }
    }
}

template <>
void StringUnaryOpHierarchy::fixed::CallKernel<StringUnaryOpHierarchy::FixedLength::reverse>(
    GPUMemory::GPUString outCol,
    GPUMemory::GPUString input,
    int32_t stringCount,
    int64_t totalCharCount)
{
    Context& context = Context::getInstance();
    kernel_reverse_string<<<context.calcGridDim(stringCount), context.getBlockDim()>>>(outCol, input, stringCount);
}
